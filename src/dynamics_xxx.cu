#include "hip/hip_runtime.h"

////////////////////////////////////////
#include"pub.h"
#include"dynamics.h"
////////////////////////////////////////
//#include"your_own_library.h"
///////////////////////////////////////
#include"dynamics_xxx.h"

using namespace GUPS_NS;
using namespace DATA_NS;

DynamicsXxx::DynamicsXxx(){}
DynamicsXxx::~DynamicsXxx(){}

int DynamicsXxx::Initialize(){
  //para setting should be finished before or within this function
  string ss;
  x=1.0f; Vars["x"]>>=x;	
  return 0;
}

int DynamicsXxx::Calculate(){
  x=x+1.0f;
  return 0;
}

int DynamicsXxx::RunFunc(string funcName){
  if (funcName=="calculate") Calculate();
  return 0;
}

int DynamicsXxx::Fix(real progress){return 0;}

string DynamicsXxx::Get(string ss){
  string ans;
  if (ss=="x") return ans<<x;
  return "nan";
}
