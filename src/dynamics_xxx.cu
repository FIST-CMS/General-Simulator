#include "hip/hip_runtime.h"

////////////////////////////////////////
#include"pub.h"
#include"dynamics.h"
////////////////////////////////////////
//#include"your_own_library.h"
///////////////////////////////////////
#include"dynamics_xxx.h"

using namespace GS_NS;
using namespace DATA_NS;

DynamicsXxx::DynamicsXxx(){}
DynamicsXxx::~DynamicsXxx(){}

int DynamicsXxx::Initialize(){
  //para setting should be finished before or within this function
  string ss;
  Vars["x"]>>=x;	
  Matrix = &((*Datas)["matrix"]);
  return 0;
}

int DynamicsXxx::Calculate(){
  (*Matrix)=(*Matrix)*x;
  return 0;
}

int DynamicsXxx::RunFunc(string funcName){
  if (funcName=="calculate") Calculate();
  return 0;
}

int DynamicsXxx::Fix(real progress){return 0;}

string DynamicsXxx::Get(string ss){
  string ans;
  if (ss=="x") return ans<<x;
  if (ss=="sumofmatrix") return ans<<(Matrix->TotalHost());
  return "nan";
}
