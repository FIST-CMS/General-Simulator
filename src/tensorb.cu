#include "hip/hip_runtime.h"

#include"../include/datamain.th"
#include"tensorb.h"
using namespace GUPS_NS;
using namespace DATA_NS;

/*
Real part(Data<Real> &data,int p1){return data(p1);}
Real part(Data<Real> &data,int p1,int p2){return data(p1,p2);}
Real part(Data<Real> &data,int p1,int p2,int p3){return data(p1,p2,p3);}
Real part(Data<Real> &data,int p1,int p2,int p3,int p4){return data(p1,p2,p3,p4);}
Real part(Data<Real> &data,int p1,int p2,int p3,int p4,int p5){return data(p1,p2,p3,p4,p5);}
Real part(Data<Real> &data,int p1,int p2,int p3,int p4,int p5,int p6){return data(p1,p2,p3,p4,p5,p6);}

Complex part(Data<Complex> &data,int p1){return data(p1);}
Complex part(Data<Complex> &data,int p1,int p2){return data(p1,p2);}
Complex part(Data<Complex> &data,int p1,int p2,int p3){return data(p1,p2,p3);}
Complex part(Data<Complex> &data,int p1,int p2,int p3,int p4){return data(p1,p2,p3,p4);}
Complex part(Data<Complex> &data,int p1,int p2,int p3,int p4,int p5){return data(p1,p2,p3,p4,p5);}
Complex part(Data<Complex> &data,int p1,int p2,int p3,int p4,int p5,int p6){return data(p1,p2,p3,p4,p5,p6);}
*/

real TensorB::VF(int x,int n,float dx){
  int tem; if (x<n/2) tem=x; else tem=x-n;
  return 2.0*3.14*dx/n*tem; 
}

int  TensorB::UnitVector(real dx,real dy, real dz){
  int nx= _g.Dimension[1], ny= _g.Dimension[2], nz= _g.Dimension[3];
  Real gx,gy,gz,mo;
  for (int ix=0;ix<nx;ix++){
    for (int iy=0;iy<ny;iy++){
	 for (int iz=0;iz<nz;iz++){
	   _g(ix,iy,iz,0)=gx=VF(ix,nx,dx);
	   _g(ix,iy,iz,1)=gy=VF(iy,ny,dy);
	   _g(ix,iy,iz,2)=gz=VF(iz,nz,dz);
	   _gSquare(ix,iy,iz)=(gx^2)+(gy^2)+(gz^2);
	   mo = sqrt(_gSquare(ix,iy,iz));
	   if (mo > 0.0000005){
		 unitVector(ix,iy,iz,0)=gx/mo;
		 unitVector(ix,iy,iz,1)=gy/mo;
		 unitVector(ix,iy,iz,2)=gz/mo;
	   }else{
		 unitVector(ix,iy,iz,0)=0;
		 unitVector(ix,iy,iz,1)=0;
		 unitVector(ix,iy,iz,2)=0;
	   }
	 }
    }
  }
  return 0;
}

TensorB::TensorB(){
}
  
int TensorB::InitB(int variantN, int lx, int ly, int lz,real a1, real a2, real a3, Data<Real> &tensor ){

  SetCalPos(Data_HOST);
  Init(5,variantN, variantN, lx, ly, lz,Data_HOST_DEV);
  _g.Init(4,lx,ly,lz,3,Data_HOST_DEV);
  _gSquare.Init(3,lx,ly,lz,Data_HOST_DEV);
  unitVector.Init(4,lx,ly,lz,3,Data_HOST_DEV);
  sigma.Init(3,variantN,3,3,Data_HOST_DEV);
  //unitvector
  UnitVector(a1,a2,a3);
  
  //stress  sigma(sa,j,i)
  cijkl.Init(4,3,3,3,3); cijkl=0.f;
  cijkl(0,0,0,0) =C00; cijkl(1,1,1,1) =C00; cijkl(2,2,2,2) =C00;
  cijkl(0,0,1,1) =C01; cijkl(1,1,2,2) =C01; cijkl(2,2,0,0) =C01;
  cijkl(0,1,0,1) =C33; cijkl(1,2,1,2) =C33; cijkl(2,0,2,0) =C33;
  cijkl(1,1,0,0) =C01; cijkl(2,2,1,1) =C01; cijkl(0,0,2,2) =C01;
  cijkl(1,0,1,0) =C33; cijkl(0,1,1,0) =C33; cijkl(1,0,0,1) =C33;
  cijkl(2,1,2,1) =C33; cijkl(2,1,1,2) =C33; cijkl(1,2,2,1) =C33;
  cijkl(0,2,0,2) =C33; cijkl(0,2,2,0) =C33; cijkl(2,0,0,2) =C33;			// 

  SetCalPos(Data_HOST);
  sigma=0.f;
  for (int sa=0;sa<variantN;sa++){
    for (int i=0;i<3;i++)
	 for (int j=0;j<3;j++){
	   for (int k=0;k<3;k++)
		for (int l=0;l<3;l++){
		  sigma(sa,i,j)+=cijkl(i,j,k,l)*tensor(sa,k,l);
		}
	 }
  }

  //-------------------------------------------------------------------
  //omega
  omega.Init(2,3,3,Data_HOST);
  iomega.Init(2,3,3,Data_HOST);
  temp.Init(2,3,3,Data_HOST);
  for (int ix=0;ix<lx;ix++)
    for (int iy=0;iy<ly;iy++)
	  for (int iz=0;iz<lz;iz++){

		for (int i=0;i<3;i++)
		  for (int j=0;j<3;j++){
			iomega(i,j)=0.0;
			for (int k=0;k<3;k++){
			  for (int l=0;l<3;l++){
				iomega(i,j) = iomega(i,j) + cijkl(i,k,l,j)*unitVector(ix,iy,iz,k)*unitVector(ix,iy,iz,l);
			  }
			}
		  }

		if (_gSquare(ix,iy,iz)==0){ omega=0;}
		else GaussCMInverse(iomega.Arr,temp.Arr,omega.Arr,3);

		Real term=0;
		for (int sa=0;sa<variantN;sa++){
		  for (int sap=0;sap<variantN;sap++){
			int index[5]={sa,sap,ix,iy,iz};
			Part(index) =0.0f;
			for (int i=0;i<3;i++)
			  for (int j=0;j<3;j++)
				for (int k=0;k<3;k++)
				  for (int l=0;l<3;l++){ // (/ 31.4 (sqrt 1001)) (/ 3.92 (sqrt 1001))
					term=-unitVector(ix,iy,iz,i)*sigma(sa,i,j)*omega(j,k)*sigma(sap,k,l)*unitVector(ix,iy,iz,l);
					Part(index) +=
					  cijkl(i,j,k,l)*tensor(sa,i,j)*tensor(sap,k,l)
					  +term;
				  }
		  }
		}
	  }
  HostToDevice();
  _g.HostToDevice();
  _gSquare.HostToDevice();
  unitVector.HostToDevice();
  return 0;
  
}
