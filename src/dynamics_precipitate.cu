#include "hip/hip_runtime.h"
#define DEBUG 0
////////////////////////////////////////
#include"pub.h"
#include"dynamics.h"
////////////////////////////////////////
#include<hiprand.h>
#include<hipfft/hipfft.h>
#include"random.h"
#include"gtensorb.h"


#include"dynamics_precipitate.h"


using namespace GS_NS;
using namespace DATA_NS;

int Dynamics_precipitate::Initialize(){
  //para setting should be finished before or within this function
  string ss;
  ss=(*Vars)["gridsize"];    			if (ss!="") ss>>nx>>ny>>nz>>dx>>dy>>dz;
  ss=(*Vars)["deltatime"];             if (ss!="") ss>>DeltaTime;else DeltaTime=0.1;
  ss=(*Vars)["coefficient"];           if (ss!="") ss>>A[1]>>A[2]>>A[3]>>A[4]>>A[5]>>A[6]>>A[7]; else {A[1]=54.0;A[2]=-17.0;A[3]=7.0;A[4]=2.5;A[5]=0.2;A[6]=0.2;A[7]=0.2;}
  ss=(*Vars)["arfi"]; if (ss!="") ss>>Arfi; else Arfi=300.0f;
  ss=(*Vars)["beta"]; if (ss!="") ss>>Beta; else Beta=300.0f;
  ss=(*Vars)["meta"]; if (ss!="") ss>>Meta; else Meta=0.4f;
  ss=(*Vars)["lpp"];  if (ss!="") ss>>Lpp; else Lpp = 0.4f;
  ss=(*Vars)["xi"];   if (ss!="") ss>>Xi; else Xi=400.f;
  ss=(*Vars)["concentration"]; if (ss!="") ss>>Concentration1>>Concentration2; else { Concentration1=0.1; Concentration2=0.44; }
  ss=(*Vars)["weightnoise"]; if (ss!="") { ss>>weightEtaNoise; weightConNoise = weightEtaNoise; } else{ weightEtaNoise= 1.0f; weightConNoise= 1.0f;}
  ////////////////////////////////////////////////////////////
  StrainTensor = &((*Datas)["varianttensor"]);
  if (StrainTensor->Arr == NULL){
	GV<0>::LogAndError<<"Error: variants' strain tensor deoos not set while initialize dynamics\n";
	return -1;
  }
  VariantN=StrainTensor->Dimension[1]; 
  // it is called to initialize the --run-- function
  // allocate memory initial size and default values
  ////////////////////////////////////////////////////////////
  //Init(3,nx,ny,nz,Data_NONE);
  SetCalPos(Data_HOST_DEV);
  Concentration = &((*Datas)["concentration"]);
  if ( Concentration->Arr == NULL ){
	Concentration->Init(3,nx,ny,nz,Data_HOST_DEV);
	SetCalPos(Data_DEV);
	(*Concentration)=Concentration1+(Concentration2-Concentration1)/3; 
  }else{ Concentration->HostToDevice(); }
  Eta = &((*Datas)["eta"]); // may create here
  if ( Eta->Arr == NULL ){
	Eta->Init(4,VariantN,nx,ny,nz,Data_HOST_DEV);
	SetCalPos(Data_DEV);
	(*Eta)=0.0f; 
  }else{ Eta->HostToDevice(); }
  ////////////////////////////////////////////////////////////
  real C00=3.5,C01=1.5,C33=1.0;
  ss=(*Vars)["modulus"];if (ss!="") ss>>C00>>C01>>C33;
  Data<Real> cijkl; SetCalPos(Data_HOST);
  cijkl.Init(4,3,3,3,3,Data_HOST_DEV); cijkl=0.f;
  cijkl(0,0,0,0) =C00; cijkl(1,1,1,1) =C00; cijkl(2,2,2,2) =C00;
  cijkl(0,0,1,1) =C01; cijkl(1,1,2,2) =C01; cijkl(2,2,0,0) =C01;
  cijkl(0,1,0,1) =C33; cijkl(1,2,1,2) =C33; cijkl(2,0,2,0) =C33;
  cijkl(1,1,0,0) =C01; cijkl(2,2,1,1) =C01; cijkl(0,0,2,2) =C01;
  cijkl(1,0,1,0) =C33; cijkl(0,1,1,0) =C33; cijkl(1,0,0,1) =C33;
  cijkl(2,1,2,1) =C33; cijkl(2,1,1,2) =C33; cijkl(1,2,2,1) =C33;
  cijkl(0,2,0,2) =C33; cijkl(0,2,2,0) =C33; cijkl(2,0,0,2) =C33;			// 
  cijkl.HostToDevice();
  Data<Real> vstrain(3,2*VariantN,3,3,Data_HOST_DEV);
  for (int i=0; i<vstrain.N(); i++)
	vstrain.Arr[i]=StrainTensor->Arr[i%(VariantN*3*3)];
  vstrain.HostToDevice();
  GV<0>::LogAndError<<"calculating space structure tensor \n";
  B.InitB(VariantN,VariantN,nx,ny,nz,dx.Re,dy.Re,dz.Re,vstrain,cijkl); 
  GV<0>::LogAndError<<"finished calculating space structure tensor \n";
  ////////////////////////////////////////////////////////////
  int rank=3,ns[3]={nx,ny,nz},dist=nx*ny*nz,stride=1;
  GV<0>::LogAndError<<"create cuda fft plan \n";
  if (hipfftPlanMany(&plan_vn,rank,ns,ns,stride,dist,ns,stride,dist,HIPFFT_C2C,VariantN)==HIPFFT_SUCCESS)
	GV<0>::LogAndError<<"finish creating cuda fft plan vn \n";
  else GV<0>::LogAndError<<"cuda fft plan vn fails to create\n";
  if (hipfftPlan3d(&plan_n,nx,ny,nz,HIPFFT_C2C)==HIPFFT_SUCCESS)
	GV<0>::LogAndError<<"cuda fft plan n created\n";
  else GV<0>::LogAndError<<"cuda fft plan n fails to create\n";

  int ndim[5]={3,nx,ny,nz};
  int vndim[6]={4,VariantN,nx,ny,nz};

  EtaLFE.Init(vndim,Data_HOST_DEV);
  EtaLFE_CT.Init(vndim,Data_HOST_DEV);
  ConLFE.Init(ndim,Data_HOST_DEV);
  EtaLFE_CT.Init(vndim,Data_HOST_DEV);

  ElasticEnergy.Init(ndim,Data_HOST_DEV);
  ElasticForce.Init(vndim,Data_HOST_DEV);

  Con_CT.Init(ndim,Data_HOST_DEV);
  ConRan_CT.Init(ndim,Data_HOST_DEV);
  RTermCon_CT.Init(ndim,Data_HOST_DEV);

  Eta_CT.Init(vndim,Data_HOST_DEV);
  EtaRan_CT.Init(vndim,Data_HOST_DEV);
  RTermEta_CT.Init(vndim,Data_HOST_DEV);
  ElasticTerm_CT.Init(vndim,Data_HOST_DEV);
  
  Noise_vn.InitRandom(4,VariantN,nx,ny,nz, 0, 0.001, 0,0);
  Noise_n.InitRandom(3,nx,ny,nz, 0, 0.001, 0,0);


  return 0;

}

Dynamics_precipitate::Dynamics_precipitate(){}
Dynamics_precipitate::~Dynamics_precipitate(){
  if (plan_n) hipfftDestroy(plan_n);
  if (plan_vn) hipfftDestroy(plan_vn);
}

__global__ void LocalConFreeEnergyCalculate_Diffuse_Kernel(Real * ConLFE, Real * Eta, Real* Concentration, Real A1,Real A2, Real Concentration1 , int VariantN ){
  int nx = gridDim.x, ny = gridDim.y, nz = blockDim.x;
  int x=blockIdx.x, y= blockIdx.y, z=threadIdx.x;
  int tid = (x*ny + y)*nz+z; // (x,y,z)
  int nn= nx*ny*nz;
  ConLFE[ tid ] = 0.f;
  for ( int va=0 ; va<VariantN; va++ )
	ConLFE[ tid]+= (A2/2.0f) * ( Eta[ tid + va *nn ]^2 );
  ConLFE[ tid ]+= A1 * ( Concentration[tid] - Concentration1);
}

int Dynamics_precipitate::LocalConFreeEnergyCalculate(){
  dim3 bn(nx,ny), tn(nz);
  LocalConFreeEnergyCalculate_Diffuse_Kernel<<<bn,tn>>>
	(ConLFE.Arr_dev, Eta->Arr_dev, Concentration->Arr_dev, A[1], A[2], Concentration1, VariantN);
  return 0;
}

__global__ void LocalEtaFreeEnergyCalculate_Diffuse_Kernel
(Real* EtaLFE, Real* Eta, Real* Concentration,
 Real A1,Real A2,Real A3, Real A4, Real A5, Real A6, Real A7, Real Concentration2 ){
  int nx = gridDim.x, ny = gridDim.y, nv =gridDim.z, nz = blockDim.x;
  int x  = blockIdx.x, y = blockIdx.y, v =blockIdx.z, z = threadIdx.x;
  int nn = nx * ny * nz;
  int tid  = ( x * ny + y) * nz + z;  //(x,y,z)
  int ntid = (( v * nx + x )* ny + y) *nz + z; //(v,x,y,z)
  Real term2 =0, term4 =0, term22 =0;
  EtaLFE[ ntid ] = 0.f;
  for (int i=0; i<nv; i++)
	if (i!=v){
	term2+= (Eta[tid+i*nn]^2);
	term4+= (Eta[tid+i*nn]^4);
  }
  for (int i=0; i<nv; i++)
	for (int j=0; j<nv; j++)
	  if ( v!=i && v!=j && i!= j)
		term22+= (Eta[i*nn + tid]^2)*(Eta[j*nn + tid ]^2);
  EtaLFE[ ntid ] =
	(2.f * A5 * Eta[ntid] + 4.f * A6 * (Eta[ntid] ^3))* term2
	+(2.0f * A6 * Eta[ntid]) * term4
	+(2.0f * A7 * Eta[ntid]) * term22
	+(A2 * Eta[ntid] * ( Concentration[tid] - Concentration2 ))
	-(A3 * (Eta[ntid]^3))
	+(A4 * (Eta[ntid]^5));
}

int Dynamics_precipitate::LocalEtaFreeEnergyCalculate(){
  dim3 bn(nx,ny,VariantN), tn(nz,1,1);
  LocalEtaFreeEnergyCalculate_Diffuse_Kernel<<<bn,tn>>>(EtaLFE.Arr_dev,Eta->Arr_dev,Concentration->Arr_dev,A[1],A[2],A[3],A[4],A[5],A[6],A[7],Concentration2);
  return 0;
}

__global__ void ElasticEnergyForceCalculate_Diffuse_Kernel(Complex *RTerm,Complex*Eta_sq,Real* B){
  int VariantN=gridDim.z;
  int nx= gridDim.x, ny= gridDim.y, nz = blockDim.x;
  int x = blockIdx.x, y = blockIdx.y, z = threadIdx.x, v = blockIdx.z;
  RTerm[((v*nx+x)*ny+y)*nz+z]=0.f;
  for (int i=0;i<VariantN;i++){
	RTerm[((v*nx+x)*ny+y)*nz+z]+=B[(((v*VariantN+i)*nx+x)*ny+y)*nz+z]* Eta_sq[((i*nx+x)*ny+y)*nz+z];
  }
}

int Dynamics_precipitate::ElasticForceCalculate(){
  SetCalPos(Data_DEV);
  Eta_CT=(*Eta)*(*Eta); //Store it in the buffer area
  ///////////////////////////////////////////////////////////////
  hipfftExecC2C(plan_vn,(hipfftComplex*)Eta_CT.Arr_dev,(hipfftComplex*)Eta_CT.Arr_dev,HIPFFT_FORWARD);
  Eta_CT = Eta_CT/Eta_CT.N()*VariantN; // equavilent to /(nx*ny*nz)
  ///////////////////////////////////////////////////////////////
  dim3 bn(nx,ny,VariantN);
  dim3 tn(nz);
  ElasticEnergyForceCalculate_Diffuse_Kernel<<<bn,tn>>>
	(RTermEta_CT.Arr_dev,Eta_CT.Arr_dev,B.Arr_dev);
  hipfftExecC2C(plan_vn,(hipfftComplex*)RTermEta_CT.Arr_dev,(hipfftComplex*)RTermEta_CT.Arr_dev,HIPFFT_BACKWARD);
  ///////////////////////////////////////////////////////////////
  ElasticForce = 2.0f* RTermEta_CT* (*Eta); // the coefficient 2.0f is ....????
  return 0;
}

__global__ void ConcentrationUpdate_Diffuse_Kernel(Complex *Con_CT, Complex* ConLFE_CT, Complex* ConRan_CT, Real* gSquare, Real dt, Real meta,Real beta, Real weightConNoise){
  //int nx = gridDim.x;
  int ny = gridDim.y, nz = blockDim.x;
  int x=blockIdx.x, y= blockIdx.y, z=threadIdx.x;
  int tid = (x*ny + y)*nz+z; //(x,y,z)
  //int nn= nx*ny*nz;
  Con_CT[tid] =
	( Con_CT[tid] - meta * gSquare[tid] *dt * ( ConLFE_CT[tid] + 0.0001f*weightConNoise * ConRan_CT[tid] ) )
	/ ( 1.0f + dt * meta * beta * gSquare[tid] * gSquare[tid] );
}

int Dynamics_precipitate::ConcentrationUpdate(){
  SetCalPos(Data_DEV);
  Noise_n.NewNormal_device(); hipDeviceSynchronize();
  set_device(ConRan_CT.Arr_dev,Noise_n.Arr_dev, ConRan_CT.N());
  Con_CT = *Concentration; ///real((nx*ny*nz));
  ConLFE_CT = ConLFE;
  if (DEBUG){ConRan_CT.DeviceToHost(); Con_CT.DeviceToHost(); ConLFE_CT.DeviceToHost(); }
  ///////////////////////////////////////////////////////////
  hipfftExecC2C(plan_n,(hipfftComplex*)ConRan_CT.Arr_dev,(hipfftComplex*)ConRan_CT.Arr_dev, HIPFFT_FORWARD); 
  hipfftExecC2C(plan_n,(hipfftComplex*)Con_CT.Arr_dev,(hipfftComplex*)Con_CT.Arr_dev, HIPFFT_FORWARD); 
  hipfftExecC2C(plan_n,(hipfftComplex*)ConLFE_CT.Arr_dev,(hipfftComplex*)ConLFE_CT.Arr_dev, HIPFFT_FORWARD); 
  //////////////////
  divi_device(Con_CT.Arr_dev,Con_CT.Arr_dev,real(nx*ny*nz),Con_CT.N());
  divi_device(ConRan_CT.Arr_dev, ConRan_CT.Arr_dev,real(nx*ny*nz),ConRan_CT.N());
  divi_device(ConLFE_CT.Arr_dev, ConLFE_CT.Arr_dev,real(nx*ny*nz),ConLFE_CT.N());
  if (DEBUG){ConRan_CT.DeviceToHost(); Con_CT.DeviceToHost(); ConLFE_CT.DeviceToHost(); }
  if (DEBUG) { ConRan_CT=0.f; }
  /////////////////
  ///////////////////////////////////////////////////////////
  // the factor nx*ny*nz within the transformation
  dim3 bn(nx,ny);
  dim3 tn(nz);
  ConcentrationUpdate_Diffuse_Kernel<<<bn,tn>>>(Con_CT.Arr_dev,ConLFE_CT.Arr_dev, ConRan_CT.Arr_dev, B._gSquare.Arr_dev, DeltaTime, Meta ,Beta, weightConNoise);
  hipfftExecC2C(plan_n,(hipfftComplex*)Con_CT.Arr_dev, (hipfftComplex*)Con_CT.Arr_dev, HIPFFT_BACKWARD);
  ///////////////////////////////////////////////////////////
  (*Concentration) = Con_CT; // / real(sqrt(nx*ny*nz)); // be done before the update
  if (DEBUG) { Concentration->DeviceToHost(); }
  return 0;
}

__global__ void EtaUpdate_Diffuse_Kernel(
	Complex* Eta, Complex* ElasticTerm, Complex* EtaRan, Real* gSquare,
	Real DeltaTime, Real weightEtaNoise,Real lpp,Real arfi) {
  //int nv = gridDim.z;
  int nx=gridDim.x,ny=gridDim.y,nz = blockDim.x;
  int v = blockIdx.z;
  int x=blockIdx.x, y=blockIdx.y, z=threadIdx.x;
  int tid = (( v* nx+x )*ny + y)*nz+z; //(v,x,y,z)
  int ntid = ( x*ny + y )*nz+ z;       //(x,y,z)

  Eta[tid]=
	(Eta[tid]
	 - DeltaTime * lpp *( ElasticTerm[tid] + 0.0001f*weightEtaNoise* EtaRan[tid]))  
	/(1.0f + DeltaTime* lpp* arfi * gSquare[ntid] );
}

int Dynamics_precipitate::EtaUpdate(){
  SetCalPos(Data_DEV);
  Noise_vn.NewNormal_device();
  set_device(EtaRan_CT.Arr_dev, Noise_vn.Arr_dev, EtaRan_CT.N());
  EtaRan_CT = EtaRan_CT; ///real(sqrt(nx*ny*nz));
  Eta_CT = (*Eta);///real(sqrt(nx*ny*nz));
  ///////////////////////////////////////////////////////////
  ElasticTerm_CT = ( Xi * ElasticForce + EtaLFE); ///real(sqrt(nx*ny*nz));
  ///////////////////////////////////////////////////////////
  if (DEBUG) { Eta_CT.DeviceToHost(); ElasticTerm_CT.DeviceToHost();ElasticForce.DeviceToHost(); EtaLFE.DeviceToHost(); }
  hipfftExecC2C(plan_vn, (hipfftComplex*)EtaRan_CT.Arr_dev,(hipfftComplex*)EtaRan_CT.Arr_dev,HIPFFT_FORWARD);
  hipfftExecC2C(plan_vn, (hipfftComplex*)Eta_CT.Arr_dev, (hipfftComplex*)Eta_CT.Arr_dev, HIPFFT_FORWARD);
  hipfftExecC2C(plan_vn, (hipfftComplex*)ElasticTerm_CT.Arr_dev, (hipfftComplex*) ElasticTerm_CT.Arr_dev, HIPFFT_FORWARD);
  divi_device(EtaRan_CT.Arr_dev , EtaRan_CT.Arr_dev,real(nx*ny*nz),EtaRan_CT.N());
  divi_device(Eta_CT.Arr_dev,Eta_CT.Arr_dev, real(nx*ny*nz),Eta_CT.N());
  divi_device(ElasticTerm_CT.Arr_dev, ElasticTerm_CT.Arr_dev,real(nx*ny*nz),ElasticTerm_CT.N());
  if (DEBUG) { Eta_CT.DeviceToHost(); ElasticTerm_CT.DeviceToHost(); }
  if (DEBUG) { SetCalPos(Data_DEV); EtaRan_CT=0.f; }
  ///////////////////////////////////////////////////////////
  dim3 bn(nx,ny,VariantN), tn(nz);
  EtaUpdate_Diffuse_Kernel<<<bn,tn>>>
	(Eta_CT.Arr_dev, ElasticTerm_CT.Arr_dev , EtaRan_CT.Arr_dev, B._gSquare.Arr_dev,
	 DeltaTime, weightEtaNoise, Lpp, Arfi );
  ///////////////////////////////////////////////////////////
  hipfftExecC2C(plan_vn, (hipfftComplex*)Eta_CT.Arr_dev, (hipfftComplex*)Eta_CT.Arr_dev, HIPFFT_BACKWARD);
  ///////////////////////////////////////////////////////////
  (*Eta)=Eta_CT;
  if (DEBUG) { Eta->DeviceToHost(); }
  return 0;
}


int Dynamics_precipitate::Calculate(){
  string ss;
  (*Vars)["temperature"]>>=Temperature; 
  LocalConFreeEnergyCalculate();
  LocalEtaFreeEnergyCalculate();
  ElasticForceCalculate();
  ////////////////////////////////
  ConcentrationUpdate();
  EtaUpdate();
  ////////////////////////////////
  return 0;
}

int Dynamics_precipitate::RunFunc(string funcName){ return 0;}


int Dynamics_precipitate::Fix(real progress){
  string ss,mode;
  ss = (*Vars)["fix"];
  while( ss!= "" ){
	ss>>mode;
	if      (mode=="temperature"	){
	  real st,et; //start and end temperature
	  ss>>st>>et;
	  ((*Vars)["temperature"])<<=(st+ progress*(et- st));
	} else if (mode=="pressure"		){ 
	} else{
	  GV<0>::LogAndError<<"Error: fix style "<<mode<<" does not find!\n";
	}
  }

  return 0;
}

string Dynamics_precipitate::Get(string ss){ // return the statistic info.
  string var; ss>>var;
  if (var == "temperature") return ToString(Temperature); 
  if (var == "eta_average") return ToString(Eta->TotalDevice()/Eta->N());
  else return "nan";
}
