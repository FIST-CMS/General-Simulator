#include "hip/hip_runtime.h"

////////////////////////////////////////
#include"pub.h"
#include"dynamics.h"
////////////////////////////////////////
//#include"your_own_library.h"
///////////////////////////////////////
#include"dynamics_multi.h"

using namespace GS_NS;
using namespace DATA_NS;

Dynamics_multi::Dynamics_multi(){}

int Dynamics_multi::Initialize(){
  return 0;
}

int Dynamics_multi::Calculate(){
  return 0;
}

int Dynamics_multi::RunFunc(string func){
  string fname; func>>fname;
  if (fname=="matrix_multi") return Matrix_multi(func);
  return Code_COMMAND_UNKNOW;
}

int Dynamics_multi::Fix(real progress){return 0;}

string Dynamics_multi::Get(string ss){
  return "nan";
}

Dynamics_multi::~Dynamics_multi(){}

__global__ void Dynamics_multi_kernel_matrix_multi
( Real *mat1, Real*mat2,Real *mat3,
 int d_mid){
  int ny=blockDim.x;
  int x=blockIdx.x,y=threadIdx.x;
  mat3[x*ny+y]=0.0f;
  for (int i=0; i<d_mid; i++)
	mat3[x*ny+y]+=mat1[x*d_mid+i]*mat2[i*ny+y];
}

int Dynamics_multi::Matrix_multi(string para){
  Data<Real> *Mat1,*Mat2,*Mat3; string matname[3];
  ///////////////////////////////////////////////////////////////
  para>>matname[0]>>matname[1]>>matname[2];
  Mat1 = &((*Datas)[matname[0]]);
  Mat2 = &((*Datas)[matname[1]]);
  Mat3 = &((*Datas)[matname[2]]);
  ///////////////////////////////////////////////////////////////
  Mat1->HostToDevice(); Mat2->HostToDevice();
  Mat3->Init(2,Mat1->Dimension[1],Mat2->Dimension[2],Data_HOST_DEV);
  dim3 bn(Mat1->Dimension[1]),tn(Mat2->Dimension[2]);
  Dynamics_multi_kernel_matrix_multi<<<bn,tn>>>
	(Mat1->Arr_dev,Mat2->Arr_dev,Mat3->Arr_dev,
	 Mat1->Dimension[2]);
  Mat3->DeviceToHost();
  ///////////////////////////////////////////////////////////////
  return 0;
}

