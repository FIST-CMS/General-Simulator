#include "hip/hip_runtime.h"

////////////////////////////////////////
#include"pub.h"
#include"dynamics.h"
////////////////////////////////////////

#include"dynamics_cores.h"

using namespace GS_NS;
using namespace DATA_NS;
/*
  main calculation:
 */

int Dynamics_cores::Initialize(){
  //para setting should be finished before or within this function
  string ss;
  ss=(*Vars)["gridsize"];if (ss!="") { ss>>nx>>ny>>nz>>dx>>dy>>dz;} else {nx=ny=nz=16; dx=dy=dz=0.1;}
  ss=(*Vars)["variantn"];if (ss!="") { ss>>VariantN;} else {VariantN=4;}
  ss=(*Vars)["coresn"];	if (ss!="") { ss>>CoresN; }else { CoresN=5;}
  ss=(*Vars)["radius"];	if (ss!="") { ss>>Radius; }else { Radius=5;}
  ss=(*Vars)["concentration"];			if (ss!="") { ss>>Concentration1>>Concentration2; }else {Concentration1=0.2f; Concentration2 = 0.44f;}
  ss=(*Vars)["method"];  if (ss!="") { ss>>Method; } else { Method = "random"; }
  // it is called to initialize the --run-- function
  ///////////////////////////////////////////////////
  Eta = &((*Datas)["eta"]); // may create here
  Concentration = &((*Datas)["concentration"]); // may create here
  Eta->Init(4,VariantN,nx,ny,nz,Data_HOST);
  Concentration->Init(3,nx,ny,nz,Data_HOST);

  Cores.Init(2,CoresN,3,Data_HOST);
  Mark.Init(3,nx,ny,nz,Data_HOST);

  SetCalPos(Data_HOST);
  Mark=false;
  (*Concentration)=Concentration1;
  (*Eta)=0.f;

  return 0;

}

Dynamics_cores::Dynamics_cores(){}
Dynamics_cores::~Dynamics_cores(){}

int Dynamics_cores::RandomCores(){
  SetCalPos(Data_HOST);
  Mark=false;
  (*Concentration)=Concentration1;
  (*Eta)=0.f;
  for (int cn=0;cn<CoresN; cn++){
	int ox,oy,oz;
	for (int j=0;j< 100000;j++){
	  ox=random()%nx; oy= random()%ny; oz=random()%nz;
	  if (!Mark(ox,oy,oz)) break;
	}
	Cores(cn,0)=ox; Cores(cn,1)=oy; Cores(cn,2)=oz;
	int vtype=random()%VariantN;
	for (int i=0;i<nx; i++)
	  for (int j=0;j<ny; j++)
		for (int k=0;k<nz; k++)
		  if (
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2)  ||
			  pow(0.0f+i-ox+nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
			  pow(0.0f+i-ox-nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy+ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy-ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz+nz,2)<=pow(0.0f+Radius,2) || 
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz-nz,2)<=pow(0.0f+Radius,2) 
			  ){
			Mark(i,j,k)=true;
			(*Concentration)(i,j,k)=Concentration2;
			(*Eta)(vtype,i,j,k)=1.0f;
		  }
  }
  return 0;
}

int Dynamics_cores::RegularCores1D(){
  for (int cn=0;cn<CoresN; cn++){
	int ox,oy,oz;
	ox=nx/2; oy=ny/2; oz= nz/CoresN/2+nx/CoresN*cn;
	Cores(cn,0)=ox; Cores(cn,1)=oy; Cores(cn,2)=oz;
	int vtype=(int)floor(random()%VariantN);
	vtype = cn % VariantN; // for debug????
	for (int i=0;i<nx; i++)
	  for (int j=0;j<ny; j++)
		for (int k=0;k<nz; k++)
		  if (
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2)  ||
			  pow(0.0f+i-ox+nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
			  pow(0.0f+i-ox-nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy+ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy-ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz+nz,2)<=pow(0.0f+Radius,2) || 
			  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz-nz,2)<=pow(0.0f+Radius,2) 
			  ){
			Mark(i,j,k)=true;
			(*Concentration)(i,j,k)=Concentration2;
			(*Eta)(vtype,i,j,k)=1.0f;
		  }
  }
  return 0;
}

int Dynamics_cores::RegularCores2D(){
  for (int cn1=0;cn1<CoresN; cn1++){
	for (int cn2=0;cn2<CoresN; cn2++){
	  int ox,oy,oz;
	  ox=nx/2;
	  oy= ny/CoresN/2+ny/CoresN*cn2;
	  oz= nz/CoresN/2+nx/CoresN*cn1;
	  Cores(cn1*CoresN+cn2,0)=ox; Cores(cn1*CoresN+cn2,1)=oy; Cores(cn1*CoresN+cn2,2)=oz;
	  int vtype=(int)floor(random()*VariantN);
	  for (int i=0;i<nx; i++)
		for (int j=0;j<ny; j++)
		  for (int k=0;k<nz; k++)
			if (
				pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2)  ||
				pow(0.0f+i-ox+nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
				pow(0.0f+i-ox-nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
				pow(0.0f+i-ox,2)+pow(0.0f+j-oy+ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
				pow(0.0f+i-ox,2)+pow(0.0f+j-oy-ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
				pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz+nz,2)<=pow(0.0f+Radius,2) || 
				pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz-nz,2)<=pow(0.0f+Radius,2) 
				){
			  Mark(i,j,k)=true;
			  (*Concentration)(i,j,k)=Concentration2;
			  (*Eta)(vtype,i,j,k)=1.0f;
			}
	}
  }
  return 0;
}
int Dynamics_cores::RegularCores3D(){
  for (int cn1=0;cn1<CoresN; cn1++){
	for (int cn2=0;cn2<CoresN; cn2++){
	  for (int cn3=0;cn3<CoresN; cn3++){
		int ox,oy,oz;
		ox= nx/CoresN/2+nx/CoresN*cn3;
		oy= ny/CoresN/2+ny/CoresN*cn2;
		oz= nz/CoresN/2+nx/CoresN*cn1;
		Cores((cn1*CoresN+cn2)*CoresN+cn3,0)=ox; Cores((cn1*CoresN+cn2)*CoresN+cn3,1)=oy; Cores((cn1*CoresN+cn2)*CoresN+cn3,2)=oz;
		int vtype=(int)floor(random()*VariantN);
		for (int i=0;i<nx; i++)
		  for (int j=0;j<ny; j++)
			for (int k=0;k<nz; k++)
			  if (
				  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2)  ||
				  pow(0.0f+i-ox+nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
				  pow(0.0f+i-ox-nx,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
				  pow(0.0f+i-ox,2)+pow(0.0f+j-oy+ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) || 
				  pow(0.0f+i-ox,2)+pow(0.0f+j-oy-ny,2)+pow(0.0f+k-oz,2)<=pow(0.0f+Radius,2) ||
				  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz+nz,2)<=pow(0.0f+Radius,2) || 
				  pow(0.0f+i-ox,2)+pow(0.0f+j-oy,2)+pow(0.0f+k-oz-nz,2)<=pow(0.0f+Radius,2) 
				  ){
				Mark(i,j,k)=true;
				(*Concentration)(i,j,k)=Concentration2;
				(*Eta)(vtype,i,j,k)=1.0f;
			  }
	  }
	}
  }
  return 0;
}

int Dynamics_cores::Calculate(){
  if (Method == "random") RandomCores();
  else if (Method =="regular") RegularCores1D();
  else if (Method =="regular2d") RegularCores2D();
  else if (Method =="regular3d") RegularCores3D();

  return 0;
}

int Dynamics_cores::RunFunc(string funcName){return 0;}

int Dynamics_cores::Fix(real progress){return 0;}

string Dynamics_cores::Get(string ss){ // return the statistic info.
  string var; ss>>var;
  return "nan";
}
