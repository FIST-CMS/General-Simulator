
#include"pub.h"
#include"pub_main.h"
#include"input.h"

using namespace std;
using namespace GS_NS;

INPUT::INPUT(){
  Vars.Vars_gs = Gs.Vars;
}
INPUT::~INPUT(){}

int INPUT::Phrasing(string script){
  int err=0;
  string ss,line;
  while (getline(ss,script)){
	line = ss;
	////////////////////
	while (ss[0]==' ') ss.erase(0,1); //delete the heading space of each line
    if (ss=="") continue;
	///////////////////////
    string command; ss>>command;
	if (command[0]==':'){ ss=command.substr(1)+" "+ss; command="expr";}
	Vars.ReplaceExpr(ss);  // all to be sub
	///////////////////////
	if 		(command== "expr"	 	)   { err= Vars.Evaluate(ss); }
	///////////////////////
	else if	(command== "shell"   	)   { err= system(ss.c_str()); }
	//////////////////////////////////////////////////////////////
	else if (command== "logon"		)	{GV<0>::LogAndError.On=true; }
	else if (command== "logoff"		)	{GV<0>::LogAndError.On=false; }
	else if (command== "break"		)   { break; }
	else if (command== "quit"		)   { err=Code_QUIT; break; }
	//////////////////////////////////////////////////////////////
	else if (command== "link"		)   { err=Gs.Link(ss); }
	/////////////
	else if (command== "info"     	)   { err=Gs.SetInfo(ss);  }
	else if (command== "dump" 		)   { err=Gs.SetDump(ss);} 
	/////////////
	else if (command== "set"		)   { err=Gs.Set(ss); } //in program
	else if (command== "read"		)   { err=Gs.Read(ss);}
	else if (command== "readhere"	)   { err=readhere(ss,script);}
	else if (command== "write"		)   { err=Gs.Write(ss);} 
	else if (command== "writehere"	)   { err=Gs.WriteHere(ss);} 
	//////////////////////////////////////////////////////////////
	else if (command== "device"		)	{ err=device(ss); }
	else if (command== "print"		)   { err=print(ss); } 
	else if (command== "sys"		)  	{ err=Gs.SetSys(ss); }
	else if (command== "run"		)   { err=Gs.Run(ss);}
	//////////////////////////////////////////////////////////////
	else if (command== "loop"		) 	{ err=stat_loop(ss,script);}
	else if (command== "while"		) 	{ err=stat_while(ss,script);}
	else if (command== "if"			) 	{ err=stat_if(ss,script);}
	//////////////////////////////////////////////////////////////
	else if (command== "runfunc"	)   { err=Gs.RunFunc(ss);}
	else {
	  err=Gs.RunFunc(command+" "+ss);
	  if (err == Code_COMMAND_UNKNOW){
	  }
	} // default leave out runfunc command

	/////////////////////////////////////////////////////////////////////////
	if (err == Code_COMMAND_UNKNOW){
	  GV<0>::LogAndError<<"Erro: unknown command : "<<line<<"\n"; 
	  return Code_ERR;
	}
	if ( err == Code_ERR ) {
	  GV<0>::LogAndError<<"Error occured when excuting command \""<<line<<"\"\n"; 
	  return Code_ERR;
	}
	if (err == Code_QUIT ) return Code_QUIT;
	/////////////////////////////////////////////////////////////////////////
  }
  return Code_NORMAL;
}

/////////////////////////////////////////////////////////////
bool INPUT::getline(string &ss, string &script){
  if (script=="") return false;
  int n=script.find("\n");
  if (n<0&&script!="") n=script.length();
  ss=script.substr(0, n);
  script.erase(0,n+1);
  if (ss!=""&&GV<0>::LogAndError.On)
	GV<0>::LogAndError<<">>>"<<ss<<"\n";
  return true;
};
int INPUT::standardize(string &script){
  script=script+"\n";
  int p;
  //repalce some cut as space
  //////////////////////////////////////////////////
  while ((p=script.find('\t'))>=0)  script[p]=' ';
  //////////////////////////////////////////////////
  while ((p=script.find(";")) >=0)  script[p]='\n';
  //////////////////////////////////////////////////
  int p1=script.find("#"),p2;
  while (p1>=0){
	p2=script.find("\n",p1);
	script.erase(p1,p2-p1);
	p1=script.find("#");
  }
  //delete extra space
  while ((p=script.find("  "))>=0) script.erase(p,1);
  ///delete space around operator is not practicle in this case, cause space is an important mark
  // delete space and other chars between two bracket---an expression
  int n_brack=0, pre_pos=-1,pos=0;
  string  temp;
  while ( pos<script.length() ){
	if (script[pos]=='(') {
	  n_brack++;
	  if (n_brack==1) pre_pos=pos;
	}else if (script[pos]==')'){
	  n_brack--;
	  if (n_brack==0){
		//standardize the expression found
		temp=script.substr(pre_pos+1,pos-pre_pos-1);
		int p=0;
		while (p<temp.length()){
		  if (temp[p]==' '||temp[p]=='\n')
			temp.erase(p,1);
		  else p++;
		}
		script.replace(pre_pos,pos-pre_pos+1,temp);
	  }
	}
	pos++;
  }
  // deel with "," spcace and "\n" in {}
  n_brack=0, pre_pos=-1,pos=0;
  while ( pos<script.length() ){
	if (script[pos]=='{') {
	  n_brack++;
	  if (n_brack==1) pre_pos=pos;
	}else if (script[pos]=='}'){
	  n_brack--;
	  if (n_brack==0){
		//standardize the expression found
		temp=script.substr(pre_pos,pos-pre_pos+1);
		int p=0;
		while (p<temp.length()){
		  if (temp[p]==' '||temp[p]=='\n'||temp[p]==','){
			temp.replace(p,1,"} {");
			p=p+2;
			continue;
		  }
		  p=p++;
		}
		script.replace(pre_pos,pos-pre_pos+1,temp);
		pos=pre_pos+temp.length();
	  }
	}
	pos++;
  }
  ///exra "," to " "
  while ((p=script.find(";")) >=0)  script[p]='\n';
  return 0;
}
/////////////////////////////////////////////////////////////

int INPUT::readhere(string sr,string &script){
  int ndim=1,nele=1,tem;
  string arrays,varname,ss,sele;
  int 	index=0;
  sr>>varname;
  while (getline(ss,script)){
	 while (ss!=""){
	   if (index==0) {
		 ss>>ndim; arrays<<ndim<<" "; }
	   else if (index<=ndim) {
		 ss>>tem; arrays<<tem<<" "; 
		 nele*=tem; 
	   }else{
		 ss>>sele; arrays<<sele<<" ";
	   }
	   index++;
	 }
	 if ( index > ndim + nele) break;//finish condition // the final index = ndim+nele+1...cause the ndim is counted as 1
  }
  //create the variant number and its corresponding tensor
  Gs.ReadHere(varname, arrays);
  return 0;
}

int INPUT::device(string ss){
  int id; ss>>id;
  if (hipSetDevice(id) )
	GV<0>::LogAndError<<"gpu device changes to devide "<<id<<"\n";
  else {
	GV<0>::LogAndError<<"Error: gpu device change operation unsuccessful!\n"; return -1;
  }
  return 0;
}

////////////////////////////////////////////////////////////

int INPUT::find_sub_script(string&script,string&sub_script){
  // each command corresponds to one and one only end
  int n_end=1;
  for (int p=0;p<script.length();p++){
	for (int i=0; i<Commands_N; i++)
	  if (script.substr(p,Commands[i].length())==Commands[i])
		n_end++;
	if (script.substr(p,3) == "end" )
	  n_end--;
	if (n_end==0){
	  sub_script = script.substr(0,p);
	  script.erase(0,p+3);
	  return 0;
	}
  }
  return Code_INPUT_UNCOMPLETE;
}

int INPUT::stat_loop(string ss,string &script){
  string sub_script,temp_sub,var; ss>>var; script=ss+"\n"+script;
  int err=0;
  find_sub_script(script,sub_script);
  err=Vars.Evaluate(var); if (err!=Code_NORMAL) return err;
  Real loop_n; var>>loop_n;
  ////////////////////////////
  for (int i=0;i<loop_n;i++){
	temp_sub=sub_script;
	err=Phrasing(temp_sub);
	if (err!=Code_NORMAL) return err;
  }
  return 0;
}

int INPUT::stat_while(string ss,string &script){
  string sub_script,temp_sub,expr,sss=ss; sss>>expr; script = sss+"\n"+script;
  int err=0;
  find_sub_script(script,sub_script);
  err=Vars.Evaluate(expr); if (err!=Code_NORMAL) return err;
  ////////////////////////////
  while( ToReal(expr)>0 ){
	temp_sub= sub_script;
	err=Phrasing(temp_sub);
	if (err!=Code_NORMAL) return err;
	ss>>=expr;
	Vars.Evaluate(expr);
  }
  return 0;
}
int INPUT::find_else_script(string&script,string&sub_script1,string &sub_script2){
  // each command corresponds to one and one only end
  int n_end=1;
  int p_else=-1,p_end=-1;
  for (int p=0;p<script.length();p++){
	for (int i=0; i<Commands_N; i++)
	  if (script.substr(p,Commands[i].length())==Commands[i])
		n_end++;
	if (script.substr(p,3) == "end" )
	  n_end--;
	if ( p_else<0 && n_end==1
		&&script.substr(p,4) == "else"){
	  p_else = p;
	}
	if (n_end==0){
	  p_end = p;
	  break;
	}
  }
  if (n_end>0) return Code_INPUT_UNCOMPLETE;
  if (p_else <0 ){
	sub_script1 = script.substr(0,p_end);
	sub_script2 = "";
  }else{
	sub_script1 = script.substr(0,p_else);
	sub_script2 = script.substr(p_else+4,p_end-p_else-4);
  }
  script.erase(0,p_end+3);
  return Code_NORMAL;
}


int INPUT::stat_if(string ss,string &script){
  string sub_script1,sub_script2,expr; ss>>=expr; script=ss+"\n"+script;
  int err=0;
  find_else_script(script,sub_script1,sub_script2);
  Vars.Evaluate(expr);
  if (ToReal(expr)>0){
	err=Phrasing(sub_script1);
  }else{
	err=Phrasing(sub_script2);
  }
  return err;
}


int INPUT::print(string ss){
  int err;
  err=Vars.ReplaceExpr(ss);
  GV<0>::LogAndError<<ss<<"\n";
  return err;
}
