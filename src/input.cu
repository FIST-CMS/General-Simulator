
#include"pub.h"
#include"pub_main.h"
#include"input.h"

using namespace std;
using namespace GS_NS;

INPUT::INPUT(){
  LineNumber=0;
}
INPUT::~INPUT(){}

bool INPUT::getline(string &ss, string &script){
  if (script=="") return false;
  LineNumber++;
  int n=script.find("\n");
  ss=script.substr(0, n);
  script.erase(0,n+1);
  GV<0>::LogAndError<<LineNumber<<": "<<ss<<"\n";
  return true;
};
int INPUT::standardize(string &script){
  script=script+"\n";
  int p;
  //repalce some cut as space
  //////////////////////////////////////////////////
  while ((p=script.find("\t"))>=0)  script[p]=' ';
  while ((p=script.find(",")) >=0)  script[p]=' ';
  while ((p=script.find(";")) >=0)  script[p]=' ';
  while ((p=script.find("{")) >=0)  script[p]=' ';
  while ((p=script.find("}")) >=0)  script[p]=' ';
  //while ((p=script.find("(")) >=0)  script[p]=' ';
  //while ((p=script.find(")")) >=0)  script[p]=' ';
  //////////////////////////////////////////////////
  int p1=script.find("#"),p2;
  while (p1>=0){
	p2=script.find("\n",p1);
	script.erase(p1,p2-p1);
	p1=script.find("#");
  }
  //delete extra space
  while ((p=script.find("  "))>=0)
	script.erase(p,2);
  return 0;
}

int INPUT::Phrasing(string script){
  standardize(script);
  int err=0;
  string ss;
  while (getline(ss,script)){
	////////////////////
	////////////////////
	while (ss[0]==' ') ss.erase(0,1); //delete the heading space of each line
    if (ss=="") continue;
    string command; ss>>command;
	if      (command== "variable" 		)   { err=Vars.Set(ss); }
	else if (command== "shell"   		)   { err=system(ss.c_str()); }
	else if (command== "quit"			)   { err=Code_QUIT; break; }
	else if (command== "break"			)   { break; }
	//////////////////////////////////////////////////////////////
	else if (command== "link"			)   { err=Gs.Link(ss); }
	else if (command== "set"			)   { Vars.SubVar(ss,1);err=Gs.Set(ss);}
	else if (command== "read"			)   { Vars.SubVar(ss,1);err=Gs.Read(ss);}
	else if (command== "dump" 			)   { Vars.SubVar(ss,1);err=Gs.SetDump(ss); } 
	else if (command== "readhere"		)   { err=readhere(ss,script);}
	else if (command== "dumphere"		)   { err=Gs.DumpHere(ss);} 
	//////////////////////////////////////////////////////////////
	else if (command== "device"			)	{ Vars.SubVar(ss,0);err=device(ss); }
	else if (command== "sys"			)  	{ Vars.SubVar(ss,0);err=Gs.SetSys(ss); }
	else if (command== "info"     		)   { Vars.SubVar(ss,0);err=Gs.SetInfo(ss);  }
	else if (command== "run"			)   { Vars.SubVar(ss,0);err=Gs.Run(ss);}
	else if (command== "runfunc"		)   { Vars.SubVar(ss,0);err=Gs.RunFunc(ss);}
	else {
	  GV<0>::LogAndError<< command <<" run as runfunc command\n"; 
	  Vars.SubVar(ss,0);
	  err=Gs.RunFunc(command+" "+ss);
	} // default leave out runfunc command
	if (err == Code_COMMAND_UNKNOW){
	  GV<0>::LogAndError<<"Command "<<command <<" is unknow.\n"; 
	  return Code_ERR;
	}
	if ( err == Code_ERR ) {
	  GV<0>::LogAndError<<"Error occured when excuting command "<<command<<"."; 
	  return Code_ERR;
	}
	if (err == Code_QUIT ) return Code_QUIT;
  }
  return Code_NORMAL;
}

int INPUT::readhere(string sr,string &script){
  int ndim=1,nele=1,tem;
  string arrays,varname,ss,sele;
  int 	index=0;
  sr>>varname;
  while (getline(ss,script)){
	 Vars.SubVar(ss,0);
	 while (ss!=""){
	   if (index==0) {
		 ss>>ndim; arrays<<ndim<<" "; }
	   else if (index<=ndim) {
		 ss>>tem; arrays<<tem<<" "; 
		 nele*=tem; 
	   }else{
		 ss>>sele; arrays<<sele<<" ";
	   }
	   index++;
	 }
	 if ( index > ndim + nele) break;//finish condition // the final index = ndim+nele+1...cause the ndim is counted as 1
  }
  //create the variant number and its corresponding tensor
  Gs.ReadHere(varname, arrays);
  return 0;
}

int INPUT::device(string ss){
  int id; ss>>id;
  if (hipSetDevice(id) )
	GV<0>::LogAndError<<"gpu device changes to devide "<<id<<"\n";
  else {
	GV<0>::LogAndError>>"Error: gpu device change operation unsuccessful!\n"; return -1;
  }
  return 0;
}
