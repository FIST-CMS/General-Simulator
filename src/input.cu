
#include"pub.h"
#include"pub_main.h"
#include"input.h"

using namespace std;
using namespace GS_NS;

INPUT::INPUT(){
}
INPUT::~INPUT(){}

int INPUT::Phrasing(string script){
  int err=0;
  string ss;
  while (getline(ss,script)){
	////////////////////
	while (ss[0]==' ') ss.erase(0,1); //delete the heading space of each line
    if (ss=="") continue;
    string command; ss>>command;
	if 		(command== "shell"   	)   { err=system(ss.c_str()); }
	//////////////////////////////////////////////////////////////
	else if (command== "logon"		)	{GV<0>::LogAndError.On=true; }
	else if (command== "logoff"		)	{GV<0>::LogAndError.On=false; }
	else if (command== "break"		)   { break; }
	else if (command== "quit"		)   { err=Code_QUIT; break; }
	//////////////////////////////////////////////////////////////
	else if (command== "link"		)   { err=Gs.Link(ss); }
	else if (command== "set"		)   { Vars.SubVar(ss,1);err=Gs.Set(ss);}
	/////////////
	else if (command== "info"     	)   { Vars.SubVar(ss,0);err=Gs.SetInfo(ss);  }
	else if (command== "dump" 	)   { Vars.SubVar(ss,1);err=Gs.SetDump(ss);} 
	/////////////
	else if (command== "read"		)   { Vars.SubVar(ss,1);err=Gs.Read(ss);}
	else if (command== "readhere"	)   { err=readhere(ss,script);}
	else if (command== "write"		)   { err=Gs.Write(ss);} 
	else if (command== "writehere"	)   { err=Gs.WriteHere(ss);} 
	//////////////////////////////////////////////////////////////
	else if (command== "device"		)	{ Vars.SubVar(ss,0);err=device(ss); }
	//else if (command== "var"	 	)   { err=Vars.Set(ss); }
	else if (command== "expr"	 	)   { err=expresion(ss); }
	else if (command== "sys"		)  	{ Vars.SubVar(ss,0); err=Gs.SetSys(ss); }
	else if (command== "run"		)   { Vars.SubVar(ss,0); err=Gs.Run(ss);}
	//////////////////////////////////////////////////////////////
	else if (command== "loop"		) 	{ err=stat_loop(ss,script);}
	else if (command== "while"		) 	{ err=stat_while(ss,script);}
	else if (command== "if"			) 	{ err=stat_if(ss,script);}
	//////////////////////////////////////////////////////////////
	else if (command== "runfunc"	)   { Vars.SubVar(ss,0);err=Gs.RunFunc(ss);}
	else {
	  Vars.SubVar(ss,0);
	  err=Gs.RunFunc(command+" "+ss);
	} // default leave out runfunc command

	/////////////////////////////////////////////////////////////////////////
	if (err == Code_COMMAND_UNKNOW){
	  GV<0>::LogAndError<<"Command "<<command <<" is unknow.\n"; 
	  return Code_ERR;
	}
	if ( err == Code_ERR ) {
	  GV<0>::LogAndError<<"Error occured when excuting command "<<command<<".\n"; 
	  return Code_ERR;
	}
	if (err == Code_QUIT ) return Code_QUIT;
	/////////////////////////////////////////////////////////////////////////
  }
  return Code_NORMAL;
}

/////////////////////////////////////////////////////////////
bool INPUT::getline(string &ss, string &script){
  if (script=="") return false;
  int n=script.find("\n");
  ss=script.substr(0, n);
  script.erase(0,n+1);
  if (ss!=""&&GV<0>::LogAndError.On)
	GV<0>::LogAndError<<">>>"<<ss<<"\n";
  return true;
};
int INPUT::standardize(string &script){
  script=script+"\n";
  int p;
  //repalce some cut as space
  //////////////////////////////////////////////////
  while ((p=script.find('\t'))>=0)  script[p]=' ';
  while ((p=script.find(",")) >=0)  script[p]=' ';
  while ((p=script.find(";")) >=0)  script[p]=' ';
  while ((p=script.find(":")) >=0)  script[p]=' ';
  while ((p=script.find("{")) >=0)  script[p]=' ';
  while ((p=script.find("}")) >=0)  script[p]=' ';
  //////////////////////////////////////////////////
  int p1=script.find("#"),p2;
  while (p1>=0){
	p2=script.find("\n",p1);
	script.erase(p1,p2-p1);
	p1=script.find("#");
  }
  //delete extra space
  while ((p=script.find("  "))>=0)
	script.erase(p,1);
  ///delete space around operator is not practicle in this case, cause space is an important mark
  // delete space and other chars between two bracket---an expression
  int n_brack=0, pre_pos=-1,pos=0;
  string  temp;
  while ( pos<script.length() ){
	if (script[pos]=='(') {
	  n_brack++;
	  if (n_brack==1) pre_pos=pos;
	}else if (script[pos]==')'){
	  n_brack--;
	  if (n_brack==0){
		//standardize the expression found
		temp=script.substr(pre_pos+1,pos-pre_pos-1);
		int p=0;
		while (p<temp.length()){
		  if (temp[p]==' '||temp[p]=='\n')
			temp.erase(p,1);
		  else p++;
		}
		script.replace(pre_pos,pos-pre_pos+1,temp);
	  }
	}
	pos++;
  }
  return 0;
}
/////////////////////////////////////////////////////////////

int INPUT::readhere(string sr,string &script){
  int ndim=1,nele=1,tem;
  string arrays,varname,ss,sele;
  int 	index=0;
  sr>>varname;
  while (getline(ss,script)){
	 Vars.SubVar(ss,0);
	 while (ss!=""){
	   if (index==0) {
		 ss>>ndim; arrays<<ndim<<" "; }
	   else if (index<=ndim) {
		 ss>>tem; arrays<<tem<<" "; 
		 nele*=tem; 
	   }else{
		 ss>>sele; arrays<<sele<<" ";
	   }
	   index++;
	 }
	 if ( index > ndim + nele) break;//finish condition // the final index = ndim+nele+1...cause the ndim is counted as 1
  }
  //create the variant number and its corresponding tensor
  Gs.ReadHere(varname, arrays);
  return 0;
}

int INPUT::device(string ss){
  int id; ss>>id;
  if (hipSetDevice(id) )
	GV<0>::LogAndError<<"gpu device changes to devide "<<id<<"\n";
  else {
	GV<0>::LogAndError<<"Error: gpu device change operation unsuccessful!\n"; return -1;
  }
  return 0;
}

int INPUT::expresion(string ss){
  string str_t,str_hold=ss;
  ss>>str_t;
  if (ss==""){
	Vars.SubVar(str_t,0);
	GV<0>::LogAndError<<str_t<<"\n";
	return 0;
  }else{
	return Vars.Set(str_hold);
  }
}
////////////////////////////////////////////////////////////

int INPUT::find_sub_script(string&script,string&sub_script){
  // each command corresponds to one and one only end
  int n_end=1;
  for (int p=0;p<script.length();p++){
	for (int i=0; i<Commands_N; i++)
	  if (script.substr(p,Commands[i].length())==Commands[i])
		n_end++;
	if (script.substr(p,3) == "end" )
	  n_end--;
	if (n_end==0){
	  sub_script = script.substr(0,p);
	  script.erase(0,p+3);
	  return 0;
	}
  }
  return Code_INPUT_UNCOMPLETE;
}

int INPUT::stat_loop(string ss,string &script){
  string sub_script,temp_sub,var; ss>>=var;
  int err=0;
  find_sub_script(script,sub_script);
  err=Vars.SubVar(var,0);
  if (err!=Code_NORMAL) return err;
  Real loop_n; var>>loop_n;
  ////////////////////////////
  for (int i=0;i<loop_n;i++){
	temp_sub=sub_script;
	err=Phrasing(temp_sub);
	if (err!=Code_NORMAL) return err;
  }
  return 0;
}

int INPUT::stat_while(string ss,string &script){
  string sub_script,temp_sub,expr; ss>>=expr;
  int err=0;
  find_sub_script(script,sub_script);
  err=Vars.SubVar(expr,0); if (err!=Code_NORMAL) return err;
  ////////////////////////////
  while( ToReal(expr)>0 ){
	temp_sub= sub_script;
	err=Phrasing(temp_sub);
	if (err!=Code_NORMAL) return err;
	ss>>=expr;
	Vars.SubVar(expr,0);
  }
  return 0;
}
int INPUT::find_else_script(string&script,string&sub_script1,string &sub_script2){
  // each command corresponds to one and one only end
  int n_end=1;
  int p_else=-1,p_end=-1;
  for (int p=0;p<script.length();p++){
	for (int i=0; i<Commands_N; i++)
	  if (script.substr(p,Commands[i].length())==Commands[i])
		n_end++;
	if (script.substr(p,3) == "end" )
	  n_end--;
	if ( p_else<0 && n_end==1
		&&script.substr(p,4) == "else"){
	  p_else = p;
	}
	if (n_end==0){
	  p_end = p;
	  break;
	}
  }
  if (n_end>0) return Code_INPUT_UNCOMPLETE;
  if (p_else <0 ){
	sub_script1 = script.substr(0,p_end);
	sub_script2 = "";
  }else{
	sub_script1 = script.substr(0,p_else);
	sub_script2 = script.substr(p_else+4,p_end-p_else-4);
  }
  script.erase(0,p_end+3);
  return Code_NORMAL;
}


int INPUT::stat_if(string ss,string &script){
  string sub_script1,sub_script2,expr; ss>>=expr;
  int err=0;
  find_else_script(script,sub_script1,sub_script2);
  Vars.SubVar(expr,0);
  if (ToReal(expr)>0){
	err=Phrasing(sub_script1);
  }else{
	err=Phrasing(sub_script2);
  }
  return err;
}

