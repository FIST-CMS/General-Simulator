#include "hip/hip_runtime.h"

#include"pub.h"
#include"pub_main.h"
#include"input.h"

using namespace std;
using namespace GUPS_NS;

INPUT::INPUT(string infile){
  fin.open(infile.c_str());
  LineNumber=0;
}

INPUT::~INPUT(){
  if (fin)  fin.close();
}

string  INPUT::standardize(string ss){
  int p,p1;
  p=ss.find("#");
  ss=ss.substr(0,p); ss+="\n";
  //repalce other cut as space
  while ((p=ss.find("\n")) >=0)  ss[p]=' ';
  while ((p=ss.find("\t"))>=0) ss[p]=' ';
  while ((p=ss.find(",")) >=0)  ss[p]=' ';
  while ((p=ss.find(";")) >=0)  ss[p]=' ';
  while ((p=ss.find("{")) >=0)  ss[p]=' ';
  while ((p=ss.find("}")) >=0)  ss[p]=' ';
  while ((p=ss.find("(")) >=0)  ss[p]=' ';
  while ((p=ss.find(")")) >=0)  ss[p]=' ';
  while (ss[0]==' ')  ss=ss.erase(0,1);
  //delete extra space
  p=ss.find(" ");
  while (p>=0){
    p1=ss.find(" ",p+1);
    if (p1==p+1) 
	 ss.erase(p+1,1);
    else{
	 p = ss.find(" ",p+1);
    }
  }
  return ss;
}

bool INPUT::fgets_str(ifstream &is, string &ss){
  if (getline(is,ss)){ LineNumber++; return 1; }
  else return 0;
}

int INPUT::Phrasing(){
  int err=0;
  string ss;
  while (fgets_str(fin,ss)){
	GV<0>::LogAndError<<LineNumber<<": "<<ss<<"\n";
    ss=this->standardize(ss);
    if (ss=="") continue;
    string command; ss>>command;
	if      (command== "variable" 		)   err=Vars.Set(ss); //variable(ss);
	else if (command== "shell"   		)   err=system(ss.c_str());
	else if (command== "quit"			)   break;
	//////////////////////////////////////////////////////////////
	else if (command== "link"			)   { err= Gups.Link(ss); }
	else if (command== "set"			)   { Vars.SubVar(ss,1); err=Gups.Set(ss);}
	else if (command== "read"			)   { Vars.SubVar(ss,1); err=Gups.Read(ss);}
	else if (command== "dump" 			)   { Vars.SubVar(ss,1); err=Gups.SetDump(ss); } 
	//////////////////////////////////////////////////////////////
	else{
	  Vars.SubVar(ss,0);
	  if 	  (command== "device"		) 	err=device(ss); 
	  else if (command== "sys"			)  	err=Gups.SetSys(ss);
	  else if (command== "variant"		) 	err=variant(ss);
	  else if (command== "thermo"     	)   err=Gups.SetThermo(ss); 
	  else if (command== "run"			)   err=Gups.Run(ss);//run(ss);
	  else{
		GV<0>::LogAndError<<"Command "<<command<<" is not supported in GUPS!"; 
		return -1;
	  }
	}
	if (err<0) return -1;
  }
  return 0;
}

int INPUT::variant(string sr){
  int n;
  sr>>n;
  if (n<=0){
	GV<0>::LogAndError>>"Variant number is less than one!\n";
    return -1;
  }

  real *arr=new real[n*3*3];
  string 	ss;
  int 	index=0;
  while (fgets_str(fin,ss)){
	 ss=this->standardize(ss);
	 Vars.SubVar(ss,0);
	 while (ss!=""){
	   ss>>arr[index];
	   index++;
	 }
	 if ( index >= 3*3*n) break;//finish condition
  }
  //create the variant number and its corresponding tensor
  Gups.CreateVariant(n, arr);
  delete []arr;

  //log the straintensor
  GV<0>::LogAndError<<"straintensor created\n";
  for (int v = 0; v < n; v++){
    for (int i=0;i<3;i++){
	 for (int j=0;j<3;j++){
	   GV<0>::LogAndError<<(Gups.Datas["straintensor"])(v,i,j)<<" ";
	 }
	 GV<0>::LogAndError<<"\n";
    }
	GV<0>::LogAndError<<"\n";
  }
  return 0;
}

int INPUT::device(string ss){
  int id; ss>>id;
  if (hipSetDevice(id) )
	GV<0>::LogAndError<<"gpu device changes to devide "<<id<<"\n";
  else {
	GV<0>::LogAndError>>"Error: gpu device change operation unsuccessful!\n"; return -1;
  }
  return 0;
}
