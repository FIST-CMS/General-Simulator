#include"pub.h"
#include"pub_main.h"
#include"input.h"

using namespace GUPS_NS;
using namespace std;

int main(int argn,char* args[]){
  GV<0>::LogAndError.Init("gups");
  string file;
  if (argn==1){
	file ="in.gups";	
	GV<0>::LogAndError<<"Since no input script assigned, default \"in.gups\" is used.\n";
  }else{
	file = args[1];
	GV<0>::LogAndError<<"Input script \""<<file<<"\" is used.\n";
	int device=0;
	if (argn==3) { 
	  io(args[2],device); 
	  hipSetDevice(device);
	  GV<0>::LogAndError<<"Gpu device set to "<<device<<"\n";
	}  
  }
  INPUT qin(file);				// 
  if (! qin.fin.fail())
	qin.Phrasing();	
  else
	GV<0>::LogAndError<<"Input script "<<file<<" is not found\n";
  return 0;
}

