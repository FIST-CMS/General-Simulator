#include"pub.h"
#include"pub_main.h"
#include"input.h"

using namespace GS_NS;
using namespace std;

int main(int argn,char* args[]){
  GV<0>::LogAndError.Init("gs");
  string file;
  if (argn==1){
	file ="in.gs";	
	GV<0>::LogAndError<<"Since no input script assigned, default \"in.gs\" is used.\n";
  }else{
	file = args[1];
	GV<0>::LogAndError<<"Input script \""<<file<<"\" is used.\n";
	int device=0;
	if (argn==3) { 
	  io(args[2],device); 
	  hipSetDevice(device);
	  GV<0>::LogAndError<<"Gpu device set to "<<device<<"\n";
	}  
  }
  ifstream in(file.c_str(), ios::in);
  if (in.fail()){
	GV<0>::LogAndError<<"Input script "<<file<<" is not found\n";
	return -1;
  }
  istreambuf_iterator<char> beg(in), end;
  string script(beg, end);
  in.close();

  INPUT qin;				// 
  qin.Phrasing(script);	
  return 0;
}

