#include "hip/hip_runtime.h"

#define DEBUG 0

////////////////////////////////////////
#include"pub.h"
#include"dynamics.h"
////////////////////////////////////////
#include<hiprand.h>
#include<hipfft/hipfft.h>
#include"random.h"
#include"gtensorb.h"

#include"dynamics_stress.h"

using namespace GS_NS;
using namespace DATA_NS;
/*
  main calculation:
    the structure tensor B
	derivative of order parameter and concentration
 */

int DynamicsStress::Initialize(){
  //para setting should be finished before or within this function
  string ss;

  (((((Vars["gridsize"]>>=nx)>>=ny)>>=nz)>>=dx)>>=dy)>>=dz;
  Xi=4000.0f;  Vars["xi"]>>=Xi;
  StrainTensor = &((*Datas)["varianttensor"]);
  if (StrainTensor->Arr == NULL){
	GV<0>::LogAndError>>"Error: variants' strain tensor does not set while initialize dynamics\n";
	return -1;
  }
  VariantN = StrainTensor->Dimension[1];
  ///////////////////////////////////////////////////
  Stress = &((*Datas)["stress"]); // the assign function will malloc space for Stress
  Defect = &((*Datas)["defect"]); // may create here
  Stress->Init(4,6,nx,ny,nz,Data_HOST_DEV);
  Defect->Init(3,nx,ny,nz,Data_HOST);
  ///////////////////////////////////////////////////
  Eta = &((*Datas)["eta"]); // may create here
  if ( Eta->Arr == NULL ){
	Eta->Init(4,VariantN,nx,ny,nz,Data_HOST_DEV);
	SetCalPos(Data_DEV);
	(*Eta)=0.0f; 
  }else { Eta->HostToDevice(); }
  ////////////////////////////////////////////////////
  Data<Real> cijkl;
  SetCalPos(Data_HOST);
  cijkl.Init(4,3,3,3,3); cijkl=0.f;
  cijkl(1-1,1-1,1-1,1-1) =5.39  ;//c11
  cijkl(2-1,2-1,2-1,2-1) =5.39  ;//c11
  cijkl(3-1,3-1,3-1,3-1) =5.22  ;//c11
  cijkl(1-1,1-1,2-1,2-1) =3.39  ;//c12
  cijkl(2-1,2-1,3-1,3-1) =3.56  ;//c12
  cijkl(3-1,3-1,1-1,1-1) =3.56  ;//c12
  cijkl(1-1,2-1,1-1,2-1) =0.6   ;//c44
  cijkl(2-1,3-1,2-1,3-1) =0.77  ;//c44
  cijkl(3-1,1-1,3-1,1-1) =0.77  ;//c44

  cijkl(2-1,2-1,1-1,1-1) =3.39  ;//c12
  cijkl(3-1,3-1,2-1,2-1) =3.56  ;//c12
  cijkl(1-1,1-1,3-1,3-1) =3.56  ;//c12
  cijkl(2-1,1-1,2-1,1-1) =0.60  ;//c44
  cijkl(1-1,2-1,2-1,1-1) =0.60  ;//c44
  cijkl(2-1,1-1,1-1,2-1) =0.60  ;//c44
  cijkl(3-1,2-1,3-1,2-1) =0.77  ;//c44
  cijkl(3-1,2-1,2-1,3-1) =0.77  ;//c44
  cijkl(2-1,3-1,3-1,2-1) =0.77  ;//c44
  cijkl(1-1,3-1,1-1,3-1) =0.77  ;//c44
  cijkl(1-1,3-1,3-1,1-1) =0.77  ;//c44
  cijkl(3-1,1-1,1-1,3-1) =0.77  ;//c44
  Data<Real> *modulus; modulus=&((*Datas)["modulus"]);
  if ( modulus->Arr != NULL )
	cijkl = (*modulus);
  ////////////////////////////////////////////////////
  tensor.Init(3,BaseVariantN+VariantN,3,3,Data_HOST_DEV); 
  SetCalPos(Data_HOST);
  tensor=0.f;
  tensor(0,0,0)=1.0f; tensor(1,0,1)=1.0f; tensor(2,0,2)=1.0f;
  tensor(3,1,1)=1.0f; tensor(4,1,2)=1.0f; tensor(5,2,2)=1.0f;
  for (int i=6*9; i< (6+VariantN)*9; i++)
	tensor[i]=(*StrainTensor)[i-6*9];
  
  GV<0>::LogAndError<<"Calculating space structure tensor \n";
  B.InitB(BaseVariantN,VariantN,nx,ny,nz,dx.Re,dy.Re,dz.Re,tensor,cijkl); 
  GV<0>::LogAndError<<"Calculating space structure finished\n";

  int rank=3,ns[3]={nx,ny,nz},dist=nx*ny*nz,stride=1;
  GV<0>::LogAndError<<"Cuda fft plan is to create\n";
  if (hipfftPlanMany(&plan_vn,rank,ns,ns,stride,dist,ns,stride,dist,HIPFFT_C2C,VariantN)==HIPFFT_SUCCESS)
	GV<0>::LogAndError<<"Cuda fft plan vn is created\n";
  else GV<0>::LogAndError<<"Cuda fft plan vn fails to create\n";

  if (hipfftPlanMany(&plan_bvn,rank,ns,ns,stride,dist,ns,stride,dist,HIPFFT_C2C,BaseVariantN/*6*/)==HIPFFT_SUCCESS)
	GV<0>::LogAndError<<"Cuda fft plan bvn is created\n";
  else GV<0>::LogAndError<<"Cuda fft plan bvn fails to create\n";

  int vndim[6]={4,VariantN,nx,ny,nz};
  int bvndim[6]={4,BaseVariantN,nx,ny,nz};

  Eta_CT.Init(vndim,Data_HOST_DEV);
  RTermEta_CT.Init(bvndim,Data_HOST_DEV);

  return 0;

}

DynamicsStress::DynamicsStress(){}
DynamicsStress::~DynamicsStress(){
  if (plan_vn) hipfftDestroy(plan_vn);
  if (plan_bvn) hipfftDestroy(plan_bvn);
}

__global__ void ElasticForceCalculate_Stress_Kernel(Complex *ReTerm,Complex*Eta_sq,Real* B,int VariantN,Real Xi){
  //int BaseVariantN = gridDim.z;
  int nx= gridDim.x, ny= gridDim.y, nz = blockDim.x;
  int x = blockIdx.x, y = blockIdx.y, z = threadIdx.x, v = blockIdx.z;
  int nn= nx*ny*nz;
  int nvn=  VariantN* nn;
  int pn= (x*ny +y)*nx+z;
  Complex temp = 0;
  for (int i=0;i<VariantN;i++){
	temp+=Xi*B[ v*nvn + i*nn +pn ]* Eta_sq[ i*nn + pn ];
	if (DEBUG) ReTerm[v*nn+pn]=temp;
  }
  ReTerm[ v*nn + pn ] = temp;
}

int DynamicsStress::ElasticForceCalculate(){
  SetCalPos(Data_DEV);
  //Eta_CT=(*Eta)*(*Eta); //Store it in the buffer area
  Eta_CT=(*Eta); //Store it in the buffer area
  hipfftExecC2C(plan_vn,(hipfftComplex*)Eta_CT.Arr_dev,(hipfftComplex*)Eta_CT.Arr_dev,HIPFFT_FORWARD);
  divi_device(Eta_CT.Arr_dev,Eta_CT.Arr_dev ,real(nx*ny*nz) ,Eta_CT.N() ); // seperate transformed
  dim3 bn(nx,ny,BaseVariantN/*6*/);
  dim3 tn(nz);
  ElasticForceCalculate_Stress_Kernel<<<bn,tn>>>
	(RTermEta_CT.Arr_dev,Eta_CT.Arr_dev,B.Arr_dev,VariantN,Xi);
  hipfftExecC2C(plan_bvn,(hipfftComplex*)RTermEta_CT.Arr_dev,(hipfftComplex*)RTermEta_CT.Arr_dev,HIPFFT_BACKWARD);
  *Stress = - RTermEta_CT;
  return 0;
}

int DynamicsStress::Calculate(){
  string ss;
  
  ElasticForceCalculate();

  SetCalPos(Data_HOST);
  (*Defect)=0.f;
  for (int v=0; v<VariantN; v++)
	for (int i=0; i<nx; i++)
	  for (int j=0; j<ny; j++)
		for (int k=0; k<nz; k++)
		  if (abs((*Eta)(v,i,j,k))>0.9)
			(*Defect)(i,j,k)=1.f;
  
  return 0;
}

int DynamicsStress::RunFunc(string funcName){ return 0; }
 

int DynamicsStress::Fix(real progress){
  string ss,mode;
  return 0;
}

string DynamicsStress::Get(string ss){ // return the statistic info.
  string var; ss>>var;
  return "nan"; 
}
