#include "hip/hip_runtime.h"

#include"pub.h"
#include"pub_main.h"

using namespace GS_NS;
using namespace DATA_NS;


GS::GS(){
  DynaTotal=0;
  DynaPosition=-1;
  DynaName="";
  Datas = new Map < Data<Real> >;
  Vars  = new Map < string >;
}


GS::~GS(){}

int GS::SetInfo(string ss){ss>>InfoSteps; InfoMode  = ss; return 0;}

int GS::InfoOut(){
  string sm,ss; sm=InfoMode;
  GV<0>::LogAndError<<"Step  "<<CurrentStep<<" \t";
  while (sm !=""){
	sm>>ss;
	GV<0>::LogAndError<<Dynas[DynaPosition]->Get(ss)<<" \t";
  }
  GV<0>::LogAndError<<"\n";
  return 0;
}

int GS::SetDump(string ss){
  ss>>DumpFolder>>DumpSteps;
  string cmd="if [ -s "+DumpFolder+" ]; then echo \'dump to folder"+DumpFolder+"\'; else mkdir "+DumpFolder+"; fi";
  if (system(cmd.c_str()));
  DumpMode	 = ss;
  return 0;
}

int GS::DumpOut(){
  string sm,ss;
  sm=DumpMode;
  while (sm !=""){
	sm>>ss;
	if ((*Datas)(ss) != NULL){
	  if ((*Datas)[ss].Position==Data_DEV)
		(*Datas)[ss].DeviceToHost();
	  string file; file<<DumpFolder<<"/"<<ss<<"."<<CurrentStep<<".data";
	  if (BinaryMode) (*Datas)[ss].BinaryDumpFile(file);
	  else (*Datas)[ss].DumpFile(file);
	}else GV<0>::LogAndError<<"Error: data "<<ss<<" does not exist!\n";
  }
  return 0;
}

int GS::SetSys(string ss){
  string sys; ss>>sys;
  /////////////////////////
  int pos=-1;
  if ( DynaPositions(sys)!=NULL ) {
	DynaName=sys;
	DynaPosition=DynaPositions[ sys ];
	return 0;
  }
  ////////////////////////////////////
  //create a new dyna
  pos = DynaTotal;
  bool hassys=false;
  ///////////////////////////////////
  //GS_SYS_DEFINE_START
  if (sys == "cores"	) { Dynas[ pos ] = new Dynamics_cores; hassys=true; }
  if (sys == "mart"	) { Dynas[ pos ] = new Dynamics_mart; hassys=true; }
  if (sys == "multi"	) { Dynas[ pos ] = new Dynamics_multi; hassys=true; }
  if (sys == "pow"	) { Dynas[ pos ] = new Dynamics_pow; hassys=true; }
  if (sys == "precipitate"	) { Dynas[ pos ] = new Dynamics_precipitate; hassys=true; }
  if (sys == "stress"	) { Dynas[ pos ] = new Dynamics_stress; hassys=true; }
  if (sys == "xxx"	) { Dynas[ pos ] = new Dynamics_xxx; hassys=true; }
  //GS_SYS_DEFINE_END
  ///////////////////////////////////
  if (!hassys) {
	GV<0>::LogAndError<<"Error: unknown system \""<<sys<<"\"\n";
	return -1;
  }
  ///then a new will be create
  DynasInited[sys]=false;
  DynaPositions[sys]=pos;
  DynaName=sys;
  DynaPosition=pos;
  DynaTotal++;
  return 0;
}

//////////////////////////////////////////////////
int GS::Set(string ss){
  string var,str;
  ss>>var;
  if (var=="binary"){
    ss>>str; 
    if (str=="on") BinaryMode=true;
    else BinaryMode=false;
  }else if (var=="log"){
    ss>>str;
    if (str=="off") GV<0>::LogAndError.On=false;
    else GV<0>::LogAndError.On=true;
  }else (*Vars)[var] = ss;
  return 0;
}

int GS::Link(string ss){
  string target,link;
  ss>>target>>link;
  Vars->link(target,link);
  return 0;
}

int GS::Read(string ss){
  string file,var; ss>>var>>file;
  if (BinaryMode)
    return (*Datas)[var].BinaryReadFile(file);
  else
    return (*Datas)[var].ReadFile(file);
}

int GS::ReadHere(string name, string &arrays){
  int n; arrays>>n;
  int *dim = new int[n+1]; dim[0]=n;
  for (int i=1; i<=n; i++) arrays>>dim[i];
  (*Datas)[name].Init(dim,Data_HOST);
  for (int i=0; i<(*Datas)[name].N(); i++)
	arrays>>(*Datas)[name].Arr[i];
  return 0;
}

int GS::Write(string sm){
  string ss,file;
  while (sm !=""){
	sm>>ss>>file;
	if ((*Datas)(ss) != NULL){
	  if ( (*Datas)[ss].Position==Data_DEV)
		(*Datas)[ss].DeviceToHost();
	  if (file == "") file<<ss<<".data";
	  if (BinaryMode) (*Datas)[ss].BinaryDumpFile(file);
	  else (*Datas)[ss].DumpFile(file);
	}else GV<0>::LogAndError<<"Error: unknown data \""<<ss<<"\"\n";
  }
  return 0;
}

int GS::WriteHere(string sm){
  string ss;
  while (sm !=""){
	sm>>ss;
	if ((*Datas)(ss) != NULL){
	  if ((*Datas)[ss].Position==Data_DEV)
		(*Datas)[ss].DeviceToHost();
	  cout<<(*Datas)[ss];
	  GV<0>::LogAndError.Logofs<<(*Datas)[ss];
	}else if ((*Vars)(ss) != NULL){
	  cout<<(*Vars)[ss];
	  GV<0>::LogAndError.Logofs<<(*Vars)[ss];
	}else GV<0>::LogAndError<<"Error: unknown data \""<<ss<<"\"\n";
  }
  return 0;
}


int GS::Run(string ss){
  if (DynaName=="") {
	GV<0>::LogAndError<<"Error: run before system setting\n";
	return -1;
  }
  ////////////////////////////////////
  int totalsteps=1;
  ss>>totalsteps; TotalSteps=totalsteps;
  if (totalsteps <= 0 ) return -1;
  Dynas[DynaPosition]->Datas = Datas; 
  Dynas[DynaPosition]->Vars = Vars; 
  if ( !DynasInited[DynaName] ){
	Dynas[DynaPosition]->Initialize();
	DynasInited[DynaName]=true;
	//////////////////////////////////////////////////////////////
	CurrentStep=0;
	(*Vars)["ahead_steps"]>>=CurrentStep;
  }
  //////////////////////////////////////////////////////////////
  int  infoInterval;
  if ( InfoSteps==0) infoInterval=totalsteps+1;
  else infoInterval = totalsteps/InfoSteps;
  if ( infoInterval==0 ) infoInterval=1;
  int  dumpInterval;
  if ( DumpSteps== 0 ) dumpInterval= totalsteps+1;
  else dumpInterval  = totalsteps/DumpSteps;
  if ( dumpInterval == 0 ) dumpInterval =1;
  //////////////////////////////////////////////////////////////
  string mode=InfoMode,tempss;
  GV<0>::LogAndError<<"Info \t\t";
  while (mode!=""){ mode>>tempss; GV<0>::LogAndError<<tempss<<"\t"; };
  GV<0>::LogAndError<<"\n";
  //////////////////////////////////////////////////////////////
  for (int i=1;i<=totalsteps;i++) {
	CurrentStep++; // this will be used in dump and info to identity the progress
	/////////////////////////////
	Dynas[DynaPosition]->Fix(real(i)/totalsteps);
	Dynas[DynaPosition]->Calculate();
	//////////////////////////////
	if (i % infoInterval==0|| i==totalsteps) InfoOut();
	if (i % dumpInterval  ==0|| i==totalsteps) DumpOut();
  }
  //////////////////////////////////////////////////////////////
  return 0;
}

int GS::RunFunc(string func ){
  if (DynaPositions(DynaName)==NULL){
	GV<0>::LogAndError<<"Error: no system set, runfunc commands not available\n";
	return Code_ERR;
  }
  Dynas[DynaPosition]->Datas = Datas; 
  Dynas[DynaPosition]->Vars= Vars; 
  if ( !DynasInited[DynaName] ){
	GV<0>::LogAndError<<"Warning: Initialization function called\n";
	Dynas[ DynaPosition]->Initialize();
	DynasInited[ DynaName ] = true;
  }
  return Dynas[ DynaPosition ]->RunFunc(func);
}
