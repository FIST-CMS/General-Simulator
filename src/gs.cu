#include "hip/hip_runtime.h"

#include"pub.h"
#include"pub_main.h"

using namespace GS_NS;
using namespace DATA_NS;


GS::GS(){ 
  for (int i=0;i<DynaMax;i++){
	Dyna[i] = NULL;
	IsDynaInit[i] = false;
  }
}


GS::~GS(){
  for (int i=0;i<DynaMax;i++)
	if (Dyna[i]!=NULL) delete Dyna[i];
}

int GS::SetInfo(string ss){ss>>InfoSteps; InfoMode  = ss; return 0;}

int GS::InfoOut(){
  string sm,ss; sm=InfoMode;
  GV<0>::LogAndError<<"Step  "<<CurrentStep<<" \t";
  while (sm !=""){
	sm>>ss;
	GV<0>::LogAndError<<Dyna[DynaID]->Get(ss)<<" \t";
  }
  GV<0>::LogAndError<<"\n";
  return 0;
}

int GS::SetDump(string ss){
  ss>>DumpFolder>>DumpSteps;
  string cmd="if [ -s "+DumpFolder+" ]; then echo \'dump to folder"+DumpFolder+"\'; else mkdir "+DumpFolder+"; fi";
  if (system(cmd.c_str()));
  DumpMode	 = ss;
  return 0;
}

int GS::DumpOut(){
  string sm,ss;
  sm=DumpMode;
  while (sm !=""){
	sm>>ss;
	if (Datas(ss) != NULL){
	  if (Datas[ss].Position==Data_DEV)
		Datas[ss].DeviceToHost();
	  string file; file<<DumpFolder<<"/"<<ss<<"."<<CurrentStep<<".data";
	  if (CurrentStep==TotalSteps) {
		file="";
		file<<DumpFolder<<"/"<<ss<<".final.data";
	  }
	  Datas[ss].DumpFile(file);
	}else GV<0>::LogAndError<<"Error: data "<<ss<<" does not exist!\n";
  }
  return 0;
}

int GS::SetSys(string ss){
  string sys;
  int id=0;
  ss>>sys>>id;
  DynaID=id;
  if ( id >=DynaMax || id<0 ){
	DynaID=(id%DynaMax+DynaMax)%DynaMax;
	GV<0>::LogAndError<<"Dyna ID is out of the allowed index range. It is moduled to "<<DynaID<<"\n";
	return -1;
  }
  if (Dyna[DynaID]!=NULL) // if the DynaID dyna has been created then delete it.
	delete Dyna[id];

  bool hassys=false;
  ///////////////////////////////////
  //GS_SYS_DEFINE_START
  if (sys == "cores"	) { Dyna[DynaID] = new Dynamics_cores; hassys=true; }
  if (sys == "diffuse"	) { Dyna[DynaID] = new Dynamics_diffuse; hassys=true; }
  if (sys == "mart"	) { Dyna[DynaID] = new Dynamics_mart; hassys=true; }
  if (sys == "pow2"	) { Dyna[DynaID] = new Dynamics_pow2; hassys=true; }
  if (sys == "pow"	) { Dyna[DynaID] = new Dynamics_pow; hassys=true; }
  if (sys == "stress"	) { Dyna[DynaID] = new Dynamics_stress; hassys=true; }
  if (sys == "xxx"	) { Dyna[DynaID] = new Dynamics_xxx; hassys=true; }
  //GS_SYS_DEFINE_END
  ///////////////////////////////////
  if (!hassys) {
	GV<0>::LogAndError<<"Error: System "<<ss<<"is not recognized\n";
	return -1;
  }
  return 0;
}

//////////////////////////////////////////////////
int GS::Set(string ss){
  string var;
  ss>>var;
  Vars[var] = ss;
  return 0;
}

int GS::Link(string ss){
  string target,link;
  ss>>target>>link;
  Vars.Link(target,link);
  return 0;
}

int GS::Read(string ss){
  string file,var; ss>>var>>file;
  ifstream ifs;
  ifs.open(file.c_str());
  if (ifs){
	ifs>>Datas[var];
	ifs.close();
  }else{
	GV<0>::LogAndError<<"Error: File "<<file<<" not found!\n";
	return -1;
  }
  return 0;
}

int GS::ReadHere(string name, string &arrays){
  int n; arrays>>n;
  int *dim = new int[n+1]; dim[0]=n;
  for (int i=1; i<=n; i++) arrays>>dim[i];
  Datas[name].Init(dim,Data_HOST);
  for (int i=0; i<Datas[name].N(); i++)
	arrays>>Datas[name].Arr[i];
  return 0;
}

int GS::Write(string sm){
  string ss,file;
  while (sm !=""){
	sm>>ss>>file;
	if (Datas(ss) != NULL){
	  if (Datas[ss].Position==Data_DEV)
		Datas[ss].DeviceToHost();
	  if (file == "") file<<ss<<".data";
	  Datas[ss].DumpFile(file);
	}else GV<0>::LogAndError<<"Error: data "<<ss<<" does not exist!\n";
  }
  return 0;
}

int GS::WriteHere(string sm){
  string ss;
  while (sm !=""){
	sm>>ss;
	if (Datas(ss) != NULL){
	  if (Datas[ss].Position==Data_DEV)
		Datas[ss].DeviceToHost();
	  cout<<Datas[ss];
	  GV<0>::LogAndError.Logofs<<Datas[ss];
	}if (Vars(ss) != NULL){
	  cout<<Vars[ss];
	  GV<0>::LogAndError.Logofs<<Vars[ss];
	}else GV<0>::LogAndError<<"Error: data "<<ss<<" does not exist!\n";
  }
  return 0;
}


int GS::Run(string ss){
  int totalsteps=1;
  ss>>totalsteps; TotalSteps=totalsteps; if (totalsteps <= 0 ) return -1;
  //what dyna to use???? defined in sys
  // the data is passed by reference ( big )
  // paras is passed by value ( small )
  Dyna[DynaID]->Datas = &Datas; 
  Dyna[DynaID]->Vars= Vars; 
  if ( !IsDynaInit[DynaID] ){
	Dyna[DynaID]->Initialize();
	IsDynaInit[ DynaID ] = true;
  }
  //////////////////////////////////////////////////////////////
  int  infoInterval;
  if ( InfoSteps==0) infoInterval=totalsteps+1;else infoInterval = totalsteps/InfoSteps;
  if ( infoInterval==0 ) infoInterval=1;
  int  dumpInterval;
  if ( DumpSteps== 0 ) dumpInterval= totalsteps+1; else dumpInterval  = totalsteps/DumpSteps;
  if ( dumpInterval == 0 ) dumpInterval =1;
  //////////////////////////////////////////////////////////////
  string mode=InfoMode,tempss;
  GV<0>::LogAndError<<"Info \t\t"; while (mode!=""){ mode>>tempss; GV<0>::LogAndError<<tempss<<"\t"; };
  GV<0>::LogAndError<<"\n";
  for (int i=1;i<=totalsteps;i++) {
	CurrentStep = i; // this will be used in dump and info to identity the progress
	Dyna[DynaID]->Fix(real(i)/totalsteps);
	Dyna[DynaID]->Calculate();

	if (i % infoInterval==0|| i==totalsteps) InfoOut();
	if (i % dumpInterval  ==0|| i==totalsteps) DumpOut();
  }
  //////////////////////////////////////////////////////////////
  return 0;
}

int GS::RunFunc(string func){
  if (Dyna[DynaID]==NULL){
	GV<0>::LogAndError<<"Error: no system set, runfunc commands not available\n";
	return Code_ERR;
  }
  Dyna[DynaID]->Datas = &Datas; 
  Dyna[DynaID]->Vars= Vars; 
  if ( !IsDynaInit[DynaID] ){
	GV<0>::LogAndError<<"Warning: Initialization function called\n";
	Dyna[DynaID]->Initialize();
	IsDynaInit[ DynaID ] = true;
  }
  Dyna[DynaID]->RunFunc(func);
  return 0;
}
