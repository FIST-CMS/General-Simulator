#include "hip/hip_runtime.h"
#define DEBUG 0
#include"pub.h"
#include<hiprand.h>
#include<hipfft/hipfft.h>
#include"random.h"
#include"gtensorb.h"
#include"dynamics.h"
#include"dynamics_mart.h"


using namespace GS_NS;
using namespace DATA_NS;

int Dynamics_mart::Initialize(){
  /////////////////////////////////////////////////////////
  //para setting should be finished before or within this function
  string ss;
  ss=(*Vars)["gridsize"];    			if (ss!="") ss>>nx>>ny>>nz>>dx>>dy>>dz;
<<<<<<< HEAD
  DeltaTime =0.01f; (*Vars)["deltatime"]>>=DeltaTime;
  TransitionTemperature=450.0f; (*Vars)["transitiontemperature"]>>=TransitionTemperature;

  weightGradient= 2.5f; (*Vars)["weightgradient"]>>=weightGradient;
  weightChemical= 1.0f; (*Vars)["weightchemical"]>>=weightChemical;
  weightElastic=  100000.0f;  (*Vars)["weightelastic"]>>=weightElastic;
  weightDislocation= 1.0f; (*Vars)["weightdislocation"]>>=weightDislocation;
  weightNoise = 0.0001f; (*Vars)["weightnoise"]>>=weightNoise;
=======

  weightExternal= 0.f;
  weightDislocation= 0.01f; (*Vars)["weightdislocation"]>>=weightDislocation;
  weightNoise = 1.0f; (*Vars)["weightnoise"]>>=weightNoise;
  DeltaTime =0.01f; (*Vars)["deltatime"]>>=DeltaTime;
  weightGradient= 2.5f; (*Vars)["weightgradient"]>>=weightGradient;
  weightChemical= 1.0f; (*Vars)["weightchemical"]>>=weightChemical;
  weightElastic=  100000.0f;  (*Vars)["weightelastic"]>>=weightElastic;
  TransitionTemperature=450.0f; (*Vars)["transitiontemperature"]>>=TransitionTemperature;
>>>>>>> origin/master
  /////////////////////////////////////////////////////////
  LPC[2]=32.05f; LPC[3]=37.5f;
  ss=(*Vars)["coefficient"]; if (ss!="") ss>>LPC[1]>>LPC[2]>>LPC[3];
  /////////////////////////////////////////////////////////
  StrainTensor = &((*Datas)["varianttensor"]);
  if (StrainTensor->Arr == NULL){
	GV<0>::LogAndError<<"Error: variants' strain tensor does not set while initialize dynamics\n";
	return -1;
  }
  VariantN = StrainTensor->Dimension[1];
  /////////////////////////////////////////////////////////
  // it is called to initialize the --run-- function
  // allocate memory initial size and default values
  //Init(3,nx,ny,nz,Data_NONE);
  SetCalPos(Data_HOST_DEV);
  //Eta=eta; // a pointer assign, not value or memory operation
  Eta = &((*Datas)["eta"]); // may create here
  if ( Eta->Arr == NULL ){
	Eta->Init(4,VariantN,nx,ny,nz,Data_HOST_DEV);
	SetCalPos(Data_DEV);
	(*Eta)=0.0f; 
  }else{ Eta->HostToDevice();}
  /////////////////////////////////////////////////////////
  int dim[5]={3,nx,ny,nz};
  int dimN[6]={4,VariantN,nx,ny,nz};

<<<<<<< HEAD
  Noise.InitRandom(4,VariantN,nx,ny,nz);
=======
  Noise.InitRandom(4,VariantN,nx,ny,nz, 0, 0.001, 0,0);
>>>>>>> origin/master

  Gradient.Init(dimN,Data_HOST_DEV);
  GradientForce.Init(dimN,Data_HOST_DEV);

  ChemicalForce.Init(dimN,Data_HOST_DEV);
<<<<<<< HEAD
  //ChemicalFreeEnergy.Init(dim,Data_DEV);
=======
>>>>>>> origin/master
  /////////////////////////////////////////////////////////////////
  real C00=3.5f, C01=1.5f, C33=1.0f;//defaut values
  Data<Real> cijkl(4,3,3,3,3,Data_HOST_DEV); SetCalPos(Data_HOST);
  cijkl=0.0f;
  cijkl(0,0,0,0) =C00; cijkl(1,1,1,1) =C00; cijkl(2,2,2,2) =C00;
  cijkl(0,0,1,1) =C01; cijkl(1,1,2,2) =C01; cijkl(2,2,0,0) =C01;
  cijkl(0,1,0,1) =C33; cijkl(1,2,1,2) =C33; cijkl(2,0,2,0) =C33;
  cijkl(1,1,0,0) =C01; cijkl(2,2,1,1) =C01; cijkl(0,0,2,2) =C01;
  cijkl(1,0,1,0) =C33; cijkl(0,1,1,0) =C33; cijkl(1,0,0,1) =C33;
  cijkl(2,1,2,1) =C33; cijkl(2,1,1,2) =C33; cijkl(1,2,2,1) =C33;
  cijkl(0,2,0,2) =C33; cijkl(0,2,2,0) =C33; cijkl(2,0,0,2) =C33;			// 
  Data<Real> *modulus; modulus=&((*Datas)["modulus"]);
  if ( modulus->Arr != NULL )
	cijkl = (*modulus);
  ///////////////////////////
  Data<Real> vstrain(3,2*VariantN,3,3,Data_HOST_DEV);
  for (int i=0; i<2*VariantN*3*3; i++)
	vstrain.Arr[i]=StrainTensor->Arr[i%(VariantN*3*3)];
  ///////////////////////////
  GV<0>::LogAndError<<"Space structure tensor is calculating\n";
  B.InitB(VariantN,VariantN,nx,ny,nz,dx.Re,dy.Re,dz.Re,vstrain,cijkl); 
  GV<0>::LogAndError<<"Calculating of space structure tensor relating to the elastic terms is finished\n";
<<<<<<< HEAD
=======
  if (DEBUG){ 
    B.DeviceToHost(); B.DumpFile("data.b"); 
  }
>>>>>>> origin/master
  /////////////////////////////////////////////////////////////////
  ElasticForce.Init(dimN,Data_HOST_DEV);
  Eta_RT.Init(dimN,Data_HOST_DEV);
  Eta_CT.Init(dimN,Data_HOST_DEV);
  ReciprocalTerm.Init(dimN,Data_HOST_DEV);
  /////////////////////////////////////////////////////////////////
  int rank=3,ns[3]={nx,ny,nz},dist=nx*ny*nz,stride=1;
  GV<0>::LogAndError<<"Cuda fft plan is to creat\n";
  if (hipfftPlanMany(&planAll_Cuda,rank,ns,ns,stride,dist,ns,stride,dist,HIPFFT_C2C,VariantN)==HIPFFT_SUCCESS)
	GV<0>::LogAndError<<"Cuda fft plan is created\n";
  else GV<0>::LogAndError<<"Cuda fft plan fails to create\n";

  Defect = &((*Datas)["defect"]);
  if (Defect->Arr==NULL){
	Defect->Init(dim,Data_HOST_DEV); // it will be init when read in
	SetCalPos(Data_HOST_DEV);
	(*Defect)=0.0f;
  }else { Defect->HostToDevice();}
  
  /////////////////////////////////////////////////////////////////
  // the 6 component form should be rewriten to 3*3 form
  DislocationStressOForm= &((*Datas)["dislocationstress"]);
  int dim33[6]={5,3,3,nx,ny,nz}; 
  DislocationStress.Init(dim33,Data_HOST_DEV);//it will be also init when read in
  if (DislocationStressOForm->Arr==NULL){
	SetCalPos(Data_HOST_DEV);
	(DislocationStress)=0.0f;
  }else {
	SetCalPos(Data_HOST);
	for (int i=0; i<nx; i++)
	  for (int j=0; j<ny; j++)
		for (int k=0; k<nz; k++){
		  DislocationStress(0,0,i,j,k)=(*DislocationStressOForm)(0,i,j,k);
		  DislocationStress(0,1,i,j,k)=(*DislocationStressOForm)(1,i,j,k);
		  DislocationStress(0,2,i,j,k)=(*DislocationStressOForm)(2,i,j,k);
		  DislocationStress(1,1,i,j,k)=(*DislocationStressOForm)(3,i,j,k);
		  DislocationStress(1,2,i,j,k)=(*DislocationStressOForm)(4,i,j,k);
		  DislocationStress(2,2,i,j,k)=(*DislocationStressOForm)(5,i,j,k);
		  DislocationStress(1,0,i,j,k)=(*DislocationStressOForm)(1,i,j,k);
		  DislocationStress(2,0,i,j,k)=(*DislocationStressOForm)(2,i,j,k);
		  DislocationStress(2,1,i,j,k)=(*DislocationStressOForm)(4,i,j,k);
		}
	DislocationStress.HostToDevice();
  }

  DislocationForce.Init(dimN,Data_HOST_DEV);
  DislocationForceConst.Init(dimN,Data_HOST_DEV);
  DislocationForceInit(); //this only need one calculation
<<<<<<< HEAD

  if (1&&DEBUG){
    SetCalPos(Data_HOST);
    for (int i=0;i<nx;i++)
	 for (int j=0;j<ny;j++)
	   for (int k=0;k<nz;k++)
		(*Eta)(0,i,j,k)=.01f;
    for (int v=1;v<4;v++)
	 for (int i=0;i<nx;i++)
	   for (int j=0;j<ny;j++)
		for (int k=0;k<nz;k++)
		  (*Eta)(v,i,j,k)=0.f;
    Eta->HostToDevice();
  }
=======
>>>>>>> origin/master
  return 0;
}

Dynamics_mart::Dynamics_mart(){
}
Dynamics_mart::~Dynamics_mart(){
  if (planAll_Cuda) hipfftDestroy(planAll_Cuda);
}

__global__ void Grad_Mart_Kernel(Real *Gradient_arr,  Real* Eta_arr,int *dim, Real dx, Real dy, Real dz){
  // (* 4 128 128) (* 4 128)
  int x=blockIdx.x, y= blockIdx.y, z=threadIdx.x, v=blockIdx.z;
  /**/PPart(Gradient_arr,dim,v,x,y,z)=
	 (PPart(Eta_arr,dim,v,x+1,y,z)+PPart(Eta_arr,dim,v,x-1,y,z)-2*PPart(Eta_arr,dim,v,x,y,z))/(2.0f* dx)/3.0f
	+(PPart(Eta_arr,dim,v,x,y+1,z)+PPart(Eta_arr,dim,v,x,y-1,z)-2*PPart(Eta_arr,dim,v,x,y,z))/(2.0f* dy)/3.0f	
	+(PPart(Eta_arr,dim,v,x,y,z+1)+PPart(Eta_arr,dim,v,x,y,z-1)-2*PPart(Eta_arr,dim,v,x,y,z))/(2.0f* dz)/3.0f	; // */
}
int Dynamics_mart::GradientCalculate(){
  dim3 bn(nx,ny,VariantN);
  dim3 tn(nz);
  Grad_Mart_Kernel<<<bn,tn>>>(Gradient.Arr_dev,  Eta->Arr_dev, Eta->Dimension_dev, dx,dy,dz);
  return 0;
}


int Dynamics_mart::GradientForceCalculate(){
  GradientCalculate();
<<<<<<< HEAD
  GradientForce= weightGradient* Gradient;
=======
  GradientForce= Gradient;
>>>>>>> origin/master
  return 0;
}

int Dynamics_mart::LPCConstruct(){
  LPC[1]=0.02f *(Temperature-TransitionTemperature);
  return 0;
}
<<<<<<< HEAD

__global__ void ChemicalFreeEnergy_mart_kernel(Real*cfn,Real*eta, int VariantN){
  int x=blockIdx.x, y=blockIdx.y, z=threadIdx.x, nx=gridDim.x, ny=gridDim.y, nz=blockDim.x;
  cfn[(x*ny+y)*nz+z]=0.f;
  for (int i=0; i<VariantN; i++) cfn[(x*ny+y)*nz+z]+=(eta[((i*nx+x)*ny+y)*nz+z]^2);
  cfn[(x*ny+y)*nz+z]=(cfn[(x*ny+y)*nz+z]^3)/6.0f;
  for (int i=0; i<VariantN; i++)
    cfn[(x*ny+y)*nz+z]+=(eta[((i*nx+x)*ny+y)*nz+z]^2)/2.0f+(eta[((i*nx+x)*ny+y)*nz+z]^4)/4.0f;
}

__global__ void ChemiFor_Mart_Kernel(Real*ChemiForce_arr,
    Real*Eta_arr,Real a1,Real a2,Real a3,Real weight){// n1*n2*n3 each variant have an driving force
=======
__global__ void ChemiFor_Mart_Kernel(Real*ChemiForce_arr,
    Real*Eta_arr,Real a1,Real a2,Real a3){// n1*n2*n3 each variant have an driving force
>>>>>>> origin/master
  int x=blockIdx.x, y=blockIdx.y, z=threadIdx.x,v=blockIdx.z, nx=gridDim.x, ny=gridDim.y, nz=blockDim.x,nv=gridDim.z;
  int tid=((v*nx+x)*ny+y)*nz+z;
  // request the same memory at the same time will lead to nan at the wrost situation
  ChemiForce_arr[tid]=0.0;
<<<<<<< HEAD
  sqrt(abs(a1/(a2-a3)));
  for (int i=0;i<nv;i++) ChemiForce_arr[tid]+=((sqrt(abs(a1/(a2-a3)))*Eta_arr[((i*nx+x)*ny+y)*nz+z])^2);
  if (Eta_arr[tid]>=0){
    ChemiForce_arr[tid]= 
	 -weight*(sqrt(abs(a1/(a2-a3)))*Eta_arr[tid])
	 *( a1 -a2*((sqrt(abs(a1/(a2-a3)))*Eta_arr[tid])^2) +a3*ChemiForce_arr[tid]);
  }else{//<0 power 2
    ChemiForce_arr[tid]= 1000.0*weight*a2*Eta_arr[tid]*Eta_arr[tid] ;
=======
  if (Eta_arr<=0){
    for (int i=0;i<nv;i++) ChemiForce_arr[tid]+=(Eta_arr[((i*nx+x)*ny+y)*nz+z]^2);
    ChemiForce_arr[tid]= Eta_arr[tid]*( a1 -a2*(Eta_arr[tid]^2) +a3*ChemiForce_arr[tid]);
  }else{
    for (int i=0;i<nv;i++) ChemiForce_arr[tid]+=(Eta_arr[((i*nx+x)*ny+y)*nz+z]);
    ChemiForce_arr[tid]= Eta_arr[tid]*( a1 -a2*Eta_arr[tid] +a3*ChemiForce_arr[tid]);
>>>>>>> origin/master
  }
}

int Dynamics_mart::ChemicalForceCalculate(){
  /////////////////////////
<<<<<<< HEAD
  dim3 bvn(nx,ny,VariantN);
  dim3 bn(nx,ny);
  dim3 tn(nz);
  LPCConstruct();
  ChemiFor_Mart_Kernel<<<bvn,tn>>>(ChemicalForce.Arr_dev, Eta->Arr_dev, LPC[1], LPC[2], LPC[3],weightChemical);
  //ChemicalFreeEnergy_mart_kernel<<<bn,tn>>>(ChemicalFreeEnergy.Arr_dev,Eta->Arr_dev,VariantN);
=======
  dim3 bn(nx,ny,VariantN);
  dim3 tn(nz);
  LPCConstruct();
  ChemiFor_Mart_Kernel<<<bn,tn>>>(ChemicalForce.Arr_dev, Eta->Arr_dev, LPC[1], LPC[2], LPC[3]);
>>>>>>> origin/master
  return 0;
} //(* 2373 0.9)

__global__ void ElaFor_Mart_Kernel(Complex *ReTerm,Complex*Eta_sq,Real* B){
  int  nx=gridDim.x, ny=gridDim.y, nz=blockDim.x,nv=gridDim.z;
  int x=blockIdx.x, y=blockIdx.y, z=threadIdx.x, v=blockIdx.z;
  ReTerm[((v*nx+x)*ny+y)*nz+z] = 0;
<<<<<<< HEAD
  for (int i=0;i<nv;i++)
=======
  for (int i=0;i<v;i++)
>>>>>>> origin/master
	ReTerm[((v*nx+x)*ny+y)*nz+z] +=  B[(((v*nv+i)*nx+x)*ny+y)*nz+z]* Eta_sq[((i*nx+x)*ny+y)*nz+z];
}
int Dynamics_mart::ElasticForceCalculate(){
  SetCalPos(Data_DEV);
  Eta_CT=(*Eta)*(*Eta); //Store it in the buffer area
  hipfftExecC2C(planAll_Cuda,(hipfftComplex*)Eta_CT.Arr_dev,(hipfftComplex*)Eta_CT.Arr_dev,HIPFFT_FORWARD);
  dim3 bn(nx,ny,VariantN);
  dim3 tn(nz);
  Eta_CT = Eta_CT/Eta_CT.N()*VariantN;
  ElaFor_Mart_Kernel<<<bn,tn>>>(ReciprocalTerm.Arr_dev,Eta_CT.Arr_dev,B.Arr_dev);
  hipfftExecC2C(planAll_Cuda,(hipfftComplex*)ReciprocalTerm.Arr_dev,(hipfftComplex*)ReciprocalTerm.Arr_dev,HIPFFT_BACKWARD);
<<<<<<< HEAD
  ElasticForce = - weightElastic* ReciprocalTerm* (*Eta);
=======
  ElasticForce = ReciprocalTerm* (*Eta);
  if (0){
    Eta_CT.DeviceToHost();
    ElasticForce.DeviceToHost();

    Eta_CT.DumpFile("data.eta_squre");
    ElasticForce.DumpFile("data.r_term");
  }
>>>>>>> origin/master
  return 0;
}

int Dynamics_mart::DislocationForceInit(){
  SetCalPos(Data_HOST);
  for (int saq=0;saq<VariantN;saq++){
    for (int i=0;i<nx;i++)
      for (int j=0;j<ny;j++)
        for (int k=0;k<nz;k++){
          DislocationForceConst(saq,i,j,k)=0;
          for (int sa=0;sa<3;sa++)
            for (int sap=0;sap<3;sap++){
              DislocationForceConst(saq,i,j,k)=DislocationForceConst(saq,i,j,k)+25.0f*DislocationStress(sa,sap,i,j,k)*(*StrainTensor)(saq,sa,sap);
            }
        }
  }
  DislocationForceConst.HostToDevice();
  SetCalPos(Data_DEV);
  return 0;
}

int Dynamics_mart::DislocationForceCalculate(){
  SetCalPos(Data_DEV);
<<<<<<< HEAD
  DislocationForce = -weightDislocation*DislocationForceConst*(*Eta);
=======
  DislocationForce=DislocationForceConst*(*Eta);
>>>>>>> origin/master
  return 0;
}

__global__ void Block_Mart_Kernel(Real *Eta_arr, Real *Defect_arr){
  int pn=
	blockIdx.y*gridDim.z*blockDim.x +blockIdx.z*blockDim.x +threadIdx.x;
  int pvn= blockIdx.x *gridDim.y *gridDim.z * blockDim.x +pn;
  Eta_arr[pvn]=Eta_arr[pvn]*(1.0f-Defect_arr[pn]);
}

int Dynamics_mart::Block(){
  dim3 bn(VariantN,nx,ny);
  dim3 tn(nz);
  Block_Mart_Kernel<<<bn,tn>>>(Eta->Arr_dev,Defect->Arr_dev);

  return 0;
}

int Dynamics_mart::Calculate(){
<<<<<<< HEAD
  if (1&&DEBUG) {
    Eta->DeviceToHost();
    Eta->DumpFile("data.eta.before");
  }

  SetCalPos(Data_DEV);
=======
>>>>>>> origin/master
  string ss;
  (*Vars)["temperature"]>>=Temperature; 
  GradientForceCalculate();
  ChemicalForceCalculate();
  ElasticForceCalculate();
  DislocationForceCalculate();
  ////////////////////////////
  Eta_RT=0.f;
<<<<<<< HEAD
  Eta_RT += GradientForce;  
  Eta_RT += ChemicalForce;
  Eta_RT += ElasticForce;
  Eta_RT += DislocationForce; 
  //////////////
  Noise.NewNormal_device(); Eta_RT += weightNoise * Noise;
  //////////////
=======
  if (weightGradient>0) Eta_RT += weightGradient*GradientForce; 
  if (weightChemical>0) Eta_RT += (0-weightChemical)*ChemicalForce;
  if (weightElastic>0) Eta_RT  += (0-weightElastic)*ElasticForce;
  if (weightDislocation>0) Eta_RT += (0-weightDislocation)*DislocationForce; 
  if (weightExternal>0) Eta_RT += (0-weightExternal)*ExternalForce; 
  if (weightNoise>0){
	Noise.NewNormal_device();
	Eta_RT += weightNoise*0.0001* Noise;
  }
>>>>>>> origin/master
  (*Eta) += DeltaTime* Eta_RT;
  //defect block
  Block();
  ///////////
<<<<<<< HEAD
  if (1&&DEBUG) {
    GradientForce.DeviceToHost();
    ChemicalForce.DeviceToHost();
    ElasticForce.DeviceToHost();
    DislocationForce.DeviceToHost();
=======
  if (DEBUG) {
    GradientForce.DeviceToHost();
    ChemicalForce.DeviceToHost();
    ElasticForce.DeviceToHost();
>>>>>>> origin/master
    Eta->DeviceToHost();
    ///
    GradientForce.DumpFile("data.gradient");
    ChemicalForce.DumpFile("data.chemical");
    ElasticForce.DumpFile("data.elastic");
<<<<<<< HEAD
    DislocationForce.DumpFile("data.dislocation");
    Eta->DumpFile("data.eta");
    Eta_RT = weightNoise * Noise; Eta_RT.DeviceToHost(); Eta_RT.DumpFile("data.noise");
    //string ss; cin>>ss;
=======
    Eta->DumpFile("data.eta");
>>>>>>> origin/master
  }
  ///////////////////////////////
  return 0;
}

int Dynamics_mart::RunFunc(string funcName){ return 0; }

int Dynamics_mart::Fix(real progress){
  string ss,mode;
  ss = (*Vars)["fix"];
  do{
	ss>>mode;
	if      (mode=="temperature"	){
<<<<<<< HEAD
	  real st,et; ss>>st>>et;
=======
	  real st,et; //start and end temperature
	  ss>>st>>et;
>>>>>>> origin/master
	  ((*Vars)["temperature"])<<=(st+ progress*(et- st));
	} else if (mode=="pressure"		){ 
	} else{
	  GV<0>::LogAndError<<"Error: fix style "<<mode<<" does not find!\n";
	}
  } while ( ss != "");

  return 0;
}

string Dynamics_mart::Get(string ss){ // return the statistic info.
  string var; ss>>var;
  if (var == "temperature") return ToString(Temperature); 
<<<<<<< HEAD
  if (var == "eta") return ToString(Eta->TotalDevice()/Eta->N()); 
  if (var == "gradient"   ) return ToString(GradientForce.TotalDevice()/GradientForce.N());
  if (var == "chemical"   ) return ToString(ChemicalForce.TotalDevice()/ChemicalForce.N());
  if (var == "elastic"    ) return ToString(ElasticForce.TotalDevice()/ElasticForce.N());
  if (var == "dislocation") return ToString(DislocationForce.TotalDevice()/DislocationForce.N());
  //if (var == "chemical.free.energy") return ToString(ChemicalFreeEnergy.TotalDevice()/ChemicalFreeEnergy.N());
=======
>>>>>>> origin/master
  else return "nan";
}

