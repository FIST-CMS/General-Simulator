#include "hip/hip_runtime.h"
#define DEBUG 0
#include"pub.h"
#include"../include/datamain.th"
#include<hiprand.h>
#include<hipfft/hipfft.h>
#include"random.h"
//#include"tensorb.h"
#include"gtensorb.h"

#include"dynamics.h"
#include"dynamics_mart.h"


using namespace GS_NS;
using namespace DATA_NS;

int Dynamics_mart::Initialize(){
  /////////////////////////////////////////////////////////
  //para setting should be finished before or within this function
  string ss;
  ss=Vars["gridsize"];    			if (ss!="") ss>>nx>>ny>>nz>>dx>>dy>>dz;


  weightExternal= 0.f;
  weightDislocation= 0.01f; Vars["weightdislocation"]>>=weightDislocation;
  weightNoise = 1.0f; Vars["weightnoise"]>>=weightNoise;
  DeltaTime =0.01f; Vars["deltatime"]>>=DeltaTime;
  weightGradient= 2.5f; Vars["weightgradient"]>>=weightGradient;
  weightChemical= 1.0f; Vars["weightchemical"]>>=weightChemical;
  weightElastic=  100000.0f;  Vars["weightelastic"]>>=weightElastic;
  TransitionTemperature=450.0f; Vars["transitiontemperature"]>>=TransitionTemperature;
  /////////////////////////////////////////////////////////
  LPC[2]=32.05f; LPC[3]=37.5f;
  ss=Vars["coefficient"]; if (ss!="") ss>>LPC[1]>>LPC[2]>>LPC[3];
  /////////////////////////////////////////////////////////
  StrainTensor = &((*Datas)["varianttensor"]);
  if (StrainTensor->Arr == NULL){
	GV<0>::LogAndError>>"Error: variants' strain tensor does not set while initialize dynamics\n";
	return -1;
  }
  VariantN = StrainTensor->Dimension[1];
  /////////////////////////////////////////////////////////
  // it is called to initialize the --run-- function
  // allocate memory initial size and default values
  //Init(3,nx,ny,nz,Data_NONE);
  SetCalPos(Data_HOST_DEV);
  //Eta=eta; // a pointer assign, not value or memory operation
  Eta = &((*Datas)["eta"]); // may create here
  if ( Eta->Arr == NULL ){
	Eta->Init(4,VariantN,nx,ny,nz,Data_HOST_DEV);
	SetCalPos(Data_DEV);
	(*Eta)=0.0f; 
  }else{ Eta->HostToDevice();}
  /////////////////////////////////////////////////////////
  int dim[5]={3,nx,ny,nz};
  int dimN[6]={4,VariantN,nx,ny,nz};

  Noise.InitRandom(4,VariantN,nx,ny,nz, 0, 0.001, 0,0);

  Gradient.Init(dimN,Data_HOST_DEV);
  GradientEnergy.Init(dim,Data_HOST_DEV);
  GradientForce.Init(dimN,Data_HOST_DEV);

  ChemicalEnergy.Init(dim,Data_HOST_DEV);
  ChemicalForce.Init(dimN,Data_HOST_DEV);
  /////////////////////////////////////////////////////////////////
  real C00=3.5f, C01=1.5f, C33=1.0f;//defaut values
  Data<Real> cijkl(4,3,3,3,3,Data_HOST_DEV); SetCalPos(Data_HOST);
  cijkl(0,0,0,0) =C00; cijkl(1,1,1,1) =C00; cijkl(2,2,2,2) =C00;
  cijkl(0,0,1,1) =C01; cijkl(1,1,2,2) =C01; cijkl(2,2,0,0) =C01;
  cijkl(0,1,0,1) =C33; cijkl(1,2,1,2) =C33; cijkl(2,0,2,0) =C33;
  cijkl(1,1,0,0) =C01; cijkl(2,2,1,1) =C01; cijkl(0,0,2,2) =C01;
  cijkl(1,0,1,0) =C33; cijkl(0,1,1,0) =C33; cijkl(1,0,0,1) =C33;
  cijkl(2,1,2,1) =C33; cijkl(2,1,1,2) =C33; cijkl(1,2,2,1) =C33;
  cijkl(0,2,0,2) =C33; cijkl(0,2,2,0) =C33; cijkl(2,0,0,2) =C33;			// 
  Data<Real> *modulus; modulus=&((*Datas)["modulus"]);
  if ( modulus->Arr != NULL )
	cijkl = (*modulus);
  ///////////////////////////
  Data<Real> vstrain(3,2*VariantN,3,3,Data_HOST_DEV);
  for (int i=0; i<2*VariantN*3*3; i++)
	vstrain.Arr[i]=StrainTensor->Arr[i%(VariantN*3*3)];
  ///////////////////////////
  GV<0>::LogAndError<<"Space structure tensor is calculating\n";
  B.InitB(VariantN,VariantN,nx,ny,nz,dx.Re,dy.Re,dz.Re,vstrain,cijkl); 
  GV<0>::LogAndError<<"Calculating of space structure tensor relating to the elastic terms is finished\n";
  /////////////////////////////////////////////////////////////////
  ElasticEnergy.Init(dim,Data_HOST_DEV);
  ElasticForce.Init(dimN,Data_HOST_DEV);
  Eta_RT.Init(dimN,Data_HOST_DEV);
  Eta_CT.Init(dimN,Data_HOST_DEV);
  ReciprocalTerm.Init(dimN,Data_HOST_DEV);
  /////////////////////////////////////////////////////////////////
  int rank=3,ns[3]={nx,ny,nz},dist=nx*ny*nz,stride=1;
  GV<0>::LogAndError<<"Cuda fft plan is to creat\n";
  if (hipfftPlanMany(&planAll_Cuda,rank,ns,ns,stride,dist,ns,stride,dist,HIPFFT_C2C,VariantN)==HIPFFT_SUCCESS)
	GV<0>::LogAndError<<"Cuda fft plan is created\n";
  else GV<0>::LogAndError<<"Cuda fft plan fails to create\n";

  Defect = &((*Datas)["defect"]);
  if (Defect->Arr==NULL){
	Defect->Init(dim,Data_HOST_DEV); // it will be init when read in
	SetCalPos(Data_HOST_DEV);
	(*Defect)=0.0f;
  }else { Defect->HostToDevice();}
  
  /////////////////////////////////////////////////////////////////
  // the 6 component form should be rewriten to 3*3 form
  DislocationStressOForm= &((*Datas)["dislocationstress"]);
  int dim33[6]={5,3,3,nx,ny,nz}; 
  DislocationStress.Init(dim33,Data_HOST_DEV);//it will be also init when read in
  if (DislocationStressOForm->Arr==NULL){
	SetCalPos(Data_HOST_DEV);
	(DislocationStress)=0.0f;
  }else {
	SetCalPos(Data_HOST);
	for (int i=0; i<nx; i++)
	  for (int j=0; j<ny; j++)
		for (int k=0; k<nz; k++){
		  DislocationStress(0,0,i,j,k)=(*DislocationStressOForm)(0,i,j,k);
		  DislocationStress(0,1,i,j,k)=(*DislocationStressOForm)(1,i,j,k);
		  DislocationStress(0,2,i,j,k)=(*DislocationStressOForm)(2,i,j,k);
		  DislocationStress(1,1,i,j,k)=(*DislocationStressOForm)(3,i,j,k);
		  DislocationStress(1,2,i,j,k)=(*DislocationStressOForm)(4,i,j,k);
		  DislocationStress(2,2,i,j,k)=(*DislocationStressOForm)(5,i,j,k);
		  DislocationStress(1,0,i,j,k)=(*DislocationStressOForm)(1,i,j,k);
		  DislocationStress(2,0,i,j,k)=(*DislocationStressOForm)(2,i,j,k);
		  DislocationStress(2,1,i,j,k)=(*DislocationStressOForm)(4,i,j,k);
		}
	DislocationStress.HostToDevice();
  }

  DislocationForce.Init(dimN,Data_HOST_DEV);
  DislocationForceConst.Init(dimN,Data_HOST_DEV);
  DislocationForceInit(); //this only need one calculation
  return 0;
}

Dynamics_mart::Dynamics_mart(){
  //default weight values
}
Dynamics_mart::~Dynamics_mart(){
  if (planAll_Cuda) hipfftDestroy(planAll_Cuda);
}

__global__ void Grad_Mart_Kernel(Real *Gradient_arr,  Real* Eta_arr,int *dim, Real dx, Real dy, Real dz){
  // (* 4 128 128) (* 4 128)
  int x=blockIdx.x, y= blockIdx.y, z=threadIdx.x, v=blockIdx.z;
  /**/PPart(Gradient_arr,dim,v,x,y,z)=
	 (PPart(Eta_arr,dim,v,x+1,y,z)+PPart(Eta_arr,dim,v,x-1,y,z)-2*PPart(Eta_arr,dim,v,x,y,z))/(2.0f* dx)/3.0f
	+(PPart(Eta_arr,dim,v,x,y+1,z)+PPart(Eta_arr,dim,v,x,y-1,z)-2*PPart(Eta_arr,dim,v,x,y,z))/(2.0f* dy)/3.0f	
	+(PPart(Eta_arr,dim,v,x,y,z+1)+PPart(Eta_arr,dim,v,x,y,z-1)-2*PPart(Eta_arr,dim,v,x,y,z))/(2.0f* dz)/3.0f	; // */
  /*PPart(Gradient_arr,dim,v,x,y,z)=
	 (PPart(Eta_arr,dim,v,x+1,y,z)+PPart(Eta_arr,dim,v,x-1,y,z)-2*PPart(Eta_arr,dim,v,x,y,z))/(dx^2)
	+(PPart(Eta_arr,dim,v,x,y+1,z)+PPart(Eta_arr,dim,v,x,y-1,z)-2*PPart(Eta_arr,dim,v,x,y,z))/(dy^2)	
	+(PPart(Eta_arr,dim,v,x,y,z+1)+PPart(Eta_arr,dim,v,x,y,z-1)-2*PPart(Eta_arr,dim,v,x,y,z))/(dz^2)	; // */

}
int Dynamics_mart::GradientCalculate(){
  dim3 bn(nx,ny,VariantN);
  dim3 tn(nz);
  Grad_Mart_Kernel<<<bn,tn>>>(Gradient.Arr_dev,  Eta->Arr_dev, Eta->Dimension_dev, dx,dy,dz);
  if (DEBUG) Gradient.DeviceToHost();
  return 0;
}

int Dynamics_mart::GradientEnergyCalculate(){
  return 0;
};

int Dynamics_mart::GradientForceCalculate(){
  GradientCalculate();
  GradientForce= Gradient;
  if (DEBUG) GradientForce.DeviceToHost();
  return 0;
}

int Dynamics_mart::LPCConstruct(){
  LPC[1]=0.02f *(Temperature-TransitionTemperature);
  return 0;
}
__global__ void ChemiEner_Mart_Kernel(Real *ChemE_arr,Real*Eta_arr,int v,Real a1, Real a2, Real a3){
  int tid=blockIdx.x* gridDim.y* blockIdx.x + blockIdx.y* blockDim.x + threadIdx.x ;
  int vn=gridDim.x*gridDim.y*blockDim.x;
  Real term1=0.f,term2=0.f,term3=0.f;
  for (int i=0; i<v; i++) term1=term1+(Eta_arr[tid+i*vn]^2);
  for (int i=0; i<v; i++) term2=term2+(Eta_arr[tid+i*vn]^4);
  term3=(term1^3);
  ChemE_arr[tid]= a1* term1 - a2* term2 + a3* term3;
}

int Dynamics_mart::ChemicalEnergyCalculate(){
  LPCConstruct();
  dim3 bn(nx,ny);
  dim3 tn(nz);
  LPCConstruct();
  ChemiEner_Mart_Kernel<<<bn,tn>>>(ChemicalEnergy.Arr_dev,Eta->Arr_dev,VariantN,LPC[1],LPC[2],LPC[3]);
  if (DEBUG) ChemicalEnergy.DeviceToHost();

  return 0;
}

__global__ void ChemiFor_Mart_Kernel(Real*ChemiForce_arr,Real*Eta_arr,int v,Real a1,Real a2,Real a3){// n1*n2*n3 each variant have an driving force
  int tid=blockIdx.x* gridDim.y* blockDim.x
	+ blockIdx.y* blockDim.x 
	+ threadIdx.x ;
  int vn=gridDim.x*gridDim.y*blockDim.x;
  Real term3=0;
  for (int i=0;i<v;i++)
	term3=term3+(Eta_arr[tid+i*vn]^2);
  for (int i=0;i<v;i++){
	if (Eta_arr[tid+i*vn]<0){//1 2 3 and for energy it is 2 3 4 
	  ChemiForce_arr[tid+i*vn]= Eta_arr[tid+i*vn]* ( a1 - a2* Eta_arr[tid+i*vn] + a3*term3 );
	}else{//1 3 5  //For energy it is 2 4 6
	  ChemiForce_arr[tid+i*vn]= Eta_arr[tid+i*vn]* ( a1 - a2*(Eta_arr[tid+i*vn]^2) + a3*(term3^2) );
	} //(sqrt (/ 9608  37.5))
  }
}

int Dynamics_mart::ChemicalForceCalculate(){
  dim3 bn(nx,ny);
  dim3 tn(nz);
  LPCConstruct();
  ChemiFor_Mart_Kernel<<<bn,tn>>>(ChemicalForce.Arr_dev, Eta->Arr_dev, VariantN, LPC[1], LPC[2], LPC[3]);
  if (DEBUG) ChemicalForce.DeviceToHost();
  return 0;
} //(* 2373 0.9)

__global__ void ElaFor_Mart_Kernel(Complex *ReTerm,Complex*Eta_sq,Real* B){
  int v=gridDim.x;
  int nv= gridDim.x* gridDim.y *gridDim.z *blockDim.x;
  int n=  gridDim.y *gridDim.z *blockDim.x;
  int pvv = blockIdx.x;
  int pn= blockIdx.y *gridDim.z*blockDim.x + blockIdx.z *blockDim.x + threadIdx.x;
  ReTerm[pvv*n +pn] = 0;
  for (int i=0;i<v;i++)
	ReTerm[pvv*n +pn] +=  B[pvv*nv + i*n +pn ]* Eta_sq[i*n + pn ];
}
int Dynamics_mart::ElasticForceCalculate(){
  SetCalPos(Data_DEV);
  Eta_CT=(*Eta)*(*Eta); //Store it in the buffer area
  hipDeviceSynchronize();
  if (DEBUG) Eta_CT.DeviceToHost();
  hipfftExecC2C(planAll_Cuda,(hipfftComplex*)Eta_CT.Arr_dev,(hipfftComplex*)Eta_CT.Arr_dev,HIPFFT_FORWARD);
  if (DEBUG) Eta_CT.DeviceToHost();
  dim3 bn(VariantN,nx,ny);
  dim3 tn(nz);
  hipDeviceSynchronize();
  Eta_CT = Eta_CT/Eta_CT.N()*VariantN;
  hipDeviceSynchronize();
  if (DEBUG) {Eta_CT.DeviceToHost();}
  ElaFor_Mart_Kernel<<<bn,tn>>>(ReciprocalTerm.Arr_dev,Eta_CT.Arr_dev,B.Arr_dev);
  hipDeviceSynchronize();
  if (DEBUG) ReciprocalTerm.DeviceToHost();
  hipfftExecC2C(planAll_Cuda,(hipfftComplex*)ReciprocalTerm.Arr_dev,(hipfftComplex*)ReciprocalTerm.Arr_dev,HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  if (DEBUG) ReciprocalTerm.DeviceToHost();
  ElasticForce = ReciprocalTerm* (*Eta);
  hipDeviceSynchronize();
  if (DEBUG) ElasticForce.DeviceToHost();

  return 0;
}

int Dynamics_mart::DislocationForceInit(){
  SetCalPos(Data_HOST);
  for (int saq=0;saq<VariantN;saq++){
    for (int i=0;i<nx;i++)
      for (int j=0;j<ny;j++)
        for (int k=0;k<nz;k++){
          DislocationForceConst(saq,i,j,k)=0;
          for (int sa=0;sa<3;sa++)
            for (int sap=0;sap<3;sap++){
              DislocationForceConst(saq,i,j,k)=DislocationForceConst(saq,i,j,k)+25.0f*DislocationStress(sa,sap,i,j,k)*(*StrainTensor)(saq,sa,sap);
            }
        }
  }
  DislocationForceConst.HostToDevice();
  SetCalPos(Data_DEV);
  return 0;
}

int Dynamics_mart::DislocationForceCalculate(){
  SetCalPos(Data_DEV);
  DislocationForce=DislocationForceConst*(*Eta);
  return 0;
}

__global__ void Block_Mart_Kernel(Real *Eta_arr, Real *Defect_arr){
  int pn=
	blockIdx.y*gridDim.z*blockDim.x +blockIdx.z*blockDim.x +threadIdx.x;
  int pvn= blockIdx.x *gridDim.y *gridDim.z * blockDim.x +pn;
  Eta_arr[pvn]=Eta_arr[pvn]*(1.0f-Defect_arr[pn]);
}

int Dynamics_mart::Block(){
  dim3 bn(VariantN,nx,ny);
  dim3 tn(nz);
  Block_Mart_Kernel<<<bn,tn>>>(Eta->Arr_dev,Defect->Arr_dev);

  return 0;
}

int Dynamics_mart::Calculate(){
  string ss;
  Vars["temperature"]>>=Temperature; 
  GradientForceCalculate();
  ChemicalForceCalculate();
  ElasticForceCalculate();
  DislocationForceCalculate();
  ////////////////////////////
  Eta_RT=0.f;
  if (weightGradient>0) Eta_RT += weightGradient*GradientForce; hipDeviceSynchronize();
  if (weightChemical>0) Eta_RT += (0-weightChemical)*ChemicalForce; hipDeviceSynchronize();
  if (weightElastic>0) Eta_RT  += (0-weightElastic)*ElasticForce; hipDeviceSynchronize();
  if (weightDislocation>0) Eta_RT += (0-weightDislocation)*DislocationForce; hipDeviceSynchronize();
  if (weightExternal>0) Eta_RT += (0-weightExternal)*ExternalForce; hipDeviceSynchronize();
  if (weightNoise>0){
	/*/dim3 bn(VariantN,Dimension[1],Dimension[2]); dim3 tn(Dimension[3]);
	  fnoise<<<bn,tn>>>(Noise.Arr_dev);// */
	Noise.NewNormal_device();
	Eta_RT += weightNoise*0.0001* Noise;
  }
  (*Eta) += DeltaTime* Eta_RT;
  //defect block
  Block();

  ///////////////////////////////
  return 0;
}

int Dynamics_mart::RunFunc(string funcName){ return 0; }

int Dynamics_mart::Fix(real progress){
  string ss,mode;
  ss = Vars["fix"];
  do{
	ss>>mode;
	if      (mode=="temperature"	){
	  real st,et; //start and end temperature
	  ss>>st>>et;
	  (Vars["temperature"])<<=(st+ progress*(et- st));
	} else if (mode=="pressure"		){ 
	} else{
	  GV<0>::LogAndError>>"Error: fix style ">>mode>>" does not find!\n";
	}
  } while ( ss != "");

  return 0;
}

string Dynamics_mart::Get(string ss){ // return the statistic info.
  string ans="";
  string var; ss>>var;
  if (var == "temperature") return ans<<Temperature; 
  else return "nan";
}

