
#include<hiprand.h>
#include"../include/datamain.th"
#include"random.h"

using namespace GS_NS;
using namespace DATA_NS;

<<<<<<< HEAD
int Random::InitRandom(int *dimArr, real mean, real variance,int seed_host,int seed_dev){
  hiprandCreateGenerator(&Gen_dev,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandCreateGeneratorHost(&Gen_host,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(Gen_dev,seed_dev);
  hiprandSetPseudoRandomGeneratorSeed(Gen_host,seed_host);
  Init(dimArr,Data_HOST_DEV);
  Mean=mean;
  Variance=variance;
  Seed_dev=seed_dev;
  Seed_host=seed_host;
=======
int Random::InitRandom(int *dimArr, real mean, real variance,real Seed_host,real Seed_dev){
  hiprandCreateGenerator(&Gen_dev,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandCreateGeneratorHost(&Gen_host,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(Gen_dev,Seed_dev);
  hiprandSetPseudoRandomGeneratorSeed(Gen_host,Seed_host);
  Init(dimArr,Data_HOST_DEV);
  Mean=mean;
  Variance=variance;
>>>>>>> origin/master
  return 0;
}

Random::Random(){
<<<<<<< HEAD
  Mean=0.0f;
  Variance=1.0f;
  Seed_dev=0;
  Seed_host=0;
=======
  Mean=0.0;
  Variance=1.0;
  Seed_dev=0.0;
  Seed_host=0.0;
>>>>>>> origin/master
  Gen_host = NULL;
  Gen_dev  = NULL;
}
Random::~Random(){
  if (Gen_dev) hiprandDestroyGenerator(Gen_dev);
  if (Gen_host) hiprandDestroyGenerator(Gen_host);
}
int Random::InitRandom(int n, ...){
  va_list args;
  va_start(args,n);
  int *arr	=	new int [ n+ 2 ];
  arr[0]=n;
  for (int i=1; i<=n; i++)
	arr[i]	=	va_arg(args,int);
  va_end(args);
  InitRandom(arr,Mean,Variance,Seed_host,Seed_dev);
  delete[]arr;
  return 0;
}
<<<<<<< HEAD
int Random::SetParas(real mean,real variance,int seed_host,int seed_dev){
=======
int Random::SetParas(real mean,real variance,real seed_host,real seed_dev){
>>>>>>> origin/master
  Mean=mean;
  Variance=variance;
  Seed_host=seed_host;
  Seed_dev=seed_dev;
  hiprandSetPseudoRandomGeneratorSeed(Gen_dev,Seed_dev);
  hiprandSetPseudoRandomGeneratorSeed(Gen_host,Seed_host);
  return 0;
}

  
Random &Random::NewNormal_device(){
  hiprandGenerateNormal(Gen_dev,(float*)Arr_dev,N(),Mean,Variance);//??
  return *this;
}
Random &Random::NewNormal_host(){
  hiprandGenerateNormal(Gen_host,(float*)Arr,N(),Mean,Variance);
  return *this;
}

