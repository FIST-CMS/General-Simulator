
#include<hiprand.h>
#include"../include/datamain.th"
#include"random.h"

using namespace GS_NS;
using namespace DATA_NS;

int Random::InitRandom(int *dimArr, real mean, real variance,real Seed_host,real Seed_dev){
  hiprandCreateGenerator(&Gen_dev,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandCreateGeneratorHost(&Gen_host,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(Gen_dev,Seed_dev);
  hiprandSetPseudoRandomGeneratorSeed(Gen_host,Seed_host);
  Init(dimArr,Data_HOST_DEV);
  Mean=mean;
  Variance=variance;
  return 0;
}

Random::Random(){
  Mean=0.0;
  Variance=1.0;
  Seed_dev=0.0;
  Seed_host=0.0;
  Gen_host = NULL;
  Gen_dev  = NULL;
}
Random::~Random(){
  if (Gen_dev) hiprandDestroyGenerator(Gen_dev);
  if (Gen_host) hiprandDestroyGenerator(Gen_host);
}
int Random::InitRandom(int n, ...){
  va_list args;
  va_start(args,n);
  int *arr	=	new int [ n+ 2 ];
  arr[0]=n;
  for (int i=1; i<=n; i++)
	arr[i]	=	va_arg(args,int);
  va_end(args);
  InitRandom(arr,Mean,Variance,Seed_host,Seed_dev);
  delete[]arr;
  return 0;
}
int Random::SetParas(real mean,real variance,real seed_host,real seed_dev){
  Mean=mean;
  Variance=variance;
  Seed_host=seed_host;
  Seed_dev=seed_dev;
  hiprandSetPseudoRandomGeneratorSeed(Gen_dev,Seed_dev);
  hiprandSetPseudoRandomGeneratorSeed(Gen_host,Seed_host);
  return 0;
}

  
Random &Random::NewNormal_device(){
  hiprandGenerateNormal(Gen_dev,(float*)Arr_dev,N(),Mean,Variance);//??
  return *this;
}
Random &Random::NewNormal_host(){
  hiprandGenerateNormal(Gen_host,(float*)Arr,N(),Mean,Variance);
  return *this;
}

