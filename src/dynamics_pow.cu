#include "hip/hip_runtime.h"

////////////////////////////////////////
#include"pub.h"
#include"dynamics.h"
////////////////////////////////////////
//#include"your_own_library.h"
///////////////////////////////////////
#include"dynamics_pow.h"

using namespace GS_NS;
using namespace DATA_NS;

Dynamics_pow::Dynamics_pow(){}




int Dynamics_pow::Initialize(){
  x=2.0f; Vars["x"]>>=x;// x default set to 2.0
  Matrix = &((*Datas)["matrix"]);
  return 0;
}





int Dynamics_pow::Calculate(){
  (*Matrix)=(*Matrix)*x;
  return 0;
}




int Dynamics_pow::RunFunc(string func){ return Code_COMMAND_UNKNOW;}

int Dynamics_pow::Fix(real progress){return 0;}



string Dynamics_pow::Get(string var){
  if (var=="x") return ToString(x);
  if (var=="sum_of_matrix")
	return ToString(Matrix->TotalHost());
  return "nan";
}





Dynamics_pow::~Dynamics_pow(){}
