
#define DEBUG 0

#include"pub.h"
#include"pub_main.h"

using namespace GUPS_NS;
using namespace DATA_NS;


GUPS::GUPS(){ 
  for (int i=0;i<DynaMax;i++){
	Dyna[i] = NULL;
	IsDynaInit[i] = false;
  }
}


GUPS::~GUPS(){
  for (int i=0;i<DynaMax;i++)
	if (Dyna[i]!=NULL) delete Dyna[i];
}

int GUPS::ReadHere(string name, string &arrays){
  int n; arrays>>n;
  int *dim = new int[n+1]; dim[0]=n;
  for (int i=1; i<=n; i++) arrays>>dim[i];
  Datas[name].Init(dim,Data_HOST);
  for (int i=0; i<Datas[name].N(); i++)
	arrays>>Datas[name].Arr[i];
  return 0;
}

int GUPS::SetInfo(string ss){ss>>InfoSteps; InfoMode  = ss; return 0;}

int GUPS::InfoOut(){
  string sm,ss; sm=InfoMode;
  GV<0>::LogAndError<<"Step  "<<CurrentStep<<" \t";
  while (sm !=""){
	sm>>ss;
	GV<0>::LogAndError<<Dyna[DynaID]->Get(ss)<<" \t";
  }
  GV<0>::LogAndError<<"\n";
  return 0;
}

int GUPS::SetDump(string ss){
  ss>>DumpFolder>>DumpSteps;
  string cmd="if [ -s "+DumpFolder+" ]; then echo \'dump to folder"+DumpFolder+"\'; else mkdir "+DumpFolder+"; fi";
  if (system(cmd.c_str()));
  DumpMode	 = ss;
  return 0;
}

template<class type> int GUPS::DumpFile(string str,Data<type> &data){
  data.DeviceToHost();
  string file; file<<DumpFolder<<"/"<<str<<CurrentStep<<".data";
  if (CurrentStep==TotalSteps) {file="";file<<DumpFolder<<"/"<<str<<".final.data";}
  ofstream of(file.c_str());
  hipDeviceSynchronize();
  of<<data;
  of.close();
  return 0;
}
  
int GUPS::DumpOut(){
  string sm,ss;
  sm=DumpMode;
  //if (DumpMode==""){//dedaut mode
  //DumpFile("eta",Datas["eta"]);
  //}else
  while (sm !=""){
	sm>>ss;
	if (Datas(ss) != NULL)
	  DumpFile(ss,Datas[ss]);
	else GV<0>::LogAndError>>"Error: data ">>ss>>" does not exist!\n";
  }
  return 0;
}

int GUPS::SetSys(string ss){
  string sys;
  int id=0;
  ss>>sys>>id;
  DynaID=id;
  if ( id >=DynaMax || id<0 ){
	DynaID=(id%DynaMax+DynaMax)%DynaMax;
	GV<0>::LogAndError<<"Dyna ID is out of the allowed index range. It is moduled to "<<DynaID<<"\n";
	return -1;
  }
  if (Dyna[DynaID]!=NULL) // if the DynaID dyna has been created then delete it.
	delete Dyna[id];

  bool hassys=false;
  ///////////////////////////////////
  //GUPS_SYS_DEFINE_START
  if (sys == "cores"	) { Dyna[DynaID] = new DynamicsCores; hassys=true; }
  if (sys == "diffuse"	) { Dyna[DynaID] = new DynamicsDiffuse; hassys=true; }
  if (sys == "mart"	) { Dyna[DynaID] = new DynamicsMart; hassys=true; }
  if (sys == "stress"	) { Dyna[DynaID] = new DynamicsStress; hassys=true; }
  if (sys == "xxx"	) { Dyna[DynaID] = new DynamicsXxx; hassys=true; }
  //GUPS_SYS_DEFINE_END
  ///////////////////////////////////
  if (!hassys) {
	GV<0>::LogAndError>>"System ">>ss>>"is not recognized\n";
	return -1;
  }
  return 0;
}

//////////////////////////////////////////////////
int GUPS::Set(string ss){
  string var;
  ss>>var;
  Vars[var] = ss;
  return 0;
}

int GUPS::Link(string ss){
  string target,link;
  ss>>target>>link;
  Vars.Link(target,link);
  return 0;
}

int GUPS::Read(string ss){
  string file,var; ss>>var>>file;
  ifstream ifs;
  ifs.open(file.c_str());
  if (ifs){
	ifs>>Datas[var];
	ifs.close();
  }else{
	GV<0>::LogAndError>>"File ">>file>>" not found!\n";
	return -1;
  }
  return 0;
}

int GUPS::Run(string ss){
  int totalsteps=1;
  ss>>totalsteps; TotalSteps=totalsteps; if (totalsteps <= 0 ) return -1;
  //what dyna to use???? defined in sys
  // the data is passed by reference ( big )
  // paras is passed by value ( small )
  Dyna[DynaID]->Datas = &Datas; 
  Dyna[DynaID]->Vars= Vars; 
  if ( !IsDynaInit[DynaID] ){
	Dyna[DynaID]->Initialize();
	IsDynaInit[ DynaID ] = true;
  }
  //////////////////////////////////////////////////////////////
  int  infoInterval;
  if ( InfoSteps==0) infoInterval=totalsteps+1;else infoInterval = totalsteps/InfoSteps;
  if ( infoInterval==0 ) infoInterval=1;
  int  dumpInterval;
  if ( DumpSteps== 0 ) dumpInterval= totalsteps+1; else dumpInterval  = totalsteps/DumpSteps;
  if ( dumpInterval == 0 ) dumpInterval =1;
  //////////////////////////////////////////////////////////////
  string mode=InfoMode,tempss;
  GV<0>::LogAndError<<"Info style: \t"; while (mode!=""){ mode>>tempss; GV<0>::LogAndError<<tempss<<"\t"; };
  GV<0>::LogAndError<<"\n";
  for (int i=1;i<=totalsteps;i++) {
	CurrentStep = i; // this will be used in dump and info to identity the progress
	Dyna[DynaID]->Fix(real(i)/totalsteps);
	Dyna[DynaID]->Calculate();

	if (i % infoInterval==0|| i==totalsteps) InfoOut();
	if (i % dumpInterval  ==0|| i==totalsteps) DumpOut();
  }
  //////////////////////////////////////////////////////////////
  return 0;
}

int GUPS::RunFunc(string funcName){
  Dyna[DynaID]->Datas = &Datas; 
  Dyna[DynaID]->Vars= Vars; 
  if ( !IsDynaInit[DynaID] ){
	GV<0>::LogAndError<<"runfunc command run before dynamics being initialized, run initialize function first\n";
	Dyna[DynaID]->Initialize();
	IsDynaInit[ DynaID ] = true;
	GV<0>::LogAndError<<"Initialize function called\n";
  }
  Dyna[DynaID]->RunFunc(funcName);
  return 0;
}
