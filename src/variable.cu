#include "hip/hip_runtime.h"

#include"pub.h"
#include"../include/datamain.th"
#include"variable.h"
using namespace DATA_NS;
using namespace GS_NS;
using namespace std;

Variable::Variable(){
  Vars = new Map< string >;
};

Variable::~Variable(){};

bool _is_number(string ss){
  int pos;
  static const int s_N=9;
  static const string special_case[s_N]={"E+","E-","e+","e-","E","e","+","-","."};
  for (int i=0; i<s_N; i++){
	while ( (pos=ss.find(special_case[i]))>=0 )
	  ss.erase( pos, special_case[i].length() );
  }
  for (int i=0;i<ss.length(); i++)
	if (ss[i]<'0'||ss[i]>'9') return false;
  return true;
}
  
bool _not_operator_for_sure(string &expr, int &pos){
  if (pos>0 && (expr[pos]=='+'||expr[pos]=='-')&&(expr[pos-1]=='e'||expr[pos-1]=='E'))
	return true;
  return false;
}

bool _is_var_char(char ch){
  if (ch>='a'&&ch<='z') return true;
  if (ch>='A'&&ch<='Z') return true;
  if (ch>='0'&&ch<='9') return true;
  if (ch=='_') return true;
  return false;
};

<<<<<<< HEAD
int Variable::shell(string ss){ 
  int err=0; string temp; 
  err=shell(ss,temp); 
  GV<0>::LogAndError<<temp<<"\n";
  return err;
}
int Variable::shell(string ss,string &result){
  string st; st =ss+" 2>&1 >.gs.shell.tmp"; 
  if (system(st.c_str()));
  ifstream in(".gs.shell.tmp",ios::in);
  if (in.fail()){
    GV<0>::LogAndError<<"Error: fail to read shell results\n";
    return 0;
  }
  istreambuf_iterator<char> beg(in), end;
  result=string(beg,end);
  in.close();
  if (result.length()>0) result.erase(result.length()-1,1);
  return 0;
};

int Variable::ReplaceExpr(string &ss){//ignore the first startP words
  int err=0;
  int p_left=-1,p_right=-1,pl;
  while( (p_left=ss.find("`"))>=0){
    p_right=ss.find("`",p_left+1);
    if ( p_right<0 ){
	 GV<0>::LogAndError<<"Error: uncomplete expression\n";
	 return Code_ERR;
    }
    ////////////////////////////////////
    string temp_str,result_str;
    temp_str=ss.substr(p_left+1,p_right-p_left-1);
    err=shell(temp_str,result_str); if(err<0)return err;
    ss.replace(p_left,p_right - p_left+1, result_str);
  }
  p_left=-1;
  while( (p_left=ss.find("{"))>=0){
    p_right=-1;
	do {
=======
int Variable::ReplaceExpr(string &ss){//ignore the first startP words
  int err=0;
  int p_left=-1,p_right=-1,pl;
  while( (p_left=ss.find("{"))>=0){
	p_right=ss.find("}");
	if (p_right<0) return 0; //no substitution
	do {
	  pl = ss.find("{",p_left+1);
	  if (pl>p_right||pl<0){
		break;
	  }
>>>>>>> origin/master
	  p_right=ss.find("}",p_right+1);
	  if ( p_right<0 ){
		GV<0>::LogAndError<<"Error: uncomplete expression\n";
		return Code_ERR;
	  }
<<<<<<< HEAD
	  pl = ss.find("{",p_left+1);
	  if (pl>p_right||pl<0){
		break;
	  }
=======
>>>>>>> origin/master
	}while (1);
	////////////////////////////////////
	string temp_str,result_str;
	temp_str=ss.substr(p_left+1,p_right-p_left-1);
	ExprTree tree;
	err=tree.Init(temp_str,Vars,Vars_gs); if(err<0)return err;
	result_str = tree.Expr;
	ss.replace(p_left,p_right - p_left+1, result_str);
  }
  return 0;
}

<<<<<<< HEAD
=======

>>>>>>> origin/master
int Variable::Evaluate(string &ss){//calculate an expression
  int pos;
  while ((pos=ss.find('{'))>=0) ss.replace(pos,1,"(");
  while ((pos=ss.find('}'))>=0) ss.replace(pos,1,")");
  ExprTree tree;
  tree.Init(ss,Vars,Vars_gs);
  ss = tree.Expr;
  return 0;
}

<<<<<<< HEAD
=======
int Variable::Set(string ss){
  string var,val;
  //////////////////////////////
  ss>>var>>val;
  (*Vars)[var]= val;
  //////////////////////////////
  return 0;
}

>>>>>>> origin/master
ExprTree::ExprTree(){
  Left=NULL;
  Right=NULL;
};

ExprTree::~ExprTree(){
  if (Left!=NULL ){ delete Left;  }
  if (Right!=NULL){ delete Right; }
}

int ExprTree::Init(string expr,Map<string> *vars,Map<string> *vars_gs){
  Left=NULL; Right=NULL; Val=0; Expr="";
  if (expr=="") return 0;
  int count=0,len=expr.length();
  ////////////////////delete extra chars
  {int p; while ((p=expr.find(" "))>=0) expr.erase(p,1);}
  while (expr[0]=='('&&expr[len-1]==')') {expr=expr.substr(1,len-2); len=expr.length();}//?????
  ////////////////////seperate the expresion
  int expr_position=-1,lev=9999,operator_position=-1;
  for (int p=0;p<len;p++){
	if (expr[p]=='(') count++;
	if (expr[p]==')') count--;
	if (count!=0) continue;
	// some specific form can not be included e- e+
	if ( _not_operator_for_sure(expr,p) ) continue;
	  ////////////////////////////////////////
	for ( int i=0; i<Operator_N; i++)
	  if ( (expr.substr(p,Operators[i].length()) == Operators[i]) && (lev>=Operator_Levels[i])){
		expr_position=p;
		operator_position=i;
		lev=Operator_Levels[i];
	  }
	///////////////////////////////////////
  }
  //////////////////serperate and sum up
  if (expr_position<0) {
	string temp_s; expr>>=temp_s;
	/////////////
	if ( vars->exist(temp_s) ){
<<<<<<< HEAD
	  (*vars)[temp_s]>>=Expr;
	  Expr>>=Val;
=======
	  (*vars)[temp_s]>>=Val;
	  Expr<<Val;
>>>>>>> origin/master
	  return 0;
	}
	if (temp_s[0]=='$') temp_s.erase(0,1);
	if ( vars_gs->exist(temp_s) ){
<<<<<<< HEAD
	  (*vars_gs)[temp_s]>>=Expr;
	  Expr>>=Val;
=======
	  (*vars_gs)[temp_s]>>=Val;
	  Expr<<Val;
>>>>>>> origin/master
	  return 0;
	}
	////////////////
	if (_is_number(temp_s)){
<<<<<<< HEAD
	  Expr=temp_s;
	  temp_s>>=Val;
=======
	  temp_s>>=Val;
	  Expr<<Val;
>>>>>>> origin/master
	  return 0;
	}else{
	  GV<0>::LogAndError<<"Error: \""<<temp_s<<"\" unknown \n";
	  Val=_NAN_Var;
	  Expr="nan";
	  return Code_ERR;
	}
  }
  else{
	string l_expr, r_expr;
	Opera= Operators[operator_position];
	l_expr=expr.substr(0,expr_position);
	r_expr=expr.substr(expr_position + Opera.length() ,len-expr_position);

	Left = new ExprTree;
	Right= new ExprTree;
	Right->Init(r_expr,vars,vars_gs);
	if (Opera != "=") Left->Init(l_expr,vars,vars_gs);

	if (Left->Expr=="nan" || Right->Expr=="nan"){
	  Expr="nan";
	  Val=_NAN_Var;
	  return 0;
	}
	if      (Opera=="+") { Val= Left->Val + Right->Val; Expr=ToString(Val); }
	else if (Opera=="-") { Val= Left->Val - Right->Val; io(Val,Expr);}
	else if (Opera=="*") { Val= Left->Val * Right->Val; io(Val,Expr);}
	else if (Opera=="/") { if (Right->Val==0) {Expr="nan"; Val=_NAN_Var;} else{ Val= Left->Val/Right->Val;io(Val,Expr);} }
<<<<<<< HEAD
	else if (Opera=="^") { Val= (Real(Left->Val) ^ ((int)Right->Val)).Re; io(Val,Expr); }
=======
	else if (Opera=="^") { Val= (Real(Left->Val) ^ Right->Val).Re; io(Val,Expr); }
>>>>>>> origin/master
	else if (Opera=="<="){ if (Left->Val<=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">="){ if (Left->Val>=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="=="){ if (Left->Val==Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="!="){ if (Left->Val!=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="<") { if (Left->Val <Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">") { if (Left->Val >Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="=") {
	  if (l_expr[0]!='$') (*vars)[l_expr]=Right->Expr; else (*vars_gs)[l_expr.substr(1)]=Right->Expr;
	  Expr=Right->Expr; Val=Right->Val;
	}
	else if (Opera=="+="){ExprTree treenew;treenew.Init(l_expr+"="+l_expr+"+"+r_expr,vars,vars_gs);Val=treenew.Val;Expr=treenew.Expr;}
	else if (Opera=="-="){ExprTree treenew;treenew.Init(l_expr+"="+l_expr+"-"+r_expr,vars,vars_gs);Val=treenew.Val;Expr=treenew.Expr;}
	else if (Opera=="*="){ExprTree treenew;treenew.Init(l_expr+"="+l_expr+"*"+r_expr,vars,vars_gs);Val=treenew.Val;Expr=treenew.Expr;}
	else if (Opera=="/="){ExprTree treenew;treenew.Init(l_expr+"="+l_expr+"/"+r_expr,vars,vars_gs);Val=treenew.Val;Expr=treenew.Expr;}
  }
  return 0;
}
