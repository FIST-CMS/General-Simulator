#include "hip/hip_runtime.h"

#include"pub.h"
#include"../include/datamain.th"
#include"variable.h"
using namespace DATA_NS;
using namespace GS_NS;
using namespace std;

Variable::Variable(){};

Variable::~Variable(){};

int Variable::Calculate(string expr, string &val){
  CalTree ct;
  ct.Init(expr);
  val= ct.Expr;
  return 0;
};

bool _not_operator(string &expr, int &pos){
  if (pos>0 && (expr[pos]=='+'||expr[pos]=='-')&&(expr[pos-1]=='e'||expr[pos-1]=='E'))
	return true;
  return false;
}

int _postion_of_operator(string expr,string opera,int pos){
  int npos;
  npos=expr.find(opera,pos);
  while ( npos>=0 && _not_operator(expr,npos) )
	npos=expr.find(opera,npos+1);
  return npos;
}

int Variable::SubVar(string &ss,int startp){//ignore the first startP words
  string st;
  map<string,string>::iterator p;
  //first replace exist variables
  p=Variables.begin();
  while (p!=Variables.end()){
	int l=p->first.length();
	int po=-1;
	string pf=p->first,ps=p->second;
	for (int i=1;i<=startp;i++)//find a proper start index in order to ignore startp words
	  po=ss.find(" ",po+1);
	po = ss.find(pf,po+1);
	while (po>=0){
	  if ( (po>0) && ( (ss[po-1]>='a'&&ss[po-1]<='z') || (ss[po-1]>='A'&&ss[po-1]<='Z') ||( ss[po+l]>='0' && ss[po+l]<='9'))) { po = ss.find(pf,po+1); continue;}
	  if ( (po+l< ss.length() )&& ( (ss[po+l]>='a'&&ss[po+l]<='z')||(ss[po+l]>='a'&&ss[po+l]<='z') ||( ss[po+l]>='0' && ss[po+l]<='9')) ) { po = ss.find(pf,po+1); continue; }
	  ss.replace(po,l,ps);
	  po = ss.find(pf,po+ps.length());
	}
	p++;
  }

  //if + - * / ^ exists then do a math evaluation.
  for (int i=0;i<Operator_N;i++){
	string opera = Operators[i];
	string ssub;
	int po=-1,ph,pe;
	for (int i=1;i<=startp;i++)//find start index to ignore startp words; changes when under operates
	  po=ss.find(" ",po+1);
	po=_postion_of_operator(ss,opera,po+1);
	while (po>=0){
	  ph=po-1; while (ph>=0&&ss[ph]!=' ') ph--;
	  pe=po+1; while (pe<=ss.length()&&ss[pe]!=' ') pe++;
	  ssub=ss.substr(ph+1,pe-ph-1);
	  Calculate(ssub,st);
	  ss.replace(ph+1,pe-ph-1,st);
	  po=_postion_of_operator(ss,opera,ph+2);
	}
  }
  return 0;
}



int Variable::Set(string ss){
  string var,expr;
  ss>>var>>expr;
  SubVar(expr,0); 
  Variables[var]= expr;
  return 0;
}

string Variable::operator()(string id){// return value or ""
  map<string,string>::iterator viter;
  viter=Variables.find(id);
  if ( viter == Variables.end() )
	return "";
  else return viter->second;
}
string &Variable::operator[](string id){//create new if necc.
  return Variables[id];
}


CalTree::CalTree(){};

CalTree::~CalTree(){
  if (Left!=NULL ){ delete Left;  }
  if (Right!=NULL){ delete Right; }
}

int CalTree::Init(string expr){
  Left=NULL; Right=NULL; Val=0;
  if (expr=="") return 0;
  int count=0,len=expr.length();
  ////////////////////delete extra chars
  {int p; while ((p=expr.find(" ")) >=0) expr.erase(p,1);}
  while (expr[0]=='('&&expr[len-1]==')') {expr=expr.substr(1,len-2); len=expr.length();}//?????
  ////////////////////seperate the expresion
  int expr_position=-1,lev=9999,operator_position=-1;
  for (int p=0;p<len;p++){
	if (expr[p]=='(') count++;
	if (expr[p]==')') count--;
	if (count!=0) continue;
	// some specific form can not be included e- e+
	if ( _not_operator(expr,p) ) continue;
	  ////////////////////////////////////////
	for ( int i=0; i<Operator_N; i++)
	  if ( (expr.substr(p,Operators[i].length()) == Operators[i]) && (lev>=Operator_Levels[i])){
		expr_position=p;
		operator_position=i;
		lev=Operator_Levels[i];
	  }
	///////////////////////////////////////
  }
  //////////////////serperate and sum up
  if (expr_position<0) {
	Expr=expr;
	expr>>=Val;
	/*
	int len=expr.length();
	for (int i=0;i<len; i++)
	  if (expr[i]>'9' || expr[i]<'0'){
		GV<0>::LogAndError<<"Error: expression "<<expr<<" unknown\n";
		Expr="nan";
		Val=-9.9999999e+23;
		return 0;
	  }
	*/
  }
  else{
	Opera= Operators[operator_position];
	Left = new CalTree;
	Left->Init(expr.substr(0,expr_position));
	Right= new CalTree;
	Right->Init(expr.substr( expr_position + Opera.length() ,len-expr_position));
	if (Left->Expr=="nan" || Right->Expr=="nan"){
	  Expr="nan";
	  Val=-9.9999999e+23;
	  return 0;
	}
	if      (Opera=="+") { Val= Left->Val + Right->Val; Expr=ToString(Val); }
	else if (Opera=="-") { Val= Left->Val - Right->Val; io(Val,Expr);}
	else if (Opera=="*") { Val= Left->Val * Right->Val; io(Val,Expr);}
	else if (Opera=="/") { if (Right->Val==0) {Expr="nan"; Val=-9.9999e+23;}else{ Val= Left->Val/Right->Val;io(Val,Expr);} }
	else if (Opera=="^") { Val= (Real(Left->Val) ^ Right->Val).Re; io(Val,Expr); }
	else if (Opera=="<") { if (Left->Val <Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">") { if (Left->Val >Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="<="){ if (Left->Val<=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">="){ if (Left->Val>=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="=="){ if (Left->Val==Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="!="){ if (Left->Val!=Right->Val) Val=1; else Val=0; io(Val,Expr);}
  }
  return 0;
}

