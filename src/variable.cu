#include "hip/hip_runtime.h"

#include"pub.h"
#include"../include/datamain.th"
#include"variable.h"
using namespace DATA_NS;
using namespace GS_NS;
using namespace std;

Variable::Variable(){
  Vars = new Map< string >;
  Datas = new Map< Data<Real> >;
};

Variable::~Variable(){};


bool _is_number(string ss){
  int pos;
  static const int s_N=9;
  static const string special_case[s_N]={"E+","E-","e+","e-","E","e","+","-","."};
  for (int i=0; i<s_N; i++){
	while ( (pos=ss.find(special_case[i]))>=0 )
	  ss.erase( pos, special_case[i].length() );
  }
  for (int i=0;i<ss.length(); i++)
	if (ss[i]<'0'||ss[i]>'9') return false;
  return true;
}
  
bool _not_operator_for_sure(string &expr, int &pos){
  if (pos>0 && (expr[pos]=='+'||expr[pos]=='-')&&(expr[pos-1]=='e'||expr[pos-1]=='E'))
	return true;
  return false;
}

int _postion_of_operator(string expr,string opera,int pos){
  int npos;
  npos=expr.find(opera,pos);
  while ( npos>=0 && _not_operator_for_sure(expr,npos) )
	npos=expr.find(opera,npos+1);
  return npos;
}

bool _is_var_char(char ch){
  if (ch>='a'&&ch<='z') return true;
  if (ch>='A'&&ch<='Z') return true;
  if (ch>='0'&&ch<='9') return true;
  if (ch=='_') return true;
  return false;
};

int Variable::ReplaceExpr(string &ss){//ignore the first startP words
  int err=0;
  int p_left=-1,p_right=-1,pl;
  while( (p_left=ss.find("{"))>=0){
	p_right=ss.find("}");
	if (p_right<0) return 0; //no substitution
	do {
	  pl = ss.find("{",p_left+1);
	  if (pl>p_right||pl<0){
		break;
	  }
	  p_right=ss.find("}",p_right+1);
	  if ( p_right<0 ){
		GV<0>::LogAndError<<"Error: uncomplete expression\n";
		return Code_ERR;
	  }
	}while (1);
	////////////////////////////////////
	string temp_str,result_str;
	temp_str=ss.substr(p_left+1,p_right-p_left-1);
	ExprTree tree;
	err=tree.Init(temp_str,Vars); if(err<0)return err;
	result_str = tree.Expr;
	ss.replace(p_left,p_right - p_left+1, result_str);
  }
  return 0;
}


int Variable::Evaluate(string &ss){//calculate an expression
  int pos;
  while ((pos=ss.find('{'))>=0) ss.replace(pos,1,"(");
  while ((pos=ss.find('}'))>=0) ss.replace(pos,1,")");
  ExprTree tree;
  tree.Init(ss,Vars);
  ss = tree.Expr;
  return 0;
}

int Variable::Set(string ss){
  string var,val;
  //////////////////////////////
  ss>>var>>val;
  (*Vars)[var]= val;
  //////////////////////////////
  return 0;
}

ExprTree::ExprTree(){
  Left=NULL;
  Right=NULL;
};

ExprTree::~ExprTree(){
  if (Left!=NULL ){ delete Left;  }
  if (Right!=NULL){ delete Right; }
}

int ExprTree::Init(string expr,Map<string> *vars){
  Left=NULL; Right=NULL; Val=0; Expr="";
  if (expr=="") return 0;
  Vars=vars;
  int count=0,len=expr.length();
  ////////////////////delete extra chars
  {int p; while ((p=expr.find(" "))>=0) expr.erase(p,1);}
  while (expr[0]=='('&&expr[len-1]==')') {expr=expr.substr(1,len-2); len=expr.length();}//?????
  ////////////////////seperate the expresion
  int expr_position=-1,lev=9999,operator_position=-1;
  for (int p=0;p<len;p++){
	if (expr[p]=='(') count++;
	if (expr[p]==')') count--;
	if (count!=0) continue;
	// some specific form can not be included e- e+
	if ( _not_operator_for_sure(expr,p) ) continue;
	  ////////////////////////////////////////
	for ( int i=0; i<Operator_N; i++)
	  if ( (expr.substr(p,Operators[i].length()) == Operators[i]) && (lev>=Operator_Levels[i])){
		expr_position=p;
		operator_position=i;
		lev=Operator_Levels[i];
	  }
	///////////////////////////////////////
  }
  //////////////////serperate and sum up
  if (expr_position<0) {
	string temp_s; expr>>=temp_s;
	if ( Vars->exist(temp_s) ){
	  (*Vars)[temp_s]>>=Val;
	  Expr<<Val;
	}else{
	  if (_is_number(temp_s)){
		temp_s>>=Val;
		Expr<<Val;
	  }else{
		GV<0>::LogAndError<<"Error: \""<<temp_s<<"\" unknown \n";
		Val=_NAN_Var;
		Expr="nan";
		return Code_ERR;
	  }
	}
  }
  else{
	string l_expr, r_expr;
	Opera= Operators[operator_position];
	l_expr=expr.substr(0,expr_position);
	r_expr=expr.substr(expr_position + Opera.length() ,len-expr_position);

	Left = new ExprTree;
	Right= new ExprTree;
	Right->Init(r_expr,Vars);
	if (Opera != "=") Left->Init(l_expr,Vars);

	if (Left->Expr=="nan" || Right->Expr=="nan"){
	  Expr="nan";
	  Val=_NAN_Var;
	  return 0;
	}
	if      (Opera=="+") { Val= Left->Val + Right->Val; Expr=ToString(Val); }
	else if (Opera=="-") { Val= Left->Val - Right->Val; io(Val,Expr);}
	else if (Opera=="*") { Val= Left->Val * Right->Val; io(Val,Expr);}
	else if (Opera=="/") { if (Right->Val==0) {Expr="nan"; Val=_NAN_Var;} else{ Val= Left->Val/Right->Val;io(Val,Expr);} }
	else if (Opera=="^") { Val= (Real(Left->Val) ^ Right->Val).Re; io(Val,Expr); }
	else if (Opera=="<="){ if (Left->Val<=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">="){ if (Left->Val>=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="=="){ if (Left->Val==Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="!="){ if (Left->Val!=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="<") { if (Left->Val <Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">") { if (Left->Val >Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="=") { (*Vars)[l_expr]=Right->Expr; Expr=Right->Expr;}
	else if (Opera=="+=") { Val=Left->Val+Right->Val; io(Val,Expr); (*Vars)[l_expr]=Expr;}
	else if (Opera=="-=") { Val=Left->Val-Right->Val; io(Val,Expr); (*Vars)[l_expr]=Expr;}
	else if (Opera=="*=") { Val=Left->Val*Right->Val; io(Val,Expr); (*Vars)[l_expr]=Expr;}
	else if (Opera=="/=") { if (Right->Val==0) {Expr="nan"; Val=_NAN_Var;(*Vars)[l_expr]=Expr;} else{ Val= Left->Val/Right->Val;io(Val,Expr);(*Vars)[l_expr]=Expr;} }
  }
  return 0;
}

/*
int Variable::SubVar(string &ss){//ignore the first startP words
  return SubVar(ss,0,0);
}

int Variable::SubVar(string &ss,int startp){//ignore the first startP words
  return SubVar(ss,startp,0);
}

int Variable::SubVar(string &ss,int startp,int num){//ignore the first startP words
  int err=0;
  //////////////////standarlize;
  int pos; ss=ss+" "; while ((pos=ss.find("  "))>=0) ss.erase(pos,1);
  ///////////////////////////////////
  map<string,string>::iterator p;
  ////////
  p=Vars->begin();
  while (p!=Vars->end()){
	string v_name="{"+p->first+"}", v_value=(*Vars)[p];
	int pos = ss.find(v_name);
	while (pos>=0){
	  ss.replace(pos,v_name.length(),v_value);
	  pos = ss.find(v_name,pos + v_value.length());
	}
	p++;
  }
  /////////////////////////////////

  //if operator exists then do a math evaluation.
  string st;
  for (int i=0;i<Operator_N;i++){
	string opera = Operators[i];
	string ssub;
	int po=-1,ph,pe;
	po=_postion_of_operator(ss,opera,po+1);
	while (po>=0){
	  ph=po-1; while (ph>=0&&ss[ph]!=' ') ph--;
	  pe=po+1; while (pe<=ss.length()&&ss[pe]!=' ') pe++;
	  ssub=ss.substr(ph+1,pe-ph-1);
	  ///////////////////////
	  CalTree ct;
	  err =ct.Init(ssub); if (err<0) return err;
	  st = ct.Expr;
	  ////////////////////////
	  ss.replace(ph+1,pe-ph-1,st);
	  po=_postion_of_operator(ss,opera,ph+2);
	}
  }
  return 0;
}
*/

/*

CalTree::CalTree(){};

CalTree::~CalTree(){
  if (Left!=NULL ){ delete Left;  }
  if (Right!=NULL){ delete Right; }
}

int CalTree::Init(string expr){
  Left=NULL; Right=NULL; Val=0;
  if (expr=="") return 0;
  int count=0,len=expr.length();
  ////////////////////delete extra chars
  {int p; while ((p=expr.find(" ")) >=0) expr.erase(p,1);}
  while (expr[0]=='('&&expr[len-1]==')') {expr=expr.substr(1,len-2); len=expr.length();}//
  ////////////////////seperate the expresion
  int expr_position=-1,lev=9999,operator_position=-1;
  for (int p=0;p<len;p++){
	if (expr[p]=='(') count++;
	if (expr[p]==')') count--;
	if (count!=0) continue;
	// some specific form can not be included e- e+
	if ( _not_operator_for_sure(expr,p) ) continue;
	  ////////////////////////////////////////
	for ( int i=0; i<Operator_N; i++ )
	  if ( (expr.substr(p,Operators[i].length() ) == Operators[i]) && (lev>=Operator_Levels[i])){
		expr_position=p;
		operator_position=i;
		lev=Operator_Levels[i];
	  }
	///////////////////////////////////////
  }
  //////////////////serperate and sum up
  string temps;
  if (expr_position<0) {//final condition
	expr>>=Val;
	Expr<<Val;
  }else{
	Opera= Operators[operator_position];
	Left = new CalTree;
	Left->Init(expr.substr(0,expr_position));
	Right= new CalTree;
	Right->Init(expr.substr( expr_position + Opera.length() ,len-expr_position));
	if (Left->Expr=="nan" || Right->Expr=="nan"){
	  Expr="nan";
	  Val=-9.9999999e+23;
	  return 0;
	}
	if      (Opera=="+") { Val= Left->Val + Right->Val; Expr=ToString(Val); }
	else if (Opera=="-") { Val= Left->Val - Right->Val; io(Val,Expr);}
	else if (Opera=="*") { Val= Left->Val * Right->Val; io(Val,Expr);}
	else if (Opera=="/") { if (Right->Val==0) {Expr="nan"; Val=-9.9999e+23;}else{ Val= Left->Val/Right->Val;io(Val,Expr);} }
	else if (Opera=="^") { Val= (Real(Left->Val) ^ Right->Val).Re; io(Val,Expr); }
	else if (Opera=="<="){ if (Left->Val<=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">="){ if (Left->Val>=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="=="){ if (Left->Val==Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="!="){ if (Left->Val!=Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera=="<") { if (Left->Val <Right->Val) Val=1; else Val=0; io(Val,Expr);}
	else if (Opera==">") { if (Left->Val >Right->Val) Val=1; else Val=0; io(Val,Expr);}
  }
  return 0;
}
*/

/*
int Variable::Calculate(string expr, string &val){
  CalTree ct;
  ct.Init(expr);
  val= ct.Expr;
  return 0;
};
*/
