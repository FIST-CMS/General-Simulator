#include "hip/hip_runtime.h"

#define DEBUG 0
#include"../include/datamain.th"
#include"gtensorb.h"
using namespace GUPS_NS;
using namespace DATA_NS;


__host__ __device__ real GTensorB_VF(int x,int n,float dx){
  int tem; if (x<n/2) tem=x; else tem=x-n;
  return 2.0f*3.14f*dx/n*tem; 
}

__global__ void unitVector_gtensorb_kernel(Real *_gSquare, Real *unitVector,real dx, real dy, real dz){
  int x=blockIdx.x, y=blockIdx.y,z=threadIdx.x;
  int nx=gridDim.x, ny=gridDim.y,nz=blockDim.x;
  int tid=(x*ny+y)*nz+z; //tid =(x,y,z);
  int tidd=tid*3; // tidd+d = (x,y,z,d); d:0 1 2
  Real gx,gy,gz,mo;
  gx=GTensorB_VF(x,nx,dx);
  gy=GTensorB_VF(y,ny,dy);
  gz=GTensorB_VF(z,nz,dz);
  _gSquare[tid]=(gx^2)+(gy^2)+(gz^2);
  mo = sqrt(_gSquare[tid]);
  if (mo > 0.0000005){
	unitVector[tidd+0]=gx/mo;
	unitVector[tidd+1]=gy/mo;
	unitVector[tidd+2]=gz/mo;
  }else{
	unitVector[tidd+0]=0;
	unitVector[tidd+1]=0;
	unitVector[tidd+2]=0;
  }
}

int  GTensorB::UnitVector(real dx,real dy, real dz){
  int nx= unitVector.Dimension[1], ny= unitVector.Dimension[2],
	nz= unitVector.Dimension[3];
  dim3 bn(nx,ny),tn(nz);
  unitVector_gtensorb_kernel<<<bn,tn>>>
	(_gSquare.Arr_dev, unitVector.Arr_dev,
	 dx,dy,dz);
  return 0;
}

__global__ void gtensorb_kernel_calculate
(
 Real *B,
 Real*modulus,Real*sigma,Real*tensor,
 Real*unitVector,
 int VariantN1,int VariantN2,
 Real*iomega,Real*omega
 );

int GTensorB::InitB (int variantN1,int variantN2,
					int lx, int ly, int lz,
					real a1, real a2, real a3,
					 Data<Real> &tensor, Data<Real>&modulus ){

  Init(5,variantN1, variantN2, lx, ly, lz,Data_HOST_DEV);
  _gSquare.Init(3,lx,ly,lz,Data_HOST_DEV);
  unitVector.Init(4,lx,ly,lz,3,Data_HOST_DEV);
  ///////////////////////////////////////////////////////
  UnitVector(a1,a2,a3);
  _gSquare.DeviceToHost(); // ..
  unitVector.DeviceToHost(); // ..
  ////////////////////////////////////////////
  Data<Real> sigma;
  sigma.Init(3,variantN1+variantN2,3,3,Data_HOST_DEV);
  sigma=0.f;
  for (int sa=0; sa<variantN1+variantN2; sa++){
    for (int i=0;i<3;i++)
	 for (int j=0;j<3;j++){
	   for (int k=0;k<3;k++)
		for (int l=0;l<3;l++){
		  sigma(sa,i,j)+=modulus(i,j,k,l)*tensor(sa,k,l);
		}
	 }
  }
  sigma.HostToDevice();
  tensor.HostToDevice();
  modulus.HostToDevice();
  //-------------------------------------------------------------------
  ////////////////////////////////////////////////////////////////////
  Data<Real> iomega(5,lx,ly,lz,3,3,Data_DEV); // in order to free in time
  Data<Real> omega(5,lx,ly,lz,3,3,Data_DEV); //as temp variables
  dim3 bn(lx,ly,1), tn(lz,1,1);
  gtensorb_kernel_calculate<<<bn,tn>>>
	(
	 Arr_dev,
	 modulus.Arr_dev,sigma.Arr_dev,tensor.Arr_dev,
	 unitVector.Arr_dev,
	 variantN1,variantN2,
	 iomega.Arr_dev,omega.Arr_dev
	 );

  DeviceToHost();
  //DumpFile("data.structure.factor");
  return 0;
  
}

__global__ void gtensorb_kernel_calculate
(
 Real *B,
 Real *modulus,Real *sigma,Real *tensor,
 Real *unitVector,
 int variantN1,int variantN2,
 Real*iomega,Real*omega
 ){
  int nx= gridDim.x,  ny= gridDim.y,  nz=blockDim.x;
  int ix= blockIdx.x, iy= blockIdx.y, iz=threadIdx.x;
  /////////////////
  for (int i=0;i<3;i++)
	for (int j=0;j<3;j++){
	  iomega[(((ix*ny+iy)*nz+iz)*3+i)*3+j]=0.0f;
	  iomega[(((ix*ny+iy)*nz+iz)*3+i)*3+j]=0.0f;
	  for (int k=0;k<3;k++){
		for (int l=0;l<3;l++){
		  iomega[(((ix*ny+iy)*nz+iz)*3+i)*3+j] += 
			( modulus[((i*3+k)*3+l)*3+j]
			   * unitVector[((ix*ny+iy)*nz+iz)*3+k]
			   * unitVector[((ix*ny+iy)*nz+iz)*3+l]
			   );
		}
	  }
	}
  //////////////////////////////////////////////////////////////////////
  //gtensorb_Inverse3x3((float*)iomega,(float*)omega);
  Real det_m1; Real *m1,*m2; m1=iomega; m2=omega;
  det_m1=
	-m1[((ix*ny+iy)*nz+iz)*9+2]*m1[((ix*ny+iy)*nz+iz)*9+4]*m1[((ix*ny+iy)*nz+iz)*9+6]
	+ m1[((ix*ny+iy)*nz+iz)*9+1]*m1[((ix*ny+iy)*nz+iz)*9+5]*m1[((ix*ny+iy)*nz+iz)*9+6]
	+ m1[((ix*ny+iy)*nz+iz)*9+2]*m1[((ix*ny+iy)*nz+iz)*9+3]*m1[((ix*ny+iy)*nz+iz)*9+7]
	- m1[((ix*ny+iy)*nz+iz)*9+0]*m1[((ix*ny+iy)*nz+iz)*9+5]*m1[((ix*ny+iy)*nz+iz)*9+7]
	- m1[((ix*ny+iy)*nz+iz)*9+1]*m1[((ix*ny+iy)*nz+iz)*9+3]*m1[((ix*ny+iy)*nz+iz)*9+8]
	+ m1[((ix*ny+iy)*nz+iz)*9+0]*m1[((ix*ny+iy)*nz+iz)*9+4]*m1[((ix*ny+iy)*nz+iz)*9+8];
  if (det_m1>=0.00000001f||det_m1<=-0.00000001f){
	m2[((ix*ny+iy)*nz+iz)*9+0]=( -m1[((ix*ny+iy)*nz+iz)*9+5]*m1[((ix*ny+iy)*nz+iz)*9+7] + m1[((ix*ny+iy)*nz+iz)*9+4]*m1[((ix*ny+iy)*nz+iz)*9+8])/det_m1; 
	m2[((ix*ny+iy)*nz+iz)*9+1]=( m1[((ix*ny+iy)*nz+iz)*9+2]*m1[((ix*ny+iy)*nz+iz)*9+7] - m1[((ix*ny+iy)*nz+iz)*9+1]*m1[((ix*ny+iy)*nz+iz)*9+8])/det_m1;
	m2[((ix*ny+iy)*nz+iz)*9+2]=( -m1[((ix*ny+iy)*nz+iz)*9+2]*m1[((ix*ny+iy)*nz+iz)*9+4] + m1[((ix*ny+iy)*nz+iz)*9+1]*m1[((ix*ny+iy)*nz+iz)*9+5])/det_m1;
	m2[((ix*ny+iy)*nz+iz)*9+3]=( m1[((ix*ny+iy)*nz+iz)*9+5]*m1[((ix*ny+iy)*nz+iz)*9+6] - m1[((ix*ny+iy)*nz+iz)*9+3]*m1[((ix*ny+iy)*nz+iz)*9+8])/det_m1;
	m2[((ix*ny+iy)*nz+iz)*9+4]=( -m1[((ix*ny+iy)*nz+iz)*9+2]*m1[((ix*ny+iy)*nz+iz)*9+6] + m1[((ix*ny+iy)*nz+iz)*9+0]*m1[((ix*ny+iy)*nz+iz)*9+8])/det_m1; 
	m2[((ix*ny+iy)*nz+iz)*9+5]=( m1[((ix*ny+iy)*nz+iz)*9+2]*m1[((ix*ny+iy)*nz+iz)*9+3] - m1[((ix*ny+iy)*nz+iz)*9+0]*m1[((ix*ny+iy)*nz+iz)*9+5])/det_m1;
	m2[((ix*ny+iy)*nz+iz)*9+6]=( -m1[((ix*ny+iy)*nz+iz)*9+4]*m1[((ix*ny+iy)*nz+iz)*9+6] + m1[((ix*ny+iy)*nz+iz)*9+3]*m1[((ix*ny+iy)*nz+iz)*9+7])/det_m1; 
	m2[((ix*ny+iy)*nz+iz)*9+7]=( m1[((ix*ny+iy)*nz+iz)*9+1]*m1[((ix*ny+iy)*nz+iz)*9+6] - m1[((ix*ny+iy)*nz+iz)*9+0]*m1[((ix*ny+iy)*nz+iz)*9+7])/det_m1;
	m2[((ix*ny+iy)*nz+iz)*9+8]=( -m1[((ix*ny+iy)*nz+iz)*9+1]*m1[((ix*ny+iy)*nz+iz)*9+3] + m1[((ix*ny+iy)*nz+iz)*9+0]*m1[((ix*ny+iy)*nz+iz)*9+4])/det_m1;
  }else{
	for (int i=0; i<9; i++) m2[((ix*ny+iy)*nz+iz)*9+i]=0.f;
  }

  //////////////////////////////////////////////////////////////////////
  Real term=0;
  for (int sa=0; sa<variantN1; sa++){
	for (int sap=0; sap<variantN2; sap++){
	  B[((((sa*variantN2+sap)*nx+ix)*ny+iy)*nz+iz)]=0.0f;
	  for (int i=0;i<3;i++)
		for (int j=0;j<3;j++)
		  for (int k=0;k<3;k++)
			for (int l=0;l<3;l++){ 
			  term=
				-unitVector[((ix*ny+iy)*nz+iz)*3+i]
				*sigma[(sa*3+i)*3+j]
				*omega[(((ix*ny+iy)*nz+iz)*3+j)*3+k]
				*sigma[((sap+variantN1)*3+k)*3+l]
				*unitVector[((ix*ny+iy)*nz+iz)*3+l];
			  B[((((sa*variantN2+sap)*nx+ix)*ny+iy)*nz+iz)]+=
				modulus[((i*3+j)*3+k)*3+l]// an error
				*tensor[(sa*3+i)*3+j]
				*tensor[((sap+variantN1)*3+k)*3+l]
				+term;
			}
	}
  }
}

