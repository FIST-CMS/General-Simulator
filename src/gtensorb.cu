#include "hip/hip_runtime.h"

#define DEBUG 0
#include"../include/datamain.th"
#include"gtensorb.h"
using namespace GUPS_NS;
using namespace DATA_NS;


__host__ __device__ real GTensorB_VF(int x,int n,float dx){
  int tem; if (x<n/2) tem=x; else tem=x-n;
  return 2.0f*3.14f*dx/n*tem; 
}

__global__ void unitVector_gtensorb_kernel(Real *_g, Real *_gSquare, Real *unitVector,real dx, real dy, real dz){
  int x=blockIdx.x, y=blockIdx.y,z=threadIdx.x;
  int nx=gridDim.x, ny=gridDim.y,nz=blockDim.x;
  int tid=(x*ny+y)*nz+z; //tid =(x,y,z);
  int tidd=tid*3; // tidd+d = (x,y,z,d); d:0 1 2
  Real gx,gy,gz,mo;
  _g[tidd+0]=gx=GTensorB_VF(x,nx,dx);
  _g[tidd+1]=gy=GTensorB_VF(y,ny,dy);
  _g[tidd+2]=gz=GTensorB_VF(z,nz,dz);
  _gSquare[tid]=(gx^2)+(gy^2)+(gz^2);
  mo = sqrt(_gSquare[tid]);
  if (mo > 0.0000005){
	unitVector[tidd+0]=gx/mo;
	unitVector[tidd+1]=gy/mo;
	unitVector[tidd+2]=gz/mo;
  }else{
	unitVector[tidd+0]=0;
	unitVector[tidd+1]=0;
	unitVector[tidd+2]=0;
  }
}

int  GTensorB::UnitVector(real dx,real dy, real dz){  int nx= _g.Dimension[1], ny= _g.Dimension[2], nz= _g.Dimension[3];
  dim3 bn(nx,ny),tn(nz);
  unitVector_gtensorb_kernel<<<bn,tn>>>
	(_g.Arr_dev, _gSquare.Arr_dev, unitVector.Arr_dev,
	 dx,dy,dz);
  return 0;
}

int GTensorB::InitB (int variantN1,int variantN2,
					int lx, int ly, int lz,
					real a1, real a2, real a3,
					Data<Real> &tensor ){

  Init(5,variantN1, variantN2, lx, ly, lz,Data_HOST_DEV);
  _g.Init(4,lx,ly,lz,3,Data_HOST_DEV);
  _gSquare.Init(3,lx,ly,lz,Data_HOST_DEV);
  unitVector.Init(4,lx,ly,lz,3,Data_HOST_DEV);
  sigma.Init(3,variantN1+variantN2,3,3,Data_HOST_DEV);
  //unitvector
  UnitVector(a1,a2,a3);
  _g.DeviceToHost(); // will be use in this function
  _gSquare.DeviceToHost(); // ..
  unitVector.DeviceToHost(); // ..
  //stress  sigma(sa,j,i)
  SetCalPos(Data_HOST);
  cijkl.Init(4,3,3,3,3); cijkl=0.f;
  cijkl(0,0,0,0) =C00; cijkl(1,1,1,1) =C00; cijkl(2,2,2,2) =C00;
  cijkl(0,0,1,1) =C01; cijkl(1,1,2,2) =C01; cijkl(2,2,0,0) =C01;
  cijkl(0,1,0,1) =C33; cijkl(1,2,1,2) =C33; cijkl(2,0,2,0) =C33;
  cijkl(1,1,0,0) =C01; cijkl(2,2,1,1) =C01; cijkl(0,0,2,2) =C01;
  cijkl(1,0,1,0) =C33; cijkl(0,1,1,0) =C33; cijkl(1,0,0,1) =C33;
  cijkl(2,1,2,1) =C33; cijkl(2,1,1,2) =C33; cijkl(1,2,2,1) =C33;
  cijkl(0,2,0,2) =C33; cijkl(0,2,2,0) =C33; cijkl(2,0,0,2) =C33;	
  //*
  cijkl(1-1,1-1,1-1,1-1) =5.39  ;//c11
  cijkl(2-1,2-1,2-1,2-1) =5.39  ;//c11
  cijkl(3-1,3-1,3-1,3-1) =5.22  ;//c11
  cijkl(1-1,1-1,2-1,2-1) =3.39  ;//c12
  cijkl(2-1,2-1,3-1,3-1) =3.56  ;//c12
  cijkl(3-1,3-1,1-1,1-1) =3.56  ;//c12
  cijkl(1-1,2-1,1-1,2-1) =0.6   ;//c44
  cijkl(2-1,3-1,2-1,3-1) =0.77  ;//c44
  cijkl(3-1,1-1,3-1,1-1) =0.77  ;//c44

  cijkl(2-1,2-1,1-1,1-1) =3.39  ;//c12
  cijkl(3-1,3-1,2-1,2-1) =3.56  ;//c12
  cijkl(1-1,1-1,3-1,3-1) =3.56  ;//c12
  cijkl(2-1,1-1,2-1,1-1) =0.60  ;//c44
  cijkl(1-1,2-1,2-1,1-1) =0.60  ;//c44
  cijkl(2-1,1-1,1-1,2-1) =0.60  ;//c44
  cijkl(3-1,2-1,3-1,2-1) =0.77  ;//c44
  cijkl(3-1,2-1,2-1,3-1) =0.77  ;//c44
  cijkl(2-1,3-1,3-1,2-1) =0.77  ;//c44
  cijkl(1-1,3-1,1-1,3-1) =0.77  ;//c44
  cijkl(1-1,3-1,3-1,1-1) =0.77  ;//c44
  cijkl(3-1,1-1,1-1,3-1) =0.77  ;//c44
  // */

  cijkl.HostToDevice(); // main cal is on device

  sigma=0.f;
  for (int sa=0; sa<variantN1+variantN2; sa++){
    for (int i=0;i<3;i++)
	 for (int j=0;j<3;j++){
	   for (int k=0;k<3;k++)
		for (int l=0;l<3;l++){
		  sigma(sa,i,j)+=cijkl(i,j,k,l)*tensor(sa,k,l);
		}
	 }
  }
  sigma.HostToDevice();

  //-------------------------------------------------------------------
  //omega
  omega.Init(2,3,3,Data_HOST_DEV);
  iomega.Init(2,3,3,Data_HOST_DEV);
  temp.Init(2,3,3,Data_HOST_DEV);
  for (int ix=0;ix<lx;ix++)
    for (int iy=0;iy<ly;iy++)
	  for (int iz=0;iz<lz;iz++){

		for (int i=0;i<3;i++)
		  for (int j=0;j<3;j++){
			iomega(i,j)=0.0;
			for (int k=0;k<3;k++){
			  for (int l=0;l<3;l++){
				iomega(i,j) = iomega(i,j) + cijkl(i,k,l,j)*unitVector(ix,iy,iz,k)*unitVector(ix,iy,iz,l);
			  }
			}
		  }

		if (_gSquare(ix,iy,iz)==0){ omega=0;}
		else GaussCMInverse(iomega.Arr,temp.Arr,omega.Arr,3);

		Real term=0;
		for (int sa=0; sa<variantN1; sa++){
		  for (int sap=0; sap<variantN2; sap++){
			int index[5]={sa,sap,ix,iy,iz};
			Part(index) =0.0f;
			for (int i=0;i<3;i++)
			  for (int j=0;j<3;j++)
				for (int k=0;k<3;k++)
				  for (int l=0;l<3;l++){ // (/ 31.4 (sqrt 1001)) (/ 3.92 (sqrt 1001))
					term=-unitVector(ix,iy,iz,i)*sigma(sa,i,j)*omega(j,k)*sigma(sap+variantN1,k,l)*unitVector(ix,iy,iz,l);
					Part(index) +=
					  cijkl(i,j,k,l)*tensor(sa,i,j)*tensor(sap+variantN1,k,l)
					  +term;
				  }
		  }
		}
	  }
  HostToDevice();
  //_g.HostToDevice();
  //_gSquare.HostToDevice();
  //unitVector.HostToDevice();
  return 0;
  
}






