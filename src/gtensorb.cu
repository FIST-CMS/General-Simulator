#include "hip/hip_runtime.h"

#define DEBUG 0
#include"../include/datamain.th"
#include"gtensorb.h"
using namespace GUPS_NS;
using namespace DATA_NS;


__host__ __device__ real GTensorB_VF(int x,int n,float dx){
  int tem; if (x<n/2) tem=x; else tem=x-n;
  return 2.0f*3.14f*dx/n*tem; 
}

__global__ void unitVector_gtensorb_kernel(Real *_gSquare, Real *unitVector,real dx, real dy, real dz){
  int x=blockIdx.x, y=blockIdx.y,z=threadIdx.x;
  int nx=gridDim.x, ny=gridDim.y,nz=blockDim.x;
  int tid=(x*ny+y)*nz+z; //tid =(x,y,z);
  int tidd=tid*3; // tidd+d = (x,y,z,d); d:0 1 2
  Real gx,gy,gz,mo;
  gx=GTensorB_VF(x,nx,dx);
  gy=GTensorB_VF(y,ny,dy);
  gz=GTensorB_VF(z,nz,dz);
  _gSquare[tid]=(gx^2)+(gy^2)+(gz^2);
  mo = sqrt(_gSquare[tid]);
  if (mo > 0.0000005){
	unitVector[tidd+0]=gx/mo;
	unitVector[tidd+1]=gy/mo;
	unitVector[tidd+2]=gz/mo;
  }else{
	unitVector[tidd+0]=0;
	unitVector[tidd+1]=0;
	unitVector[tidd+2]=0;
  }
}

int  GTensorB::UnitVector(real dx,real dy, real dz){
  int nx= unitVector.Dimension[1], ny= unitVector.Dimension[2],
	nz= unitVector.Dimension[3];
  dim3 bn(nx,ny),tn(nz);
  unitVector_gtensorb_kernel<<<bn,tn>>>
	(_gSquare.Arr_dev, unitVector.Arr_dev,
	 dx,dy,dz);
  return 0;
}

__host__ __device__ int gtensorb_Inverse3x3(real *m1, real *m2){ // mat2 = Inverse(mat2)  matrix shape 3x3
  real det_m1;
  det_m1=
	-m1[2]*m1[4]*m1[6] + m1[1]*m1[5]*m1[6] + m1[2]*m1[3]*m1[7] - 
	m1[0]*m1[5]*m1[7] - m1[1]*m1[3]*m1[8] + m1[0]*m1[4]*m1[8];
  if (det_m1>=0.00000001f||det_m1<=-0.00000001f){
	m2[0]=( -m1[5]*m1[7] + m1[4]*m1[8])/det_m1; 
	m2[1]=( m1[2]*m1[7] - m1[1]*m1[8])/det_m1;
	m2[2]=( -m1[2]*m1[4] + m1[1]*m1[5])/det_m1;
	m2[3]=( m1[5]*m1[6] - m1[3]*m1[8])/det_m1;
	m2[4]=( -m1[2]*m1[6] + m1[0]*m1[8])/det_m1; 
	m2[5]=( m1[2]*m1[3] - m1[0]*m1[5])/det_m1;
	m2[6]=( -m1[4]*m1[6] + m1[3]*m1[7])/det_m1; 
	m2[7]=( m1[1]*m1[6] - m1[0]*m1[7])/det_m1;
	m2[8]=( -m1[1]*m1[3] + m1[0]*m1[4])/det_m1;
  }else{
	for (int i=0; i<9; i++)
	  m2[i]=0.f;
  }
  return 0;
}
__global__ void gtensorb_kernel_calculate
(
 Real *B,
 Real*modulus,Real*sigma,Real*tensor,
 Real*unitVector,
 int VariantN1,int VariantN2
 );

int GTensorB::InitB (int variantN1,int variantN2,
					int lx, int ly, int lz,
					real a1, real a2, real a3,
					 Data<Real> &tensor, Data<Real>&modulus ){

  Init(5,variantN1, variantN2, lx, ly, lz,Data_HOST_DEV);
  _gSquare.Init(3,lx,ly,lz,Data_HOST_DEV);
  unitVector.Init(4,lx,ly,lz,3,Data_HOST_DEV);
  ///////////////////////////////////////////////////////
  UnitVector(a1,a2,a3);
  _gSquare.DeviceToHost(); // ..
  unitVector.DeviceToHost(); // ..
  ////////////////////////////////////////////
  //stress  sigma(sa,j,i)
  Data<Real> sigma;
  sigma.Init(3,variantN1+variantN2,3,3,Data_HOST_DEV);
  sigma=0.f;
  for (int sa=0; sa<variantN1+variantN2; sa++){
    for (int i=0;i<3;i++)
	 for (int j=0;j<3;j++){
	   for (int k=0;k<3;k++)
		for (int l=0;l<3;l++){
		  sigma(sa,i,j)+=modulus(i,j,k,l)*tensor(sa,k,l);
		}
	 }
  }
  sigma.HostToDevice();
  tensor.HostToDevice();
  modulus.HostToDevice();
  //-------------------------------------------------------------------
  ////////////////////////////////////////////////////////////////////
  dim3 bn(lx,ly,1), tn(lz,1,1);
  gtensorb_kernel_calculate<<<bn,tn>>>
	(
	 Arr_dev,
	 modulus.Arr_dev,sigma.Arr_dev,tensor.Arr_dev,
	 unitVector.Arr_dev,
	 variantN1,variantN2
	 );

  DeviceToHost();
  return 0;
  
}

__global__ void gtensorb_kernel_calculate
(
 Real *B,
 Real *modulus,Real *sigma,Real *tensor,
 Real *unitVector,
 int variantN1,int variantN2
 ){
  int nx= gridDim.x,  ny= gridDim.y,  nz=blockDim.x;
  int ix= blockIdx.x, iy= blockIdx.y, iz=threadIdx.x;
  Real  omega[3][3];
  Real  iomega[3][3];

  /////////////////
  for (int i=0;i<3;i++)
	for (int j=0;j<3;j++){
	  iomega[i][j]=0.0;
	  for (int k=0;k<3;k++){
		for (int l=0;l<3;l++){
		  iomega[i][j] = iomega[i][j]
			+( modulus[((i*3+k)*3+l)*3+j]
			   * unitVector[((ix*ny+iy)*nz+iz)*3+k]
			   * unitVector[((ix*ny+iy)*nz+iz)*3+l]
			   );
		}
	  }
	}
  ///////////////
  gtensorb_Inverse3x3((float*)iomega,(float*)omega);
  /////////////////////
  Real term=0;
  for (int sa=0; sa<variantN1; sa++){
	for (int sap=0; sap<variantN2; sap++){
	  B[((((sa*variantN2+sap)*nx+ix)*ny+iy)*nz+iz)]=0.0f;
	  for (int i=0;i<3;i++)
		for (int j=0;j<3;j++)
		  for (int k=0;k<3;k++)
			for (int l=0;l<3;l++){ 
			  term=
				-unitVector[((ix*ny+iy)*nz+iz)*3+i]
				*sigma[(sa*3+i)*3+j]
				*omega[j][k]
				*sigma[((sap+variantN1)*3+k)*3+l]
				*unitVector[((ix*ny+iy)*nz+iz)*3+l];
			  B[((((sa*variantN2+sap)*nx+ix)*ny+iy)*nz+iz)]+=
				modulus[((i*3+j)*3+k)*3+l]// an error
				*tensor[(sa*3+i)*3+j]
				*tensor[((sap+variantN1)*3+k)*3+l]
				+term;
			}
	}
  }
}

